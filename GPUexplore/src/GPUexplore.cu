#include "hip/hip_runtime.h"
/*
 ============================================================================
 Name        : GPUexplore.cu
 Author      : Anton Wijs and Thomas Neele
 Version     :
 Copyright   : Copyright Anton Wijs and Thomas Neele
 Description : CUDA GPUexplore: On the fly state space analysis
 ============================================================================
 */

#include <stdio.h>
#include <stdlib.h>
#include <stdint.h>
#include <string.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <assert.h>
#include <time.h>
#include <math.h>

// type of elements used
#define inttype uint32_t
// type of indices in hash table
#define indextype uint64_t

enum BucketEntryStatus { EMPTY, TAKEN, FOUND };
enum PropertyStatus { NONE, DEADLOCK, SAFETY, LIVENESS };

#define MIN(a,b) \
   ({ __typeof__ (a) _a = (a); \
       __typeof__ (b) _b = (b); \
     _a < _b ? _a : _b; })

#define MAX(a,b) \
   ({ __typeof__ (a) _a = (a); \
       __typeof__ (b) _b = (b); \
     _a > _b ? _a : _b; })

// Nr of tiles processed in single kernel launch
//#define TILEITERS 10

static const int WARPSIZE = 32;
static const int HALFWARPSIZE = 16;
static const int INTSIZE = 32;
static const int BUFFERSIZE = 50;

// GPU constants
__constant__ inttype d_nrbuckets;
__constant__ inttype d_shared_q_size;
__constant__ inttype d_nr_procs;
__constant__ inttype d_max_buf_ints;
__constant__ inttype d_sv_nints;
__constant__ inttype d_bits_act;
__constant__ inttype d_nbits_offset;
__constant__ inttype d_kernel_iters;
__constant__ inttype d_nbits_syncbits_offset;
__constant__ PropertyStatus d_property;
__constant__ inttype d_apply_por;
__constant__ inttype d_check_cycle_proviso;

// GPU shared memory array
extern __shared__ inttype shared[];

// thread ids
#define WARP_ID							(threadIdx.x / WARPSIZE)
#define GLOBAL_WARP_ID					(((blockDim.x / WARPSIZE)*blockIdx.x)+WARP_ID)
#define NR_WARPS						((blockDim.x / WARPSIZE)*gridDim.x)
#define LANE							(threadIdx.x % WARPSIZE)
#define HALFLANE						(threadIdx.x % HALFWARPSIZE)
//#define ENTRY_ID						(LANE % d_sv_nints)
#define ENTRY_ID						(HALFLANE % d_sv_nints)
#define GROUP_ID						(LANE % d_nr_procs)
#define GROUP_GID						(WARP_ID * GROUPS_PER_WARP + LANE / d_nr_procs)
#define NR_GROUPS						((blockDim.x / WARPSIZE) * GROUPS_PER_WARP)
#define GROUPS_PER_WARP                 (WARPSIZE / d_nr_procs)
// Group id to lane and lane to group id macros
#define GTL(i)							(LANE - GROUP_ID + (i))
#define LTG(i)							((i) - (LANE - GROUP_ID))

//#define NREL_IN_BUCKET					((WARPSIZE / d_sv_nints))
#define NREL_IN_BUCKET					((HALFWARPSIZE / d_sv_nints)*2)
#define NREL_IN_BUCKET_HOST				((HALFWARPSIZE / sv_nints)*2)

// constant for cuckoo hashing (Alcantara et al)
static const inttype P = 979946131;
// Retry constant to determine number of retries for element insertion
#define RETRYFREQ 7
#define NR_HASH_FUNCTIONS 8
// Number of retries in local cache
#define CACHERETRYFREQ 20
// Maximum size of state vectors (in nr. of 32-bit integers)
#define MAX_SIZE 9
// Empty state vectors
static const inttype EMPTYVECT32 = 0x7FFFFFFF;
// Constant to indicate that no more work is required
# define EXPLORATION_DONE 0x7FFFFFFF
// offset in shared memory from which loaded data can be read
static const int SH_OFFSET = 5;
//static const int KERNEL_ITERS = 10;
//static const int NR_OF_BLOCKS = 3120;
//static const int BLOCK_SIZE = 512;
static const int KERNEL_ITERS = 1;
static const int NR_OF_BLOCKS = 1;
static const int BLOCK_SIZE = 32;
const size_t Mb = 1<<20;

// test macros
#define PRINTTHREADID()						{printf("Hello thread %d\n", (blockIdx.x*blockDim.x)+threadIdx.x);}
#define PRINTTHREAD(j, i)					{printf("%d: Seen by thread %d: %d\n", (j), (blockIdx.x*blockDim.x)+threadIdx.x, (i));}

// Offsets calculations for shared memory arrays
#define HASHCONSTANTSLEN				(2*NR_HASH_FUNCTIONS)
#define VECTORPOSLEN					(d_nr_procs+1)
#define LTSSTATESIZELEN					(d_nr_procs)
#define OPENTILELEN						(d_sv_nints*NR_GROUPS)
#define LASTSEARCHLEN					(blockDim.x/WARPSIZE)
#define TGTSTATELEN						(blockDim.x*d_sv_nints)
#define THREADBUFFERLEN					(NR_GROUPS*(THREADBUFFERSHARED+(d_nr_procs*d_max_buf_ints)))

#define HASHCONSTANTSOFFSET 			(SH_OFFSET)
#define VECTORPOSOFFSET 				(HASHCONSTANTSOFFSET+HASHCONSTANTSLEN)
#define LTSSTATESIZEOFFSET 				(VECTORPOSOFFSET+VECTORPOSLEN)
#define OPENTILEOFFSET 					(LTSSTATESIZEOFFSET+LTSSTATESIZELEN)
#define LASTSEARCHOFFSET				(OPENTILEOFFSET+OPENTILELEN)
#define TGTSTATEOFFSET		 			(LASTSEARCHOFFSET+LASTSEARCHLEN)
#define THREADBUFFEROFFSET	 			(TGTSTATEOFFSET+TGTSTATELEN)
#define CACHEOFFSET 					(THREADBUFFEROFFSET+THREADBUFFERLEN)

// One int for sync action counter
// One int for POR counter
#define THREADBUFFERSHARED				2
// parameter is thread id
#define THREADBUFFERGROUPSTART(i)		(THREADBUFFEROFFSET+ (((i) / WARPSIZE)*GROUPS_PER_WARP+(((i) % WARPSIZE) / d_nr_procs)) * (THREADBUFFERSHARED+(d_nr_procs*d_max_buf_ints)))
// parameter is group id
#define THREADBUFFERGROUPPOS(i, j)		shared[THREADBUFFERGROUPSTART(threadIdx.x)+THREADBUFFERSHARED+((i)*d_max_buf_ints)+(j)]
#define THREADGROUPCOUNTER				shared[(THREADBUFFERGROUPSTART(threadIdx.x))]
#define THREADGROUPPOR					shared[(THREADBUFFERGROUPSTART(threadIdx.x)) + 1]
#define OPENTILESTATEPART(i)			shared[OPENTILEOFFSET+(d_sv_nints*((WARP_ID*GROUPS_PER_WARP+(LANE / d_nr_procs))))+(i)]

#define THREADINGROUP					(LANE < (GROUPS_PER_WARP)*d_nr_procs)

#define STATESIZE(i)					(shared[LTSSTATESIZEOFFSET+(i)])
#define VECTORSTATEPOS(i)				(shared[VECTORPOSOFFSET+(i)])
#define NR_OF_STATES_IN_TRANSENTRY(i)	((31 - d_bits_act) / shared[LTSSTATESIZEOFFSET+(i)])
// SM local progress flags
#define ITERATIONS						(shared[0])
#define CONTINUE						(shared[1])
#define OPENTILECOUNT					(shared[2])
#define WORKSCANRESULT					(shared[3])
#define SCAN							(shared[4])

// BIT MANIPULATION MACROS

#define SETBIT(i, x)							{(x) = ((1L<<(i)) | (x));}
#define GETBIT(i, x)							(((x) >> (i)) & 1L)
#define SETBITS(i, j, x)						{(x) = (x) | (((1L<<(j))-1)^((1L<<(i))-1));}
#define GETPROCTRANSACT(a, t)					{bitmask = 0; SETBITS(1, 1+d_bits_act, bitmask); (a) = ((t) & bitmask) >> 1;}
#define GETPROCTRANSSYNC(a, t)					{(a) = ((t) & 1);}
#define GETPROCTRANSSTATE(a, t, i, j)			{bitmask = (1 << shared[LTSSTATESIZEOFFSET+(j)]) - 1; \
												 (a) = ((t) >> 1+d_bits_act+(i)*shared[LTSSTATESIZEOFFSET+(j)]) & bitmask; \
												}
#define GETTRANSOFFSET(a, t, i)					{bitmask = 0; SETBITS((i)*d_nbits_offset, ((i)+1)*d_nbits_offset, bitmask); (a) = ((t) & bitmask) >> ((i)*d_nbits_offset);}
#define GETSYNCOFFSET(a, t, i)					{bitmask = 0; SETBITS((i)*d_nbits_syncbits_offset, ((i)+1)*d_nbits_syncbits_offset, bitmask); \
													(a) = ((t) & bitmask) >> ((i)*d_nbits_syncbits_offset);}
#define GETSTATEVECTORSTATE(a, t, i)			{bitmask = 0; 	if (shared[VECTORPOSOFFSET+(i)]/INTSIZE == (shared[VECTORPOSOFFSET+(i)+1]-1)/INTSIZE) { \
																	SETBITS((shared[VECTORPOSOFFSET+(i)] % INTSIZE), \
																			(((shared[VECTORPOSOFFSET+(i)+1]-1) % INTSIZE)+1), bitmask); \
																	(a) = ((t)[shared[VECTORPOSOFFSET+(i)]/INTSIZE] & bitmask) >> (shared[VECTORPOSOFFSET+(i)] % INTSIZE); \
																} \
																else { \
																	SETBITS(0,(shared[VECTORPOSOFFSET+(i)+1] % INTSIZE),bitmask); \
																	(a) = (t)[shared[VECTORPOSOFFSET+(i)]/INTSIZE] >> (shared[VECTORPOSOFFSET+(i)] % INTSIZE) \
																		 | \
																		((t)[shared[VECTORPOSOFFSET+(i)+1]/INTSIZE] & bitmask) << \
																		(INTSIZE - (shared[VECTORPOSOFFSET+(i)] % INTSIZE)); \
																} \
												}
#define SETPROCTRANSACT(t, x)					{bitmask = 0; SETBITS(1, d_bits_act+1,bitmask); (t) = ((t) & ~bitmask) | ((x) << 1);}
#define SETPROCTRANSSYNC(t, x)					{(t) = ((t) & ~1) | (x);}
#define SETPROCTRANSSTATE(t, i, x, j)			{bitmask = 0; SETBITS(1+d_bits_act+((i)-1)*shared[LTSSTATESIZEOFFSET+(j)],1+d_bits_act+(i)*shared[LTSSTATESIZEOFFSET+(j)],bitmask); \
													(t) = ((t) & ~bitmask) | ((x) << (1+d_bits_act+((i)-1)*shared[LTSSTATESIZEOFFSET+(j)]));}
#define SETSTATEVECTORSTATE(t, i, x)			{bitmask = 0; 	if (shared[VECTORPOSOFFSET+(i)]/INTSIZE == (shared[VECTORPOSOFFSET+(i)+1]-1)/INTSIZE) { \
																	SETBITS((shared[VECTORPOSOFFSET+(i)] % INTSIZE), \
																			(((shared[VECTORPOSOFFSET+(i)+1]-1) % INTSIZE)+1),bitmask); \
																	(t)[shared[VECTORPOSOFFSET+(i)]/INTSIZE] = ((t)[shared[VECTORPOSOFFSET+(i)]/INTSIZE] & ~bitmask) | \
																	((x) << (shared[VECTORPOSOFFSET+(i)] % INTSIZE)); \
																} \
																else { \
																	SETBITS(0,(shared[VECTORPOSOFFSET+(i)] % INTSIZE), bitmask); \
																	(t)[shared[VECTORPOSOFFSET+(i)]/INTSIZE] = ((t)[shared[VECTORPOSOFFSET+(i)]/INTSIZE] & bitmask) | \
																	((x) << (shared[VECTORPOSOFFSET+(i)] % INTSIZE)); \
																	bitmask = 0; \
																	SETBITS((shared[VECTORPOSOFFSET+(i)+1] % INTSIZE), INTSIZE, bitmask); \
																	(t)[shared[VECTORPOSOFFSET+(i)+1]/INTSIZE] = ((t)[shared[VECTORPOSOFFSET+(i)+1]/INTSIZE] & bitmask) | \
																		((x) >> (INTSIZE - (shared[VECTORPOSOFFSET+(i)] % INTSIZE))); \
																} \
												}
// NEEDS FIX: USE BIT 32 OF FIRST INTEGER TO INDICATE STATE OR NOT (1 or 0), IN CASE MULTIPLE INTEGERS ARE USED FOR STATE VECTOR!!!
//#define ISSTATE(t)								((t)[(d_sv_nints-1)] != EMPTYVECT32)
#define ISSTATE(t)								((t)[0] != EMPTYVECT32)
#define SETNEWSTATE(t)							{	(t)[(d_sv_nints-1)] = (t)[(d_sv_nints-1)] | 0x80000000;}
#define SETOLDSTATE(t)							{	(t)[(d_sv_nints-1)] = (t)[(d_sv_nints-1)] & 0x7FFFFFFF;}
#define ISNEWSTATE(t)							((t)[(d_sv_nints-1)] >> 31)
#define ISNEWSTATE_HOST(t)						((t)[(sv_nints-1)] >> 31)
#define ISNEWINT(t)								((t) >> 31)
#define OLDINT(t)								((t) & 0x7FFFFFFF)
#define NEWINT(t)								((t) | 0x80000000)

#define SETPORSTATE(t)							{	(t)[(d_sv_nints-1)] = (t)[(d_sv_nints-1)] | 0x40000000;}
#define SETOTHERSTATE(t)						{	(t)[(d_sv_nints-1)] = (t)[(d_sv_nints-1)] & 0xBFFFFFFF;}
#define ISPORSTATE(t)							(ISPORINT((t)[(d_sv_nints-1)))
#define ISPORSTATE_HOST(t)						(ISPORINT((t)[(sv_nints-1)))
#define ISPORINT(t)								(((t) & 0x40000000) >> 30)
#define OTHERINT(t)								((t) & 0xBFFFFFFF)
#define PORINT(t)								((t) | 0x40000000)

#define STATE_FLAGS_MASK                        0x3FFFFFFF
#define STRIPSTATE(t)							{(t)[(d_sv_nints-1)] = (t)[(d_sv_nints-1)] & STATE_FLAGS_MASK;}
#define STRIPPEDSTATE(t, i)						((i == d_sv_nints-1) ? ((t)[i] & STATE_FLAGS_MASK) : (t)[i])
#define STRIPPEDENTRY(t, i)						((i == d_sv_nints-1) ? ((t) & STATE_FLAGS_MASK) : (t))
#define STRIPPEDENTRY_HOST(t, i)				((i == sv_nints-1) ? ((t) & STATE_FLAGS_MASK) : (t))
#define NEWSTATEPART(t, i)						(((i) == d_sv_nints-1) ? ((t)[d_sv_nints-1] | 0x80000000) : (t)[(i)])
#define COMPAREENTRIES(t1, t2)					(((t1) & STATE_FLAGS_MASK) == ((t2) & STATE_FLAGS_MASK))
#define OWNSSYNCRULE(a, t, i)					{if (GETBIT((i),(t))) { \
													bitmask = 0; SETBITS(0,(i),bitmask); if ((t & bitmask) > 0) {(a) = 0;} else {(a) = 1;}} \
													else {(a) = 0;}}
#define GETSYNCRULE(a, t, i)					{bitmask = 0; SETBITS((i)*d_nr_procs,((i)+1)*d_nr_procs,bitmask); (a) = ((t) & bitmask) >> ((i)*d_nr_procs);}
#define SYNCRULEISAPPLICABLE(a, t, ac)			{(a) = 1; \
												 for (int rule = (t); rule && a;) { \
													bk = __ffs(rule) - 1; \
													bj = THREADBUFFERGROUPPOS((inttype) bk,0); \
													(a) = bj == 0 ? 0 : ((bj >> 1) & ((1 << d_bits_act) - 1)) == (ac); \
													rule &= ~(1 << bk); \
												 } \
												}

// HASH TABLE MACROS

// Return 0 if not found, bit 2 is flag for new state, bit 3 is a flag for POR state, 8 if cache is full
__device__ inttype STOREINCACHE(inttype* t, inttype* d_q, inttype* address) {
	inttype bi, bj, bk, bl, bitmask;
	indextype hashtmp;
	STRIPSTATE(t);
	hashtmp = 0;
	for (bi = 0; bi < d_sv_nints; bi++) {
		hashtmp += t[bi];
		hashtmp <<= 5;
	}
	bitmask = d_sv_nints*((inttype) (hashtmp % ((d_shared_q_size - CACHEOFFSET) / d_sv_nints)));
	SETNEWSTATE(t);
	bl = 0;
	while (bl < CACHERETRYFREQ) {
		bi = atomicCAS((inttype *) &shared[CACHEOFFSET+bitmask+(d_sv_nints-1)], EMPTYVECT32, t[d_sv_nints-1]);
		if (bi == EMPTYVECT32) {
			for (bj = 0; bj < d_sv_nints-1; bj++) {
				shared[CACHEOFFSET+bitmask+bj] = t[bj];
			}
			*address = bitmask;
			return 0;
		}
		if (COMPAREENTRIES(bi, t[d_sv_nints-1])) {
			if (d_sv_nints == 1) {
				*address = bitmask;
				return 1 + (ISNEWINT(bi) << 1) + (ISPORINT(bi) << 2);
			}
			else {
				for (bj = 0; bj < d_sv_nints-1; bj++) {
					if (shared[CACHEOFFSET+bitmask+bj] != (t)[bj]) {
						break;
					}
				}
				if (bj == d_sv_nints-1) {
					*address = bitmask;
					return 1 + (ISNEWINT(bi) << 1) + (ISPORINT(bi) << 2);
				}
			}
		}
		if (!ISNEWINT(bi)) {
			bj = atomicCAS((inttype *) &shared[CACHEOFFSET+bitmask+(d_sv_nints-1)], bi, t[d_sv_nints-1]);
			if (bi == bj) {
				for (bk = 0; bk < d_sv_nints-1; bk++) {
					shared[CACHEOFFSET+bitmask+bk] = t[bk];
				}
				*address = bitmask;
				return 0;
			}
		}
		bl++;
		bitmask += d_sv_nints;
		if ((bitmask+(d_sv_nints-1)) >= (d_shared_q_size - CACHEOFFSET)) {
			bitmask = 0;
		}
	}
	return 8;
}

// Mark the state in the cache according to markNew
// This function takes POR bit into account
__device__ void MARKINCACHE(inttype* t, inttype* d_q, int markNew) {
	inttype bi, bj, bl, bitmask;
	indextype hashtmp;
	STRIPSTATE(t);
	hashtmp = 0;
	for (bi = 0; bi < d_sv_nints; bi++) {
		hashtmp += t[bi];
		hashtmp <<= 5;
	}
	bitmask = d_sv_nints*((inttype) (hashtmp % ((d_shared_q_size - CACHEOFFSET) / d_sv_nints)));
	SETNEWSTATE(t);
	bl = 0;
	while (bl < CACHERETRYFREQ) {
		bi = shared[CACHEOFFSET+bitmask+(d_sv_nints-1)];
		if (COMPAREENTRIES(bi, t[d_sv_nints-1])) {
			for (bj = 0; bj < d_sv_nints-1; bj++) {
				if (shared[CACHEOFFSET+bitmask+bj] != (t)[bj]) {
					break;
				}
			}
			if (bj == d_sv_nints-1) {
				if(markNew) {
					shared[CACHEOFFSET+bitmask+(d_sv_nints-1)] = NEWINT(OTHERINT(shared[CACHEOFFSET+bitmask+(d_sv_nints-1)] & STATE_FLAGS_MASK));
				} else if(ISPORINT(bi) && ISNEWINT(bi)){
					atomicCAS((inttype*) &shared[CACHEOFFSET+bitmask+(d_sv_nints-1)], bi, OLDINT(bi));
				}
				return;
			}
		}
		bl++;
		bitmask += d_sv_nints;
		if ((bitmask+(d_sv_nints-1)) >= (d_shared_q_size - CACHEOFFSET)) {
			bitmask = 0;
		}
	}
}

// hash functions use bj variable
#define FIRSTHASH(a, t)							{	hashtmp = 0; \
													for (bj = 0; bj < d_sv_nints; bj++) { \
														hashtmp += STRIPPEDSTATE(t,bj); \
														hashtmp <<= 5; \
													} \
													hashtmp = (indextype) (d_h[0]*hashtmp+d_h[1]); \
													(a) = WARPSIZE*((inttype) ((hashtmp % P) % d_nrbuckets)); \
												}
#define FIRSTHASHHOST(a)						{	indextype hashtmp = 0; \
													hashtmp = (indextype) h[1]; \
													(a) = WARPSIZE*((inttype) ((hashtmp % P) % q_size/WARPSIZE)); \
												}
#define HASHALL(a, i, t)						{	hashtmp = 0; \
													for (bj = 0; bj < d_sv_nints; bj++) { \
														hashtmp += STRIPPEDSTATE(t,bj); \
														hashtmp <<= 5; \
													} \
													hashtmp = (indextype) (shared[HASHCONSTANTSOFFSET+(2*(i))]*(hashtmp)+shared[HASHCONSTANTSOFFSET+(2*(i))+1]); \
													(a) = WARPSIZE*((inttype) ((hashtmp % P) % d_nrbuckets)); \
												}
#define HASHFUNCTION(a, i, t)					((HASHALL((a), (i), (t))))

#define COMPAREVECTORS(a, t1, t2)				{	(a) = 1; \
													for (bk = 0; bk < d_sv_nints-1; bk++) { \
														if ((t1)[bk] != (t2)[bk]) { \
															(a) = 0; break; \
														} \
													} \
													if ((a)) { \
														if (STRIPPEDSTATE((t1),bk) != STRIPPEDSTATE((t2),bk)) { \
															(a) = 0; \
														} \
													} \
												}

// check if bucket element associated with lane is a valid position to store data
#define LANEPOINTSTOVALIDBUCKETPOS						(HALFLANE < ((HALFWARPSIZE / d_sv_nints)*d_sv_nints))

__device__ inttype LANE_POINTS_TO_EL(inttype i)	{
	if (i < HALFWARPSIZE / d_sv_nints) {
		return (LANE >= i*d_sv_nints && LANE < (i+1)*d_sv_nints);
	}
	else {
		return (LANE >= HALFWARPSIZE+(i-(HALFWARPSIZE / d_sv_nints))*d_sv_nints && LANE < HALFWARPSIZE+(i-(HALFWARPSIZE / d_sv_nints)+1)*d_sv_nints);
	}
}

// start position of element i in bucket
#define STARTPOS_OF_EL_IN_BUCKET(i)			((i < (HALFWARPSIZE / d_sv_nints)) ? (i*d_sv_nints) : (HALFWARPSIZE + (i-(HALFWARPSIZE/d_sv_nints))*d_sv_nints))
#define STARTPOS_OF_EL_IN_BUCKET_HOST(i)	((i < (HALFWARPSIZE / sv_nints)) ? (i*sv_nints) : (HALFWARPSIZE + (i-(HALFWARPSIZE/sv_nints))*sv_nints))

// find or put element, single thread version.
__device__ inttype FINDORPUT_SINGLE(inttype* t, inttype* d_q, volatile inttype* d_newstate_flags) {
	inttype bi, bj, bk, bl;
	indextype hashtmp;
	for (bi = 0; bi < NR_HASH_FUNCTIONS; bi++) {
		HASHFUNCTION(hashtmp, bi, t);
		for (bj = 0; bj < NREL_IN_BUCKET; bj++) {
			bl = d_q[hashtmp+STARTPOS_OF_EL_IN_BUCKET(bj)+(d_sv_nints-1)];
			if (bl == EMPTYVECT32) {
				bl = atomicCAS(&d_q[hashtmp+STARTPOS_OF_EL_IN_BUCKET(bj)+(d_sv_nints-1)], EMPTYVECT32, t[d_sv_nints-1]);
				if (bl == EMPTYVECT32) {
					// Write was successful
					if (d_sv_nints > 1) {
						for (bk = 0; bk < d_sv_nints-1; bk++) {
							d_q[hashtmp+STARTPOS_OF_EL_IN_BUCKET(bj)+bk] = t[bk];
						}
					}
					__threadfence();
					// There is work available for some block
					d_newstate_flags[(hashtmp / blockDim.x) % gridDim.x] = 1;
				}
			}
			if (bl != EMPTYVECT32) {
				COMPAREVECTORS(bk, &d_q[hashtmp+STARTPOS_OF_EL_IN_BUCKET(bj)], t);
				if (bk == 1) {
					// Found state in global memory
					return 1;
				}
			}
			else {
				SETOLDSTATE(t);
				return 1;
			}
		}
	}
	return 0;
}

// find or put element, warp version. t is element stored in block cache
__device__ inttype FINDORPUT_WARP(inttype* t, inttype* d_q, volatile inttype* d_newstate_flags)	{
	inttype bi, bj, bk, bl, bitmask;
	indextype hashtmp;
	BucketEntryStatus threadstatus;
	// prepare bitmask once to reason about results of threads in the same (state vector) group
	bitmask = 0;
	if (LANEPOINTSTOVALIDBUCKETPOS) {
		SETBITS(LANE-ENTRY_ID, LANE-ENTRY_ID+d_sv_nints, bitmask);
	}
	for (bi = 0; bi < NR_HASH_FUNCTIONS; bi++) {
		HASHFUNCTION(hashtmp, bi, t);
		bl = d_q[hashtmp+LANE];
		bk = __ballot(STRIPPEDENTRY(bl, ENTRY_ID) == STRIPPEDSTATE(t, ENTRY_ID));
		// threadstatus is used to determine whether full state vector has been found
		threadstatus = EMPTY;
		if (LANEPOINTSTOVALIDBUCKETPOS) {
			if ((bk & bitmask) == bitmask) {
				threadstatus = FOUND;
			}
		}
		if (__ballot(threadstatus == FOUND) != 0) {
			// state vector has been found in bucket. mark local copy as old.
			if (LANE == 0) {
				SETOLDSTATE(t);
			}
			return 1;
		}
		// try to find empty position to insert new state vector
		threadstatus = (bl == EMPTYVECT32 && LANEPOINTSTOVALIDBUCKETPOS) ? EMPTY : TAKEN;
		// let bk hold the smallest index of an available empty position
		bk = __ffs(__ballot(threadstatus == EMPTY));
		while (bk != 0) {
			// write the state vector
			bk--;
			if (LANE >= bk && LANE < bk+d_sv_nints) {
				bl = atomicCAS(&(d_q[hashtmp+LANE]), EMPTYVECT32, t[ENTRY_ID]);
				if (bl == EMPTYVECT32) {
					// success
					if (ENTRY_ID == d_sv_nints-1) {
						SETOLDSTATE(t);
					}
					// try to claim the state vector for future work
					bl = OPENTILELEN;
					if (ENTRY_ID == d_sv_nints-1) {
						// try to increment the OPENTILECOUNT counter
						bl = atomicAdd((inttype *) &OPENTILECOUNT, d_sv_nints);
						if (bl < OPENTILELEN) {
							d_q[hashtmp+LANE] = t[d_sv_nints-1];
						} else {
							// There is work available for some block
							__threadfence();
							d_newstate_flags[(hashtmp / blockDim.x) % gridDim.x] = 1;
						}
					}
					// all active threads read the OPENTILECOUNT value of the first thread, and possibly store their part of the vector in the shared memory
					bl = __shfl(bl, LANE-ENTRY_ID+d_sv_nints-1);
					if (bl < OPENTILELEN) {
						// write part of vector to shared memory
						shared[OPENTILEOFFSET+bl+ENTRY_ID] = NEWSTATEPART(t, ENTRY_ID);
					}
					// write was successful. propagate this to the whole warp by setting threadstatus to FOUND
					threadstatus = FOUND;
				}
				else {
					// write was not successful. check if the state vector now in place equals the one we are trying to insert
					bk = __ballot(STRIPPEDENTRY(bl, ENTRY_ID) == STRIPPEDSTATE(t, ENTRY_ID));
					if ((bk & bitmask) == bitmask) {
						// state vector has been found in bucket. mark local copy as old.
						if (LANE == bk) {
							SETOLDSTATE(t);
						}
						// propagate this result to the whole warp
						threadstatus = FOUND;
					}
					else {
						// state vector is different, and position in bucket is taken
						threadstatus = TAKEN;
					}
				}
			}
			// check if the state vector was either encountered or inserted
			if (__ballot(threadstatus == FOUND) != 0) {
				return 1;
			}
			// recompute bk
			bk = __ffs(__ballot(threadstatus == EMPTY));
		}
	}
	return 0;
}

// find element, warp version. t is element stored in block cache
// return 0 if not found or found and new, 1 if found and old
__device__ inttype FIND_WARP(inttype* t, inttype* d_q)	{
	inttype bi, bj, bk, bl, bitmask;
	indextype hashtmp;
	BucketEntryStatus threadstatus;
	// prepare bitmask once to reason about results of threads in the same (state vector) group
	bitmask = 0;
	if (LANEPOINTSTOVALIDBUCKETPOS) {
		SETBITS(LANE-ENTRY_ID, LANE-ENTRY_ID+d_sv_nints, bitmask);
	}
	for (bi = 0; bi < NR_HASH_FUNCTIONS; bi++) {
		HASHFUNCTION(hashtmp, bi, t);
		bl = d_q[hashtmp+LANE];
		bk = __ballot(STRIPPEDENTRY(bl, ENTRY_ID) == STRIPPEDSTATE(t, ENTRY_ID));
		// threadstatus is used to determine whether full state vector has been found
		threadstatus = EMPTY;
		if (LANEPOINTSTOVALIDBUCKETPOS) {
			if ((bk & bitmask) == bitmask) {
				threadstatus = FOUND;
			}
		}
		if (__ballot(threadstatus == FOUND) != 0) {
			// state vector has been found in bucket. mark local copy as old.
			if (threadstatus == FOUND & ISNEWINT(bl) == 0 & ENTRY_ID == d_sv_nints - 1) {
				SETOLDSTATE(t);
			}
			SETPORSTATE(t);
			return __ballot(threadstatus == FOUND & ISNEWINT(bl) == 0 & ENTRY_ID == d_sv_nints - 1);
		}
		// try to find empty position
		threadstatus = (bl == EMPTYVECT32 && LANEPOINTSTOVALIDBUCKETPOS) ? EMPTY : TAKEN;
		if(__any(threadstatus == EMPTY)) {
			// There is an empty slot in this bucket and the state vector was not found
			// State will also not be found after rehashing, so we return 0
			SETPORSTATE(t);
			return 0;
		}
	}
	SETPORSTATE(t);
	return 0;
}

// macro to print state vector
#define PRINTVECTOR(s) 							{	printf ("("); \
													for (bk = 0; bk < d_nr_procs; bk++) { \
														GETSTATEVECTORSTATE(bj, (s), bk) \
														printf ("%d", bj); \
														if (bk < (d_nr_procs-1)) { \
															printf (","); \
														} \
													} \
													printf (")\n"); \
												}


int vmem = 0;

// GPU textures
texture<inttype, 1, hipReadModeElementType> tex_proc_offsets_start;
texture<inttype, 1, hipReadModeElementType> tex_proc_offsets;
texture<inttype, 1, hipReadModeElementType> tex_proc_trans_start;
texture<inttype, 1, hipReadModeElementType> tex_proc_trans;
texture<inttype, 1, hipReadModeElementType> tex_syncbits_offsets;
texture<inttype, 1, hipReadModeElementType> tex_syncbits;

/**
 * This macro checks return value of the CUDA runtime call and exits
 * the application if the call failed.
 */
#define CUDA_CHECK_RETURN(value) {											\
	hipError_t _m_cudaStat = value;										\
	if (_m_cudaStat != hipSuccess) {										\
		fprintf(stderr, "Error %s at line %d in file %s\n",					\
				hipGetErrorString(_m_cudaStat), __LINE__, __FILE__);		\
		exit(1);															\
	} }

//wrapper around hipMalloc to count allocated memory and check for error while allocating
int cudaMallocCount ( void ** ptr,int size) {
	hipError_t err = hipSuccess;
	vmem += size;
	err = hipMalloc(ptr,size);
	if (err) {
		printf("Error %s at line %d in file %s\n", hipGetErrorString(err), __LINE__, __FILE__);
		exit(1);
	}
	fprintf (stdout, "allocated %d\n", size);
	return size;
}

//test function to print a given state vector
void print_statevector(FILE* stream, inttype *state, inttype *firstbit_statevector, inttype nr_procs, inttype sv_nints) {
	inttype i, s, bitmask;

	for (i = 0; i < nr_procs; i++) {
		bitmask = 0;
		if (firstbit_statevector[i]/INTSIZE == firstbit_statevector[i+1]/INTSIZE) {
			SETBITS(firstbit_statevector[i] % INTSIZE,firstbit_statevector[i+1] % INTSIZE, bitmask);
			s = (state[firstbit_statevector[i]/INTSIZE] & bitmask) >> (firstbit_statevector[i] % INTSIZE);
		}
		else {
			SETBITS(0, firstbit_statevector[i+1] % INTSIZE, bitmask);
			s = (state[firstbit_statevector[i]/INTSIZE] >> (firstbit_statevector[i] % INTSIZE)
					| (state[firstbit_statevector[i+1]/INTSIZE] & bitmask) << (INTSIZE - (firstbit_statevector[i] % INTSIZE))); \
		}
		fprintf (stream, "%d", s);
		if (i < (nr_procs-1)) {
			fprintf (stream, ",");
		}
	}
	fprintf (stream, " ");
	for (i = 0; i < sv_nints; i++) {
		fprintf (stream, "%d ", STRIPPEDENTRY_HOST(state[i], i));
	}
	fprintf (stream, "\n");
}

//test function to print the contents of the device queue
void print_queue(inttype *d_q, inttype q_size, inttype *firstbit_statevector, inttype nr_procs, inttype sv_nints) {
	inttype *q_test = (inttype*) malloc(sizeof(inttype)*q_size);
	hipMemcpy(q_test, d_q, q_size*sizeof(inttype), hipMemcpyDeviceToHost);
	inttype nw;
	int count = 0;
	int newcount = 0;
	for (inttype i = 0; i < (q_size/WARPSIZE); i++) {
		for (inttype j = 0; j < NREL_IN_BUCKET_HOST; j++) {
			if (q_test[(i*WARPSIZE)+STARTPOS_OF_EL_IN_BUCKET_HOST(j)+(sv_nints-1)] != EMPTYVECT32) {
				count++;
				nw = ISNEWSTATE_HOST(&q_test[(i*WARPSIZE)+STARTPOS_OF_EL_IN_BUCKET_HOST(j)]);
				if (nw) {
					newcount++;
					fprintf (stdout, "new: ");
				}
				print_statevector(stdout, &(q_test[(i*WARPSIZE)+STARTPOS_OF_EL_IN_BUCKET_HOST(j)]), firstbit_statevector, nr_procs, sv_nints);
			}
		}
	}
	fprintf (stdout, "nr. of states in hash table: %d (%d unexplored states)\n", count, newcount);
}

//test function to print the contents of the device queue
void print_local_queue(FILE* stream, inttype *q, inttype q_size, inttype *firstbit_statevector, inttype nr_procs, inttype sv_nints) {
	int count = 0, newcount = 0;
	inttype nw;
	for (inttype i = 0; i < (q_size/WARPSIZE); i++) {
		for (inttype j = 0; j < NREL_IN_BUCKET_HOST; j++) {
			if (q[(i*WARPSIZE)+STARTPOS_OF_EL_IN_BUCKET_HOST(j)+(sv_nints-1)] != EMPTYVECT32) {
				count++;

				nw = ISNEWSTATE_HOST(&q[(i*WARPSIZE)+STARTPOS_OF_EL_IN_BUCKET_HOST(j)]);
				if (nw) {
					newcount++;
					fprintf (stream, "new: ");
				}
				print_statevector(stream, &(q[(i*WARPSIZE)+STARTPOS_OF_EL_IN_BUCKET_HOST(j)]), firstbit_statevector, nr_procs, sv_nints);
			}
		}
	}
	fprintf (stream, "nr. of states in hash table: %d (%d unexplored states)\n", count, newcount);
}

//test function to count the contents of the device queue
void count_queue(inttype *d_q, inttype q_size, inttype *firstbit_statevector, inttype nr_procs, inttype sv_nints) {
	inttype *q_test = (inttype*) malloc(sizeof(inttype)*q_size);
	hipMemcpy(q_test, d_q, q_size*sizeof(inttype), hipMemcpyDeviceToHost);

	int count = 0;
	for (inttype i = 0; i < (q_size/WARPSIZE); i++) {
		for (inttype j = 0; j < NREL_IN_BUCKET_HOST; j++) {
			if (q_test[(i*WARPSIZE)+STARTPOS_OF_EL_IN_BUCKET_HOST(j)+(sv_nints-1)] != EMPTYVECT32) {
				count++;
			}
		}
	}
	fprintf (stdout, "nr. of states in hash table: %d\n", count);
}

//test function to count the contents of the host queue
void count_local_queue(inttype *q, inttype q_size, inttype *firstbit_statevector, inttype nr_procs, inttype sv_nints) {
	int count = 0, newcount = 0;
	inttype nw;
	inttype nrbuckets = q_size / WARPSIZE;
	inttype nrels = NREL_IN_BUCKET_HOST;
	for (inttype i = 0; i < nrbuckets; i++) {
		for (inttype j = 0; j < nrels; j++) {
			inttype elpos = STARTPOS_OF_EL_IN_BUCKET_HOST(j);
			inttype abselpos = (i*WARPSIZE)+elpos+sv_nints-1;
			inttype q_abselpos = q[abselpos];
			if (q_abselpos != EMPTYVECT32) {
				count++;
				nw = ISNEWSTATE_HOST(&q[(i*WARPSIZE)+elpos]);
				if (nw) {
					newcount++;
				}
			}
		}
	}
	fprintf (stdout, "nr. of states in hash table: %d (%d unexplored states)\n", count, newcount);
}

/**
 * CUDA kernel function to initialise the queue
 */
__global__ void init_queue(inttype *d_q, inttype n_elem) {
    inttype nthreads = blockDim.x*gridDim.x;
    inttype i = (blockIdx.x *blockDim.x) + threadIdx.x;

    for(; i < n_elem; i += nthreads) {
    	d_q[i] = (inttype) EMPTYVECT32;
    }
}

/**
 * CUDA kernel to store initial state in hash table
 */
__global__ void store_initial(inttype *d_q, inttype *d_h, inttype *d_newstate_flags, inttype blockdim, inttype griddim) {
	inttype bj;
	indextype hashtmp;
	inttype state[MAX_SIZE];

	for (bj = 0; bj < d_sv_nints; bj++) {
		state[bj] = 0;
	}
	SETNEWSTATE(state);
	FIRSTHASH(hashtmp, state);
	for (bj = 0; bj < d_sv_nints; bj++) {
		d_q[hashtmp+bj] = state[bj];
	}
	d_newstate_flags[(hashtmp / blockdim) % griddim] = 1;
}

/**
 * CUDA kernel function for BFS iteration state gathering
 * Order of data in the shared queue:
 * (0. index of process LTS states sizes)
 * (1. index of sync rules offsets)
 * (2. index of sync rules)
 * (1. index of open queue tile)
 * 0. the 'iterations' flag to count the number of iterations so far (nr of tiles processed by SM)
 * 1. the 'continue' flag for thread work
 * (4. index of threads buffer)
 * (5. index of hash table)
 * 2. constants for d_q hash functions (2 per function, in total 8 by default)
 * 3. state vector offsets (nr_procs+1 elements)
 * 4. sizes of states in process LTS states (nr_procs elements)
 * (9. sync rules + offsets (nr_syncbits_offsets + nr_syncbits elements))
 * 5. tile of open queue to be processed by block (sv_nints*(blockDim.x / nr_procs) elements)
 * 6. buffer for threads ((blockDim.x*max_buf_ints)+(blockDim.x/nr_procs) elements)
 * 7. hash table
 */
__global__ void
__launch_bounds__(512, 2)
gather(inttype *d_q, inttype *d_h, inttype *d_bits_state,
						inttype *d_firstbit_statevector, inttype *d_proc_offsets_start,
						inttype *d_proc_offsets, inttype *d_proc_trans, inttype *d_syncbits_offsets,
						inttype *d_syncbits, inttype *d_contBFS, inttype *d_property_violation,
						volatile inttype *d_newstate_flags, inttype *d_worktiles, inttype scan) {
	//inttype global_id = (blockIdx.x * blockDim.x) + threadIdx.x;
	//inttype group_nr = threadIdx.x / nr_procs;
	inttype i, k, l, index, offset1, offset2, tmp, cont, act, sync_offset1, sync_offset2;
	inttype* src_state = &shared[OPENTILEOFFSET+d_sv_nints*GROUP_GID];
	inttype* tgt_state = &shared[TGTSTATEOFFSET+threadIdx.x*d_sv_nints];
	inttype bitmask, bi, bj, bk;
	int pos;
	// TODO: remove this
	inttype TMPVAR;
	// is at least one outgoing transition enabled for a given state (needed to detect deadlocks)
	inttype outtrans_enabled;

	for (i = threadIdx.x; i < d_shared_q_size; i += blockDim.x) {
		shared[i] = 0;
	}
	// Locally store the state sizes and syncbits
	i = threadIdx.x;
	if (i == 0) {
		ITERATIONS = 0;
		OPENTILECOUNT = 0;
		WORKSCANRESULT = 0;
		SCAN = 0;
	}
	if ((blockIdx.x*blockDim.x)+threadIdx.x == 0) {
		(*d_contBFS) = 0;
	}
	for (i = threadIdx.x; i < HASHCONSTANTSLEN; i += blockDim.x) {
		shared[i+HASHCONSTANTSOFFSET] = d_h[i];
	}
	for (i = threadIdx.x; i < VECTORPOSLEN; i += blockDim.x) {
		shared[i+VECTORPOSOFFSET] = d_firstbit_statevector[i];
	}
	for (i = threadIdx.x; i < LTSSTATESIZELEN; i += blockDim.x) {
		shared[i+LTSSTATESIZEOFFSET] = d_bits_state[i];
	}
	// Reset the open queue tile
	if (threadIdx.x < OPENTILELEN+LASTSEARCHLEN) {
		shared[OPENTILEOFFSET+threadIdx.x] = EMPTYVECT32;
	}
	// Clean the cache
	i = threadIdx.x;
	while (i < (d_shared_q_size - CACHEOFFSET)) {
		shared[CACHEOFFSET + i] = EMPTYVECT32;
		i += blockDim.x;
	}
	__syncthreads();
	if(scan) {
		//Copy the work tile from global mem
		if (threadIdx.x < OPENTILELEN + LASTSEARCHLEN) {
			shared[OPENTILEOFFSET+threadIdx.x] = d_worktiles[(OPENTILELEN+LASTSEARCHLEN+1) * blockIdx.x + threadIdx.x];
		}
		if(threadIdx.x == 0) {
			OPENTILECOUNT = d_worktiles[(OPENTILELEN+LASTSEARCHLEN+1) * blockIdx.x + OPENTILELEN + LASTSEARCHLEN];
		}
	}
	__syncthreads();
	inttype last_search_location = shared[LASTSEARCHOFFSET + WARP_ID];
	while (ITERATIONS < d_kernel_iters) {
		if (threadIdx.x == 0 && OPENTILECOUNT < OPENTILELEN && d_newstate_flags[blockIdx.x]) {
			// Indicate that we are scanning
			d_newstate_flags[blockIdx.x] = 2;
			SCAN = 1;
		}
		__syncthreads();
		// Scan the open set for work; we use the OPENTILECOUNT flag at this stage to count retrieved elements
		if (SCAN) {
			// This block should be able to find a new state
			int found_new_state = 0;
			for (i = GLOBAL_WARP_ID; i < d_nrbuckets && OPENTILECOUNT < OPENTILELEN; i += NR_WARPS) {
				int loc = i + last_search_location;
				if(loc >= d_nrbuckets) {
					last_search_location = -i + GLOBAL_WARP_ID;
					loc = i + last_search_location;
				}
				tmp = d_q[loc*WARPSIZE+LANE];
				l = EMPTYVECT32;
				if (ENTRY_ID == (d_sv_nints-1)) {
					if (ISNEWINT(tmp)) {
						found_new_state = 1;
						// try to increment the OPENTILECOUNT counter, if successful, store the state
						l = atomicAdd((uint32_t *) &OPENTILECOUNT, d_sv_nints);
						if (l < OPENTILELEN) {
							d_q[loc*WARPSIZE+LANE] = OLDINT(tmp);
						}
					}
				}
				// all threads read the OPENTILECOUNT value of the 'tail' thread, and possibly store their part of the vector in the shared memory
				if (LANEPOINTSTOVALIDBUCKETPOS) {
					l = __shfl(l, LANE-ENTRY_ID+d_sv_nints-1);
					if (l < OPENTILELEN) {
						// write part of vector to shared memory
						shared[OPENTILEOFFSET+l+ENTRY_ID] = tmp;
					}
				}
			}
			if(i < d_nrbuckets) {
				last_search_location = i - GLOBAL_WARP_ID;
			} else {
				last_search_location = 0;
			}
			if(LANE == 0) {
				shared[LASTSEARCHOFFSET + WARP_ID] = last_search_location;
			}
			if(found_new_state || i < d_nrbuckets) {
				WORKSCANRESULT = 1;
			}
		}
		__syncthreads();
		// if work has been retrieved, indicate this
		if (threadIdx.x == 0) {
			if (OPENTILECOUNT > 0) {
				(*d_contBFS) = 1;
			}
			if(SCAN && WORKSCANRESULT == 0 && d_newstate_flags[blockIdx.x] == 2) {
				// Scanning has completed and no new states were found by this block,
				// save this information to prevent unnecessary scanning later on
				d_newstate_flags[blockIdx.x] = 0;
			} else {
				WORKSCANRESULT = 0;
			}
			scan = 0;
			CONTINUE = 1;
		}
		// is the thread part of an 'active' group?
		offset1 = 0;
		offset2 = 0;
		if (THREADINGROUP) {
			act = 1 << d_bits_act;
			for (i = 0; i < d_max_buf_ints; i++) {
				THREADBUFFERGROUPPOS(GROUP_ID, i) = 0;
			}
			// Is there work?
			if (ISSTATE(src_state)) {
				// Gather the required transition information for all states in the tile
				i = tex1Dfetch(tex_proc_offsets_start, GROUP_ID);
				// Determine process state
				GETSTATEVECTORSTATE(cont, src_state, GROUP_ID);
				// Offset position
				index = cont/(INTSIZE/d_nbits_offset);
				pos = cont - (index*(INTSIZE/d_nbits_offset));
				tmp = tex1Dfetch(tex_proc_offsets, i+index);
				GETTRANSOFFSET(offset1, tmp, pos);
				if (pos == (INTSIZE/d_nbits_offset)-1) {
					tmp = tex1Dfetch(tex_proc_offsets, i+index+1);
					GETTRANSOFFSET(offset2, tmp, 0);
				}
				else {
					GETTRANSOFFSET(offset2, tmp, pos+1);
				}
			}
			if (GROUP_ID == 0) {
				// for later, when constructing successors for this state, set action counter to maximum
				THREADGROUPCOUNTER = (1 << d_bits_act);
				THREADGROUPPOR = 0;
			}
		}
		// iterate over the outgoing transitions of state 'cont'
		// variable cont is reused to indicate whether the buffer content of this thread still needs processing
		cont = 0;
		__syncthreads();
		// while there is work to be done
		outtrans_enabled = 0;
		char generate = 1;
		char proviso_satisfied = 0;
		int cluster_trans = 1 << GROUP_ID;
		int orig_offset1 = offset1;
		while(generate > -1) {
			while (CONTINUE == 1) {
				if (offset1 < offset2 && !cont) {
					// reset act
					act = (1 << (d_bits_act));
					// reset buffer of this thread
					for (l = 0; l < d_max_buf_ints; l++) {
						THREADBUFFERGROUPPOS(GROUP_ID, l) = 0;
					}
				}
				// if not sync, store in hash table
				// loop over all transentries
				while (1) {
					i = 1;
					if(offset1 < offset2  && !cont) {
						tmp = tex1Dfetch(tex_proc_trans, offset1);
						GETPROCTRANSSYNC(i, tmp);
					}
					if (__any(i == 0)) {
						if(i == 0) {
							// no deadlock
							outtrans_enabled = 1;
							// construct state
							for (l = 0; l < d_sv_nints; l++) {
								tgt_state[l] = src_state[l];
							}
							offset1++;
						}
						// loop over this transentry
						for (l = 0; __any(i == 0 && l < NR_OF_STATES_IN_TRANSENTRY(GROUP_ID)); l++) {
							if(i == 0) {
								GETPROCTRANSSTATE(pos, tmp, l, GROUP_ID);
								if (pos > 0) {
									SETSTATEVECTORSTATE(tgt_state, GROUP_ID, pos-1);
									// check for violation of safety property, if required
									if (d_property == SAFETY) {
										if (GROUP_ID == d_nr_procs-1) {
											// pos contains state id + 1
											// error state is state 1
											if (pos == 2) {
												// error state found
												(*d_property_violation) = 1;
											}
										}
									}

									if (!d_check_cycle_proviso) {
										// Set proviso to 1 to indicate at least one state has been found
										proviso_satisfied = 1;
									}
									// store tgt_state in cache
									// if k == 8, cache is full, immediately store in global hash table
									if(generate == 1) {
										k = STOREINCACHE(tgt_state, d_q, &bi);
										if (k == 8) {
											// cache time-out; store directly in global hash table
											if (FINDORPUT_SINGLE(tgt_state, d_q, d_newstate_flags) == 0) {
												// ERROR! hash table too full. Set CONTINUE to 2
												CONTINUE = 2;
											}
										} else if(k >> 2) {
											proviso_satisfied |= (k >> 1) & 1;
										} else if (!d_check_cycle_proviso) {
											SETPORSTATE(&shared[CACHEOFFSET + bi]);
										}
									} else {
										MARKINCACHE(tgt_state, d_q, (THREADGROUPPOR >> GROUP_ID) & 1);
									}
								} else {
									i = 1;
								}
							}
							int c;
							while(generate && d_apply_por && d_check_cycle_proviso && (c = __ballot(i == 0 && (k >> 2 == 0)))) {
								int active_lane = __ffs(c) - 1;
								int cache_index = __shfl(bi, active_lane);
								bj = FIND_WARP((inttype*) &shared[CACHEOFFSET + cache_index], d_q);
								if(LANE == active_lane) {
									i = 1;
									if(bj == 0) {
										proviso_satisfied = 1;
									}
								}
							}
						}
					} else {
						break;
					}
				}

				// i is the current relative position in the buffer for this thread
				i = 0;
				if (offset1 < offset2 && !cont) {
					GETPROCTRANSACT(act, tmp);
					// store transition entry
					THREADBUFFERGROUPPOS(GROUP_ID,i) = tmp;
					atomicMin((unsigned int*)&THREADGROUPCOUNTER, act);
					cont = 1;
					i++;
					offset1++;
					while (offset1 < offset2) {
						tmp = tex1Dfetch(tex_proc_trans, offset1);
						GETPROCTRANSACT(bitmask, tmp);
						if (act == bitmask) {
							THREADBUFFERGROUPPOS(GROUP_ID,i) = tmp;
							i++;
							offset1++;
						}
						else {
							break;
						}
					}
				} else if (cont) {
					atomicMin((unsigned int*)&THREADGROUPCOUNTER, act);
				}
				__syncthreads();
				// Now, we have obtained the info needed to combine process transitions
				sync_offset1 = sync_offset2 = 0;
				if(THREADINGROUP && THREADGROUPCOUNTER < (1 << d_bits_act)) {
					// syncbits Offset position
					i = THREADGROUPCOUNTER/(INTSIZE/d_nbits_syncbits_offset);
					pos = THREADGROUPCOUNTER - (i*(INTSIZE/d_nbits_syncbits_offset));
					l = tex1Dfetch(tex_syncbits_offsets, i);
					GETSYNCOFFSET(sync_offset1, l, pos);
					if (pos == (INTSIZE/d_nbits_syncbits_offset)-1) {
						l = tex1Dfetch(tex_syncbits_offsets, i+1);
						GETSYNCOFFSET(sync_offset2, l, 0);
					}
					else {
						GETSYNCOFFSET(sync_offset2, l, pos+1);
					}
				}
				// iterate through the relevant syncbit filters
				tmp = 1;
				for (int j = GROUP_ID;__any(sync_offset1 + j / (INTSIZE/d_nr_procs) < sync_offset2 && tmp); j+=d_nr_procs) {
					index = 0;
					if(THREADINGROUP && THREADGROUPCOUNTER < (1 << d_bits_act) && sync_offset1 + j / (INTSIZE/d_nr_procs) < sync_offset2 && tmp) {
						index = tex1Dfetch(tex_syncbits, sync_offset1 + j / (INTSIZE/d_nr_procs));
					}
					GETSYNCRULE(tmp, index, j % (INTSIZE/d_nr_procs));
					l = 0;
					if(tmp) {
						SYNCRULEISAPPLICABLE(l, tmp, THREADGROUPCOUNTER);
					}
					SETOLDSTATE(tgt_state);
					int has_second_succ = 0;
					if (l) {
						// source state is not a deadlock
						outtrans_enabled = 1;
						// start combining entries in the buffer to create target states
						// if sync rule applicable, construct the first successor
						// copy src_state into tgt_state
						for (pos = 0; pos < d_sv_nints; pos++) {
							tgt_state[pos] = src_state[pos];
						}
						// construct first successor
						for (int rule = tmp; rule;) {
							pos = __ffs(rule) - 1;
							// get first state
							GETPROCTRANSSTATE(k, THREADBUFFERGROUPPOS(pos,0), 0, pos);
							SETSTATEVECTORSTATE(tgt_state, pos, k-1);
							GETPROCTRANSSTATE(k, THREADBUFFERGROUPPOS(pos,0), 1, pos);
							has_second_succ |= k;
							if(d_max_buf_ints > 1 && !k) {
								GETPROCTRANSSTATE(k, THREADBUFFERGROUPPOS(pos,1), 0, pos);
								has_second_succ |= k;
							}
							rule &= ~(1 << pos);
						}
						SETNEWSTATE(tgt_state);
					}
					int rule_proviso = 0;
					// while we keep getting new states, store them
					while (__any(ISNEWSTATE(tgt_state))) {
						l = k = TMPVAR = bitmask = 0;
						if(ISNEWSTATE(tgt_state)) {
							// check for violation of safety property, if required
							if (d_property == SAFETY) {
								GETSTATEVECTORSTATE(pos, tgt_state, d_nr_procs-1);
								if (pos == 1) {
									// error state found
									(*d_property_violation) = 1;
								}
							}

							if (!d_check_cycle_proviso) {
								// Set rule_proviso to 1 to indicate at least one state has been found
								rule_proviso = 1;
							}
							// store tgt_state in cache; if i == d_shared_q_size, state was found, duplicate detected
							// if i == d_shared_q_size+1, cache is full, immediately store in global hash table
							if(generate == 1) {
								TMPVAR = STOREINCACHE(tgt_state, d_q, &bitmask);
								if (TMPVAR == 8) {
									// cache time-out; store directly in global hash table
									if (FINDORPUT_SINGLE(tgt_state, d_q, d_newstate_flags) == 0) {
										// ERROR! hash table too full. Set CONTINUE to 2
										CONTINUE = 2;
									}
								} else if(TMPVAR >> 2) {
									rule_proviso |= (TMPVAR >> 1) & 1;
								} else if (!d_check_cycle_proviso) {
									SETPORSTATE(&shared[CACHEOFFSET + bitmask]);
								}
							} else {
								MARKINCACHE(tgt_state, d_q, (THREADGROUPPOR & tmp) == tmp);
							}
							l = 1;
							k = has_second_succ;
							if(!has_second_succ) {
								SETOLDSTATE(tgt_state);
							}
						}
						int c;
						while(generate && d_apply_por && d_check_cycle_proviso && (c = __ballot(l && TMPVAR >> 2 == 0))) {
							int active_lane = __ffs(c) - 1;
							int cache_index = __shfl(bitmask, active_lane);
							bj = FIND_WARP((inttype*) &shared[CACHEOFFSET + cache_index], d_q);
							if(LANE == active_lane) {
								l = 0;
								if(bj == 0) {
									rule_proviso = 1;
								}
							}
						}
						if(k) {
							// get next successor
							int rule;
							for (rule = tmp; rule;) {
								pos = __ffs(rule) - 1;
								int curr_st;
								GETSTATEVECTORSTATE(curr_st, tgt_state, pos);
								int st = 0;
								for (k = 0; k < d_max_buf_ints; k++) {
									for (l = 0; l < NR_OF_STATES_IN_TRANSENTRY(pos); l++) {
										GETPROCTRANSSTATE(st, THREADBUFFERGROUPPOS(pos,k), l, pos);
										if (curr_st == (st-1)) {
											break;
										}
									}
									if (curr_st == (st-1)) {
										break;
									}
								}
								// Assumption: element has been found (otherwise, 'last' was not a valid successor)
								// Try to get the next element
								if (l == NR_OF_STATES_IN_TRANSENTRY(pos) - 1) {
									if (k >= d_max_buf_ints-1) {
										st = 0;
									}
									else {
										k++;
										l = 0;
									}
								}
								else {
									l++;
								}
								// Retrieve next element, insert it in 'tgt_state' if it is not 0, and return result, otherwise continue
								if (st != 0) {
									GETPROCTRANSSTATE(st, THREADBUFFERGROUPPOS(pos,k), l, pos);
									if (st > 0) {
										SETSTATEVECTORSTATE(tgt_state, pos, st-1);
										SETNEWSTATE(tgt_state);
										break;
									}
								}
								// else, set this process state to first one, and continue to next process
								GETPROCTRANSSTATE(st, THREADBUFFERGROUPPOS(pos,0), 0, pos);
								SETSTATEVECTORSTATE(tgt_state, pos, st-1);
								rule &= ~(1 << pos);
							}
							// did we find a successor? if not, set tgt_state to old
							if (rule == 0) {
								SETOLDSTATE(tgt_state);
							}
						}
					}
					for (l = 0; l < d_nr_procs; l++) {
						// Exchange the sync rules so every thread can update its cluster_trans
						int sync_rule = __shfl(tmp, GTL((GROUP_ID + l) % d_nr_procs));
						int proviso = __shfl(rule_proviso, GTL((GROUP_ID + l) % d_nr_procs));
						if(GETBIT(GROUP_ID, sync_rule) && THREADGROUPCOUNTER == act) {
							cluster_trans |= sync_rule;
							proviso_satisfied |= proviso;
						}
					}
				}

				// only active threads should reset 'cont'
				if (cont && THREADGROUPCOUNTER == act) {
					cont = 0;
				}
				// finished an iteration of adding states.
				// Is there still work? (is another iteration required?)
				if (threadIdx.x == 0) {
					if (CONTINUE != 2) {
						CONTINUE = 0;
					}
				}
				__syncthreads();
				if (THREADINGROUP) {
					if ((offset1 < offset2) || cont) {
						if (CONTINUE != 2) {
							CONTINUE = 1;
						}
					}
				}
				if (THREADINGROUP && GROUP_ID == 0) {
					THREADGROUPCOUNTER = 1 << d_bits_act;
				}
				// FOR TEST PURPOSES!
	//			if (threadIdx.x == 0) {
	//				CONTINUE++;
	//			}
				__syncthreads();
			} // END WHILE CONTINUE == 1

			if(generate == 1 && THREADINGROUP) {
				// Choose a cluster for reduction
				if(!proviso_satisfied) {
					cluster_trans = cluster_trans & ~(1 << GROUP_ID);
				}
				THREADBUFFERGROUPPOS(GROUP_ID,0) = cluster_trans;
				__syncthreads();
				proviso_satisfied = 0;
				int to_check = cluster_trans;
				while (to_check) {
					i = __ffs(to_check) - 1;
					to_check &= ~(1 << i);
					int cluster = THREADBUFFERGROUPPOS(i, 0);
					proviso_satisfied |= GETBIT(i, cluster);
					to_check |= cluster & ~cluster_trans & ~(1 << i);
					cluster_trans |= cluster;
				}
				__syncthreads();
				if(!proviso_satisfied) {
					THREADBUFFERGROUPPOS(GROUP_ID,0) = 0;
				} else {
					THREADBUFFERGROUPPOS(GROUP_ID,0) = cluster_trans;
				}
				__syncthreads();
				if(GROUP_ID == 0) {
					int min = d_nr_procs;
					int cluster = 0xFFFFFFFF >> (INTSIZE - d_nr_procs);
					for(i = 0; i < d_nr_procs; i++) {
						if(THREADBUFFERGROUPPOS(i,0) > 0 && __popc(THREADBUFFERGROUPPOS(i,0)) < min) {
							min = __popc(THREADBUFFERGROUPPOS(i,0));
							cluster = THREADBUFFERGROUPPOS(i,0);
						}
					}
					THREADGROUPPOR = cluster;
					if(cluster < (0xFFFFFFFF >> (INTSIZE - d_nr_procs))) {
//						printf("Selected cluster %d for POR\n",cluster);
					}
				}
				__syncthreads();
			}
			offset1 = orig_offset1;
			if(THREADINGROUP) {
				THREADGROUPCOUNTER = (1 << d_bits_act);
			}
			CONTINUE = 1;
			if (d_apply_por) {
				generate--;
			} else {
				generate = -1;
			}
		} // END while(generate > -1)

		// have we encountered a deadlock state?
		// we use the shared memory to communicate this to the group leaders
		if (d_property == DEADLOCK) {
			if (THREADINGROUP) {
				if (ISSTATE(src_state)) {
					THREADBUFFERGROUPPOS(GROUP_ID, 0) = outtrans_enabled;
					// group leader collects results
					l = 0;
					if (GROUP_ID == 0) {
						for (i = 0; i < d_nr_procs; i++) {
							l += THREADBUFFERGROUPPOS(i, 0);
						}
						if (l == 0) {
							// deadlock state found
							(*d_property_violation) = 1;
						}
					}
				}
			}
		}
		int performed_work = OPENTILECOUNT != 0;
		__syncthreads();
		// Reset the open queue tile
		if (threadIdx.x < OPENTILELEN) {
			shared[OPENTILEOFFSET+threadIdx.x] = EMPTYVECT32;
		}
		if (threadIdx.x == 0) {
			OPENTILECOUNT = 0;
		}
		__syncthreads();
		// start scanning the local cache and write results to the global hash table
		k = (d_shared_q_size-CACHEOFFSET)/d_sv_nints;
		int c;
		for (i = WARP_ID; performed_work && i * WARPSIZE < k; i += (blockDim.x / WARPSIZE)) {
			int have_new_state = i * WARPSIZE + LANE < k && ISNEWSTATE(&shared[CACHEOFFSET+(i*WARPSIZE+LANE)*d_sv_nints]);
			while (c = __ballot(have_new_state)) {
				int active_lane = __ffs(c) - 1;
				if(FINDORPUT_WARP((inttype*) &shared[CACHEOFFSET + (i*WARPSIZE+active_lane)*d_sv_nints], d_q, d_newstate_flags) == 0) {
					CONTINUE = 2;
				}
				if (LANE == active_lane) {
					have_new_state = 0;
				}
			}
		}
		__syncthreads();
		// Ready to start next iteration, if error has not occurred
		if (threadIdx.x == 0) {
			if (CONTINUE == 2) {
				(*d_contBFS) = 2;
				ITERATIONS = d_kernel_iters;
			}
			else {
				ITERATIONS++;
			}
			CONTINUE = 0;
		}
		__syncthreads();
	}

	//Copy the work tile to global mem
	if (threadIdx.x < OPENTILELEN+LASTSEARCHLEN) {
		d_worktiles[(OPENTILELEN+LASTSEARCHLEN+1) * blockIdx.x + threadIdx.x] = shared[OPENTILEOFFSET+threadIdx.x];
	}
	if(threadIdx.x == 0) {
		d_worktiles[(OPENTILELEN+LASTSEARCHLEN+1) * blockIdx.x + OPENTILELEN+LASTSEARCHLEN] = OPENTILECOUNT;
	}
}

/**
 * Host function that prepares data array and passes it to the CUDA kernel.
 */
int main(int argc, char** argv) {
	FILE *fp;
	inttype nr_procs, bits_act, bits_statevector, sv_nints, nr_trans, proc_nrstates, nbits_offset, max_buf_ints, nr_syncbits_offsets, nr_syncbits, nbits_syncbits_offset;
	inttype *bits_state, *firstbit_statevector, *proc_offsets, *proc_trans, *proc_offsets_start, *syncbits_offsets, *syncbits;
	inttype contBFS;
	char stmp[BUFFERSIZE], fn[50];
	// to store constants for closed set hash functions
	int h[NR_HASH_FUNCTIONS*2];
	// size of global hash table
	size_t q_size = 0;
	PropertyStatus check_property = NONE;
	// nr of iterations in single kernel run
	int kernel_iters = KERNEL_ITERS;
	int nblocks = NR_OF_BLOCKS;
	int nthreadsperblock = BLOCK_SIZE;
	// POR options
	int apply_por = 0;
	int use_cycle_proviso = 0;
	// level of verbosity (1=print level progress)
	int verbosity = 0;
	// clock to measure time
	clock_t start, stop;
	double runtime = 0.0;

	// Start timer
	assert((start = clock())!=-1);

	hipDeviceProp_t prop;
	int nDevices;

	// GPU side versions of the input
	inttype *d_bits_state, *d_firstbit_statevector, *d_proc_offsets_start, *d_proc_offsets, *d_proc_trans, *d_syncbits_offsets, *d_syncbits, *d_h;
	// flag to keep track of progress and whether hash table errors occurred (value==2)
	inttype *d_contBFS;
	// flags to track which blocks have new states
	inttype *d_newstate_flags;
	// flag to keep track of property verification outcome
	inttype *d_property_violation;
	// Space to temporarily store work tiles
	inttype *d_worktiles;

	// GPU datastructures for calculation
	inttype *d_q;

	if (argc == 1) {
		fprintf(stderr, "ERROR: No input network given!\n");
		exit(1);
	}

	strcpy(fn, argv[1]);
	strcat(fn, ".gpf");

	int i = 2;
	while (i < argc) {
		printf ("%s\n", argv[i]);
		if (!strcmp(argv[i],"-k")) {
			// if nr. of iterations per kernel run is given, store it
			kernel_iters = atoi(argv[i+1]);
			i += 2;
		}
		else if (!strcmp(argv[i],"-b")) {
			// store nr of blocks to be used
			nblocks = atoi(argv[i+1]);
			i += 2;
		}
		else if (!strcmp(argv[i],"-t")) {
			// store nr of threads per block to be used
			nthreadsperblock = atoi(argv[i+1]);
			i += 2;
		}
		else if (!strcmp(argv[i],"-q")) {
			// store hash table size
			q_size = atoi(argv[i+1]);
			i += 2;
		}
		else if (!strcmp(argv[i],"-v")) {
			// store verbosity level
			verbosity = atoi(argv[i+1]);
			if (verbosity > 3) {
				verbosity = 3;
			}
			i += 2;
		}
		else if (!strcmp(argv[i],"-d")) {
			// check for deadlocks
			check_property = DEADLOCK;
			use_cycle_proviso = 0;
			i += 1;
		}
		else if (!strcmp(argv[i],"-p")) {
			// check a property
			check_property = SAFETY;
			use_cycle_proviso = 1;
			i += 1;
		}
		else if (!strcmp(argv[i],"--por")) {
			// apply partial-order reduction
			apply_por = 1;
			i += 1;
		}
		else if (!strcmp(argv[i],"--cycle-proviso")) {
			// use cycle proviso
			if (check_property == NONE) {
				use_cycle_proviso = 1;
			}
			i += 1;
		}
	}

	fp = fopen(fn, "r");
	if (fp) {
		// Read the input
		fgets(stmp, BUFFERSIZE, fp);
		if (check_property == SAFETY) {
			i = atoi(stmp);
			fprintf(stdout, "Property to check is ");
			if (i == 0) {
				fprintf(stdout, "not ");
			}
			fprintf(stdout, "a liveness property\n");
			if (i == 1) {
				check_property = LIVENESS;
			}
		}
		fgets(stmp, BUFFERSIZE, fp);
		nr_procs = atoi(stmp);
		fprintf(stdout, "nr of procs: %d\n", nr_procs);
		fgets(stmp, BUFFERSIZE, fp);
		bits_act = atoi(stmp);
		fprintf(stdout, "nr of bits for transition label: %d\n", bits_act);
		fgets(stmp, BUFFERSIZE, fp);
		proc_nrstates = atoi(stmp);
		fprintf(stdout, "min. nr. of proc. states that fit in 32-bit integer: %d\n", proc_nrstates);
		fgets(stmp, BUFFERSIZE, fp);
		bits_statevector = atoi(stmp) + 1;
		fprintf(stdout, "number of bits needed for a state vector: %d\n", bits_statevector);
		firstbit_statevector = (inttype*) malloc(sizeof(inttype)*(nr_procs+1));
		for (int i = 0; i <= nr_procs; i++) {
			fgets(stmp, BUFFERSIZE, fp);
			firstbit_statevector[i] = atoi(stmp);
			fprintf(stdout, "statevector offset %d: %d\n", i, firstbit_statevector[i]);
		}
		// determine the number of integers needed for a state vector
		sv_nints = (bits_statevector+31) / INTSIZE;
		bits_state = (inttype*) malloc(sizeof(inttype)*nr_procs);
		for (int i = 0; i < nr_procs; i++) {
			fgets(stmp, BUFFERSIZE, fp);
			bits_state[i] = atoi(stmp);
			fprintf(stdout, "bits for states of process LTS %d: %d\n", i, bits_state[i]);
		}
		fgets(stmp, BUFFERSIZE, fp);
		nbits_offset = atoi(stmp);
		fprintf(stdout, "size of offset in process LTSs: %d\n", nbits_offset);
		fgets(stmp, BUFFERSIZE, fp);
		max_buf_ints = atoi(stmp);
		fprintf(stdout, "maximum label-bounded branching factor: %d\n", max_buf_ints);
		proc_offsets_start = (inttype*) malloc(sizeof(inttype)*(nr_procs+1));
		for (int i = 0; i <= nr_procs; i++) {
			fgets(stmp, BUFFERSIZE, fp);
			proc_offsets_start[i] = atoi(stmp);
		}
		proc_offsets = (inttype*) malloc(sizeof(inttype)*proc_offsets_start[nr_procs]);
		for (int i = 0; i < proc_offsets_start[nr_procs]; i++) {
			fgets(stmp, BUFFERSIZE, fp);
			proc_offsets[i] = atoi(stmp);
		}
		fgets(stmp, BUFFERSIZE, fp);
		nr_trans = atoi(stmp);
		fprintf(stdout, "total number of transition entries in network: %d\n", nr_trans);
		proc_trans = (inttype*) malloc(sizeof(inttype)*nr_trans);
		for (int i = 0; i < nr_trans; i++) {
			fgets(stmp, BUFFERSIZE, fp);
			proc_trans[i] = atoi(stmp);
		}

		fgets(stmp, BUFFERSIZE, fp);
		nbits_syncbits_offset = atoi(stmp);
		//fprintf(stdout, "size of offset in sync rules: %d\n", nbits_syncbits_offset);
		fgets(stmp, BUFFERSIZE, fp);
		nr_syncbits_offsets = atoi(stmp);
		syncbits_offsets = (inttype*) malloc(sizeof(inttype)*nr_syncbits_offsets);
		for (int i = 0; i < nr_syncbits_offsets; i++) {
			fgets(stmp, BUFFERSIZE, fp);
			syncbits_offsets[i] = atoi(stmp);
			//fprintf(stdout, "syncbits offset %d: %d\n", i, syncbits_offsets[i]);
		}
		fgets(stmp, BUFFERSIZE, fp);
		nr_syncbits = atoi(stmp);
		syncbits = (inttype*) malloc(sizeof(inttype)*nr_syncbits);
		for (int i = 0; i < nr_syncbits; i++) {
			fgets(stmp, BUFFERSIZE, fp);
			syncbits[i] = atoi(stmp);
			//fprintf(stdout, "syncbits %d: %d\n", i, syncbits[i]);
		}
	}
	else {
		fprintf(stderr, "ERROR: input network does not exist!\n");
		exit(1);
	}

	// Randomly define the closed set hash functions
//	srand(time(NULL));
//	for (int i = 0; i < NR_HASH_FUNCTIONS*2; i++) {
//		h[i] = rand();
//	}
	// TODO: make random again
	h[0] = 483319424;
	h[1] = 118985421;
	h[2] = 1287157904;
	h[3] = 1162380012;
	h[4] = 1231274815;
	h[5] = 1344969351;
	h[6] = 527997957;
	h[7] = 735456672;
	h[8] = 1774251664;
	h[9] = 23102285;
	h[10] = 2089529600;
	h[11] = 2083003102;
	h[12] = 908039861;
	h[13] = 1913855526;
	h[14] = 1515282600;
	h[15] = 1691511413;

	// continue flags
	contBFS = 1;

	// Query the device properties and determine data structure sizes
	hipGetDeviceCount(&nDevices);
	if (nDevices == 0) {
		fprintf (stderr, "ERROR: No CUDA compatible GPU detected!\n");
		exit(1);
	}
	hipGetDeviceProperties(&prop, 0);
	fprintf (stdout, "global mem: %lu\n", (uint64_t) prop.totalGlobalMem);
	fprintf (stdout, "shared mem per block: %d\n", (int) prop.sharedMemPerBlock);
	fprintf (stdout, "max. threads per block: %d\n", (int) prop.maxThreadsPerBlock);
	fprintf (stdout, "max. grid size: %d\n", (int) prop.maxGridSize[0]);
	fprintf (stdout, "nr. of multiprocessors: %d\n", (int) prop.multiProcessorCount);

	// determine actual nr of blocks
	nblocks = MAX(1,MIN(prop.maxGridSize[0],nblocks));

	// Allocate memory on GPU
	cudaMallocCount((void **) &d_contBFS, sizeof(inttype));
	cudaMallocCount((void **) &d_property_violation, sizeof(inttype));
	cudaMallocCount((void **) &d_h, NR_HASH_FUNCTIONS*2*sizeof(inttype));
	cudaMallocCount((void **) &d_bits_state, nr_procs*sizeof(inttype));
	cudaMallocCount((void **) &d_firstbit_statevector, (nr_procs+1)*sizeof(inttype));
	cudaMallocCount((void **) &d_proc_offsets_start, (nr_procs+1)*sizeof(inttype));
	cudaMallocCount((void **) &d_proc_offsets, proc_offsets_start[nr_procs]*sizeof(inttype));
	cudaMallocCount((void **) &d_proc_trans, nr_trans*sizeof(inttype));
	cudaMallocCount((void **) &d_syncbits_offsets, nr_syncbits_offsets*sizeof(inttype));
	cudaMallocCount((void **) &d_syncbits, nr_syncbits*sizeof(inttype));
	cudaMallocCount((void **) &d_newstate_flags, nblocks*sizeof(inttype));
	cudaMallocCount((void **) &d_worktiles, nblocks * (sv_nints*(nthreadsperblock/nr_procs)+nthreadsperblock/WARPSIZE+1)*sizeof(inttype));


	// Copy data to GPU
	CUDA_CHECK_RETURN(hipMemcpy(d_contBFS, &contBFS, sizeof(inttype), hipMemcpyHostToDevice))
	CUDA_CHECK_RETURN(hipMemcpy(d_h, h, NR_HASH_FUNCTIONS*2*sizeof(inttype), hipMemcpyHostToDevice))
	CUDA_CHECK_RETURN(hipMemcpy(d_bits_state, bits_state, nr_procs*sizeof(inttype), hipMemcpyHostToDevice))
	CUDA_CHECK_RETURN(hipMemcpy(d_firstbit_statevector, firstbit_statevector, (nr_procs+1)*sizeof(inttype), hipMemcpyHostToDevice))
	CUDA_CHECK_RETURN(hipMemcpy(d_proc_offsets_start, proc_offsets_start, (nr_procs+1)*sizeof(inttype), hipMemcpyHostToDevice))
	CUDA_CHECK_RETURN(hipMemcpy(d_proc_offsets, proc_offsets, proc_offsets_start[nr_procs]*sizeof(inttype), hipMemcpyHostToDevice))
	CUDA_CHECK_RETURN(hipMemcpy(d_proc_trans, proc_trans, nr_trans*sizeof(inttype), hipMemcpyHostToDevice))
	CUDA_CHECK_RETURN(hipMemcpy(d_syncbits_offsets, syncbits_offsets, nr_syncbits_offsets*sizeof(inttype), hipMemcpyHostToDevice))
	CUDA_CHECK_RETURN(hipMemcpy(d_syncbits, syncbits, nr_syncbits*sizeof(inttype), hipMemcpyHostToDevice))
	CUDA_CHECK_RETURN(hipMemset(d_newstate_flags, 0, nblocks*sizeof(inttype)));
	CUDA_CHECK_RETURN(hipMemset(d_worktiles, 0, nblocks * (sv_nints*(nthreadsperblock/nr_procs)+nthreadsperblock/WARPSIZE+1)*sizeof(inttype)));

	// Bind data to textures
	hipBindTexture(NULL, tex_proc_offsets_start, d_proc_offsets_start, (nr_procs+1)*sizeof(inttype));
	hipBindTexture(NULL, tex_proc_offsets, d_proc_offsets, proc_offsets_start[nr_procs]*sizeof(inttype));
	hipBindTexture(NULL, tex_proc_trans, d_proc_trans, nr_trans*sizeof(inttype));
	hipBindTexture(NULL, tex_syncbits_offsets, d_syncbits_offsets, nr_syncbits_offsets*sizeof(inttype));
	hipBindTexture(NULL, tex_syncbits, d_syncbits, nr_syncbits*sizeof(inttype));

	size_t available, total;
	hipMemGetInfo(&available, &total);
	if (q_size == 0) {
		q_size = total / sizeof(inttype);
	}
	size_t el_per_Mb = Mb / sizeof(inttype);


	while(hipMalloc((void**)&d_q,  q_size * sizeof(inttype)) == hipErrorOutOfMemory)	{
		q_size -= el_per_Mb;
		if( q_size  < el_per_Mb) {
			// signal no free memory
			break;
		}
	}

	fprintf (stdout, "global mem queue size: %lu, number of entries: %lu\n", q_size*sizeof(inttype), (indextype) q_size);

	inttype shared_q_size = (int) prop.sharedMemPerBlock / sizeof(inttype);
	fprintf (stdout, "shared mem queue size: %lu, number of entries: %u\n", shared_q_size*sizeof(inttype), shared_q_size);
	fprintf (stdout, "nr. of blocks: %d, block size: %d, nr of kernel iterations: %d\n", nblocks, nthreadsperblock, kernel_iters);

	// copy symbols
	inttype tablesize = q_size;
	inttype nrbuckets = tablesize / WARPSIZE;
	hipMemcpyToSymbol(HIP_SYMBOL(d_nrbuckets), &nrbuckets, sizeof(inttype));
	hipMemcpyToSymbol(HIP_SYMBOL(d_shared_q_size), &shared_q_size, sizeof(inttype));
	hipMemcpyToSymbol(HIP_SYMBOL(d_nr_procs), &nr_procs, sizeof(inttype));
	hipMemcpyToSymbol(HIP_SYMBOL(d_max_buf_ints), &max_buf_ints, sizeof(inttype));
	hipMemcpyToSymbol(HIP_SYMBOL(d_sv_nints), &sv_nints, sizeof(inttype));
	hipMemcpyToSymbol(HIP_SYMBOL(d_bits_act), &bits_act, sizeof(inttype));
	hipMemcpyToSymbol(HIP_SYMBOL(d_nbits_offset), &nbits_offset, sizeof(inttype));
	hipMemcpyToSymbol(HIP_SYMBOL(d_nbits_syncbits_offset), &nbits_syncbits_offset, sizeof(inttype));
	hipMemcpyToSymbol(HIP_SYMBOL(d_kernel_iters), &kernel_iters, sizeof(inttype));
	hipMemcpyToSymbol(HIP_SYMBOL(d_property), &check_property, sizeof(inttype));
	hipMemcpyToSymbol(HIP_SYMBOL(d_apply_por), &apply_por, sizeof(inttype));
	hipMemcpyToSymbol(HIP_SYMBOL(d_check_cycle_proviso), &use_cycle_proviso, sizeof(inttype));

	// init the queue
	init_queue<<<nblocks, nthreadsperblock>>>(d_q, q_size);
	store_initial<<<1,1>>>(d_q, d_h, d_newstate_flags,nthreadsperblock,nblocks);
	for (int i = 0; i < 2*NR_HASH_FUNCTIONS; i++) {
		fprintf (stdout, "hash constant %d: %d\n", i, h[i]);
	}
	FIRSTHASHHOST(i);
	fprintf (stdout, "hash of initial state: %d\n", i);

	inttype zero = 0;
	inttype *q_test = (inttype*) malloc(sizeof(inttype)*tablesize);
	int j = 0;
	inttype scan = 0;
	CUDA_CHECK_RETURN(hipMemcpy(d_property_violation, &zero, sizeof(inttype), hipMemcpyHostToDevice))
	inttype property_violation = 0;
	while (contBFS == 1) {
		CUDA_CHECK_RETURN(hipMemcpy(d_contBFS, &zero, sizeof(inttype), hipMemcpyHostToDevice))
		gather<<<nblocks, nthreadsperblock, shared_q_size*sizeof(inttype)>>>(d_q, d_h, d_bits_state, d_firstbit_statevector, d_proc_offsets_start,
																		d_proc_offsets, d_proc_trans, d_syncbits_offsets, d_syncbits, d_contBFS, d_property_violation, d_newstate_flags, d_worktiles, scan);
		// copy progress result
		//CUDA_CHECK_RETURN(hipGetLastError());
		CUDA_CHECK_RETURN(hipDeviceSynchronize());
		CUDA_CHECK_RETURN(hipMemcpy(&contBFS, d_contBFS, sizeof(inttype), hipMemcpyDeviceToHost))
		if (check_property > 0) {
			CUDA_CHECK_RETURN(hipMemcpy(&property_violation, d_property_violation, sizeof(inttype), hipMemcpyDeviceToHost))
			if (property_violation == 1) {
				contBFS = 0;
			}
		}
		if (verbosity > 0) {
			if (verbosity == 1) {
				printf ("%d\n", j++);
			}
			else if (verbosity == 2) {
				hipMemcpy(q_test, d_q, tablesize*sizeof(inttype), hipMemcpyDeviceToHost);
				count_local_queue(q_test, tablesize, firstbit_statevector, nr_procs, sv_nints);
			}
			else if (verbosity == 3) {
				hipMemcpy(q_test, d_q, tablesize*sizeof(inttype), hipMemcpyDeviceToHost);
				print_local_queue(stdout, q_test, tablesize, firstbit_statevector, nr_procs, sv_nints);
			}
		}
		scan = 1;
	}
	// determine runtime
	stop = clock();
	runtime = (double) (stop-start)/CLOCKS_PER_SEC;
	fprintf (stdout, "Run time: %f\n", runtime);

	if (property_violation == 1) {
		switch (check_property) {
			case DEADLOCK:
				printf ("deadlock detected!\n");
				break;
			case SAFETY:
				printf ("safety property violation detected!\n");
				break;
			case LIVENESS:
				printf ("liveness property violation detected!\n");
				break;
		}
	}
	// report error if required
	if (contBFS == 2) {
		fprintf (stderr, "ERROR: problem with hash table\n");
	}
	count_queue(d_q, tablesize, firstbit_statevector, nr_procs, sv_nints);

	// Debugging functionality: print states to file
//	FILE* fout;
//	fout = fopen("/tmp/gpuexplore.debug", "w");
//	hipMemcpy(q_test, d_q, tablesize*sizeof(inttype), hipMemcpyDeviceToHost);
//	print_local_queue(fout, q_test, tablesize, firstbit_statevector, nr_procs, sv_nints);
//	fclose(fout);

	CUDA_CHECK_RETURN(hipDeviceSynchronize());	// Wait for the GPU launched work to complete
	//CUDA_CHECK_RETURN(hipGetLastError());

	return 0;
}
