#include "hip/hip_runtime.h"
/*
 ============================================================================
 Name        : GPUexplore.cu
 Author      : Anton Wijs
 Version     :
 Copyright   : Copyright Anton Wijs
 Description : CUDA compute reciprocals
 ============================================================================
 */

#include <stdio.h>
#include <stdlib.h>
#include <stdint.h>
#include <string.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <assert.h>
#include <time.h>
#include <math.h>

// type of elements used
#define inttype uint32_t

__forceinline__ inttype MIN(inttype a, inttype b) {
	return (a < b) ? a : b;
}

__forceinline__ inttype MAX(inttype a, inttype b) {
   return (a > b) ? a : b;
}

// Nr of tiles processed in single kernel launch
//#define TILEITERS 10

static const int WARPSIZE = 32;
static const int HALFWARPSIZE = 16;
static const int INTSIZE = 32;
static const int BUFFERSIZE = 32;
// constant for cuckoo hashing (Alcantara et al)
static const inttype P = 334214459;
// Retry constant to determine number of retries for element insertion
#define RETRYFREQ 7
#define NR_HASH_FUNCTIONS 8
// Number of retries in local cache
#define CACHERETRYFREQ 20
// Maximum size of state vectors (in nr. of 32-bit integers)
#define MAX_SIZE 9
// Empty state vectors
static const inttype EMPTYVECT32 = 0x7FFFFFFF;
// Constant to indicate that no more work is required
# define EXPLORATION_DONE 0x7FFFFFFF
// offset in shared memory from which loaded data can be read
static const int SH_OFFSET = 3;
static const int KERNEL_ITERS = 10;
static const int NR_OF_BLOCKS = 3120;
static const int BLOCK_SIZE = 512;
const size_t Mb = 1<<20;

// test macros
#define PRINTTHREADID()						{printf("Hello thread %d\n", (blockIdx.x*blockDim.x)+threadIdx.x);}
#define PRINTTHREAD(j, i)					{printf("%d: Seen by thread %d: %d\n", (j), (blockIdx.x*blockDim.x)+threadIdx.x, (i));}

// Offsets calculations for shared memory arrays
#define HASHCONSTANTSLEN				(2*NR_HASH_FUNCTIONS)
#define VECTORPOSLEN					(d_nr_procs+1)
#define LTSSTATESIZELEN					(d_nr_procs)
#define OPENTILELEN						(d_sv_nints*(blockDim.x/d_nr_procs))
#define THREADBUFFERLEN					((blockDim.x/d_nr_procs)*(1+(d_nr_procs*d_max_buf_ints)))

#define HASHCONSTANTSOFFSET 			(SH_OFFSET)
#define VECTORPOSOFFSET 				(SH_OFFSET+HASHCONSTANTSLEN)
#define LTSSTATESIZEOFFSET 				(SH_OFFSET+HASHCONSTANTSLEN+VECTORPOSLEN)
#define OPENTILEOFFSET 					(SH_OFFSET+HASHCONSTANTSLEN+VECTORPOSLEN+LTSSTATESIZELEN)
#define THREADBUFFEROFFSET	 			(SH_OFFSET+HASHCONSTANTSLEN+VECTORPOSLEN+LTSSTATESIZELEN+OPENTILELEN)
#define CACHEOFFSET 					(SH_OFFSET+HASHCONSTANTSLEN+VECTORPOSLEN+LTSSTATESIZELEN+OPENTILELEN+THREADBUFFERLEN)

// parameter is thread id
#define THREADBUFFERGROUPSTART(i)		THREADBUFFEROFFSET+(((i) / d_nr_procs)*(1+(d_nr_procs*d_max_buf_ints)))
// parameter is group id
#define THREADBUFFERGROUPPOS(i, j)		shared[THREADBUFFERGROUPSTART(threadIdx.x)+1+((i)*d_max_buf_ints)+(j)]
#define THREADGROUPCOUNTER				shared[(THREADBUFFERGROUPSTART(threadIdx.x))]
#define OPENTILESTATEPART(i)			shared[OPENTILEOFFSET+(d_sv_nints*(threadIdx.x / d_nr_procs))+(i)]

#define THREADINGROUP					(threadIdx.x < (blockDim.x/d_nr_procs)*d_nr_procs)

#define STATESIZE(i)					(shared[LTSSTATESIZEOFFSET+(i)])
#define VECTORSTATEPOS(i)				(shared[VECTORPOSOFFSET+(i)])
#define NR_OF_STATES_IN_TRANSENTRY(i)	((31 - d_bits_act) / shared[LTSSTATESIZEOFFSET+(i)])
// SM local progress flags
#define ITERATIONS						(shared[0])
#define CONTINUE						(shared[1])
#define OPENTILECOUNT					(shared[2])

// thread ids
#define WARP_ID							(threadIdx.x / WARPSIZE)
#define GLOBAL_WARP_ID					(((blockDim.x / WARPSIZE)*blockIdx.x)+WARP_ID)
#define NR_WARPS						((blockDim.x / WARPSIZE)*gridDim.x)
#define LANE							(threadIdx.x % WARPSIZE)
#define HALFLANE						(threadIdx.x % HALFWARPSIZE)
#define ENTRY_ID						(HALFLANE % d_sv_nints)
#define GROUP_ID						(threadIdx.x % d_nr_procs)

#define NREL_IN_BUCKET					((HALFWARPSIZE / d_sv_nints)*2)



// GPU constants
__constant__ inttype d_nrbuckets;
__constant__ inttype d_shared_q_size;
__constant__ inttype d_nr_procs;
__constant__ inttype d_max_buf_ints;
__constant__ inttype d_sv_nints;
__constant__ inttype d_bits_act;
__constant__ inttype d_nbits_offset;
__constant__ inttype d_kernel_iters;
__constant__ inttype d_nbits_syncbits_offset;
__constant__ inttype d_property;

// GPU shared memory structure
extern volatile __shared__ inttype shared[];


__device__ inttype LANE_POINTS_TO_EL(inttype i)	{
	if (i < HALFWARPSIZE / d_sv_nints) {
		return (LANE >= i*d_sv_nints && LANE < (i+1)*d_sv_nints);
	}
	else {
		return (LANE >= HALFWARPSIZE+(i-(HALFWARPSIZE / d_sv_nints))*d_sv_nints && LANE < HALFWARPSIZE+(i-(HALFWARPSIZE / d_sv_nints)+1)*d_sv_nints);
	}
}

// BIT MANIPULATION MACROS

#define SETBIT(i, x)							{(x) = ((((inttype) 1)<<(i)) | (x));}
#define GETBIT(i, x)							(((x) >> (i)) & ((inttype) 1))
#define SETBITS(i, j, x)						for (bi = (i); bi < (j); bi++) { (x) = (x) | (((inttype) 1)<<bi); }

#define GETPROCTRANSACT(a, t)					{bitmask = 0; SETBITS(1, 1+d_bits_act, bitmask); (a) = ((t) & bitmask) >> 1;}
#define GETPROCTRANSSYNC(a, t)					{(a) = ((t) & 1);}
#define GETPROCTRANSSTATE(a, t, i, j)			{bitmask = 0; SETBITS(1+d_bits_act+((i)-1)*shared[LTSSTATESIZEOFFSET+(j)], \
								1+d_bits_act+(i)*shared[LTSSTATESIZEOFFSET+(j)],bitmask); \
								(a) = ((t) & bitmask) >> 1+d_bits_act+(((i)-1)*shared[LTSSTATESIZEOFFSET+(j)]);}
#define GETTRANSOFFSET(a, t, i)					{bitmask = 0; SETBITS((i)*d_nbits_offset, ((i)+1)*d_nbits_offset, bitmask); (a) = ((t) & bitmask) >> ((i)*d_nbits_offset);}
#define GETSYNCOFFSET(a, t, i)					{bitmask = 0; SETBITS((i)*d_nbits_syncbits_offset, ((i)+1)*d_nbits_syncbits_offset, bitmask); \
													(a) = ((t) & bitmask) >> ((i)*d_nbits_syncbits_offset);}
#define GETSTATEVECTORSTATE(a, t, i)			{bitmask = 0; 	if (shared[VECTORPOSOFFSET+(i)]/INTSIZE == (shared[VECTORPOSOFFSET+(i)+1]-1)/INTSIZE) { \
																	SETBITS(shared[VECTORPOSOFFSET+(i)] % INTSIZE, \
																			((shared[VECTORPOSOFFSET+(i)+1]-1) % INTSIZE)+1, bitmask); \
																	(a) = ((t)[shared[VECTORPOSOFFSET+(i)]/INTSIZE] & bitmask) >> (shared[VECTORPOSOFFSET+(i)] % INTSIZE); \
																} \
																else { \
																	SETBITS(0,shared[VECTORPOSOFFSET+(i)+1] % INTSIZE,bitmask); \
																	(a) = (t)[shared[VECTORPOSOFFSET+(i)]/INTSIZE] >> (shared[VECTORPOSOFFSET+(i)] % INTSIZE) \
																		 | \
																		((t)[shared[VECTORPOSOFFSET+(i)+1]/INTSIZE] & bitmask) << \
																		(INTSIZE - (shared[VECTORPOSOFFSET+(i)] % INTSIZE)); \
																} \
												}
#define SETPROCTRANSACT(t, x)					{bitmask = 0; SETBITS(1, d_bits_act+1,bitmask); (t) = ((t) & ~bitmask) | ((x) << 1);}
#define SETPROCTRANSSYNC(t, x)					{(t) = ((t) & ~1) | (x);}
#define SETPROCTRANSSTATE(t, i, x, j)			{bitmask = 0; SETBITS(1+d_bits_act+((i)-1)*shared[LTSSTATESIZEOFFSET+(j)],1+d_bits_act+(i)*shared[LTSSTATESIZEOFFSET+(j)],bitmask); \
													(t) = ((t) & ~bitmask) | ((x) << (1+d_bits_act+((i)-1)*shared[LTSSTATESIZEOFFSET+(j)]));}
#define SETSTATEVECTORSTATE(t, i, x)			{bitmask = 0; 	if (shared[VECTORPOSOFFSET+(i)]/INTSIZE == (shared[VECTORPOSOFFSET+(i)+1]-1)/INTSIZE) { \
																	SETBITS(shared[VECTORPOSOFFSET+(i)] % INTSIZE, \
																			((shared[VECTORPOSOFFSET+(i)+1]-1) % INTSIZE)+1,bitmask); \
																	(t)[shared[VECTORPOSOFFSET+(i)]/INTSIZE] = ((t)[shared[VECTORPOSOFFSET+(i)]/INTSIZE] & ~bitmask) | \
																	((x) << (shared[VECTORPOSOFFSET+(i)] % INTSIZE)); \
																} \
																else { \
																	SETBITS(0,shared[VECTORPOSOFFSET+(i)] % INTSIZE, bitmask); \
																	(t)[shared[VECTORPOSOFFSET+(i)]/INTSIZE] = ((t)[shared[VECTORPOSOFFSET+(i)]/INTSIZE] & bitmask) | \
																	((x) << (shared[VECTORPOSOFFSET+(i)] % INTSIZE)); \
																	bitmask = 0; \
																	SETBITS(shared[VECTORPOSOFFSET+(i)+1] % INTSIZE, INTSIZE, bitmask); \
																	(t)[shared[VECTORPOSOFFSET+(i)+1]/INTSIZE] = ((t)[shared[VECTORPOSOFFSET+(i)+1]/INTSIZE] & bitmask) | \
																		((x) >> (INTSIZE - (shared[VECTORPOSOFFSET+(i)] % INTSIZE))); \
																} \
												}
// NEEDS FIX: USE BIT 32 OF FIRST INTEGER TO INDICATE STATE OR NOT (1 or 0), IN CASE MULTIPLE INTEGERS ARE USED FOR STATE VECTOR!!!
//#define ISSTATE(t)								((t)[(d_sv_nints-1)] != EMPTYVECT32)
#define ISSTATE(t)								((t)[0] != EMPTYVECT32)
#define SETNEWSTATE(t)							{	(t)[(d_sv_nints-1)] = (t)[(d_sv_nints-1)] | 0x80000000;}
#define SETOLDSTATE(t)							{	(t)[(d_sv_nints-1)] = (t)[(d_sv_nints-1)] & 0x7FFFFFFF;}
#define ISNEWSTATE(t)							((t)[(d_sv_nints-1)] >> 31)
#define ISNEWSTATE_HOST(t)						((t)[(sv_nints-1)] >> 31)
#define ISNEWINT(t)								((t) >> 31)
#define OLDINT(t)								((t) & 0x7FFFFFFF)
#define NEWINT(t)								((t) | 0x80000000)
// TODO: change!
// TODO: change to 30!
#define ISOLDCACHEINT(t)						(((t) >> 31 == 0) ? 1 : 0)
//#define SETOLDCACHESTATE(t)						{	(t)[(d_sv_nints-1)] = (t)[(d_sv_nints)-1] & 0x3FFFFFFF;}
// TODO: change to 3FFFFFFF!
#define SETOLDCACHEINT(t)						{	(t) = (t) | 0x7FFFFFFF;}

#define STRIPSTATE(t)							{(t)[(d_sv_nints-1)] = (t)[(d_sv_nints-1)] & 0x7FFFFFFF;}
#define STRIPPEDSTATE(t, i)						((i == d_sv_nints-1) ? ((t)[i] & 0x7FFFFFFF) : (t)[i])
#define NEWSTATEPART(t, i)						(((i) == d_sv_nints-1) ? ((t)[d_sv_nints-1] | 0x80000000) : (t)[(i)])
#define COMPAREENTRIES(t1, t2)					(((t1) & 0x7FFFFFFF) == ((t2) & 0x7FFFFFFF))
#define OWNSSYNCRULE(a, t, i)					{if (GETBIT((i),(t))) { \
													bitmask = 0; SETBITS(0,(i),bitmask); if ((t & bitmask) > 0) {(a) = 0;} else {(a) = 1;}} \
													else {(a) = 0;}}
#define GETSYNCRULE(a, t, i)					{bitmask = 0; SETBITS((i)*d_nr_procs,((i)+1)*d_nr_procs,bitmask); (a) = ((t) & bitmask) >> ((i)*d_nr_procs);}
#define SYNCRULEISAPPLICABLE(a, t, ac)			{(a) = 1; for (bk = 0; bk < d_nr_procs; bk++) { \
													if (GETBIT(bk,(t))) { \
														bj = THREADBUFFERGROUPPOS((inttype) bk,0); \
														if (bj == 0) { \
															(a) = 0; \
														} \
														else { \
															GETPROCTRANSACT(k, bj); \
															if (k != (ac)) { \
																(a) = 0; \
															} \
														}\
													} \
												} \
												}
// check if bucket element associated with lane is a valid position to store data
#define LANEPOINTSTOVALIDBUCKETPOS						((LANE % HALFWARPSIZE) < ((HALFWARPSIZE / d_sv_nints)*d_sv_nints))

// HASH TABLE MACROS

// Return d_shared_q_size if duplicate found, d_shared_q_size+1 if cache is full
__device__ inttype STOREINCACHE(inttype* t, inttype bi, inttype bj, inttype bk, inttype bl, inttype bitmask, inttype hashtmp) {
	hashtmp = 0;
	for (bi = 0; bi < d_sv_nints; bi++) {
		hashtmp += STRIPPEDSTATE(t, bi);
	}
	bitmask = d_sv_nints*((inttype) (hashtmp % ((d_shared_q_size - CACHEOFFSET) / d_sv_nints)));
	bl = 0;
	while (bl < CACHERETRYFREQ) {
		bi = atomicCAS((inttype *) &shared[CACHEOFFSET+bitmask], EMPTYVECT32, t[0]); \
		if (bi == EMPTYVECT32) {
			for (bj = 1; bj < d_sv_nints; bj++) {
				shared[CACHEOFFSET+bitmask+bj] = t[bj];
			}
			return bitmask;
		}
		if (COMPAREENTRIES(bi, t[0])) {
			for (bj = 1; bj < d_sv_nints; bj++) { \
				if (STRIPPEDSTATE(&(shared[CACHEOFFSET+bitmask]), bj) != STRIPPEDSTATE(t, bj)) {
					return 0;
				}
			}
			return d_shared_q_size;
		}
		if (!ISNEWINT(bi)) {
			bj = atomicCAS((inttype *) &shared[CACHEOFFSET+bitmask], bi, t[0]);
			if (bi == bj) {
				for (bk = 1; bk < d_sv_nints; bk++) {
					shared[CACHEOFFSET+bitmask+bk] = t[bk];
				}
				return bitmask;
			}
		}
		bl++;
		bitmask += d_sv_nints;
		if ((bitmask+(d_sv_nints-1)) >= (d_shared_q_size - CACHEOFFSET)) {
			bitmask = 0;
		}
	}
	return d_shared_q_size+1;
}

// hash functions use bj variable
#define FIRSTHASH(a, t)							{	hashtmp = 0; \
													for (bj = 0; bj < d_sv_nints; bj++) { \
														hashtmp += (uint64_t) (d_h[0]*(STRIPPEDSTATE((t), bj))+d_h[1]); \
													} \
													(a) = WARPSIZE*((inttype) ((hashtmp % P) % d_nrbuckets)); \
												}
#define FIRSTHASHHOST(a)						{	uint64_t hashtmp = 0; \
													hashtmp = (uint64_t) h[1]; \
													(a) = WARPSIZE*((inttype) ((hashtmp % P) % q_size/WARPSIZE)); \
												}
#define HASHALL(a, i, t)						{	hashtmp = 0; \
													for (bj = 0; bj < d_sv_nints; bj++) { \
														hashtmp += (uint64_t) (shared[HASHCONSTANTSOFFSET+(2*(i))]*(STRIPPEDSTATE((t), bj))+shared[HASHCONSTANTSOFFSET+(2*(i))+1]); \
													} \
													(a) = WARPSIZE*((inttype) ((hashtmp % P) % d_nrbuckets)); \
												}
#define HASHFUNCTION(a, i, t)					((HASHALL((a), (i), (t))))

__device__ inttype COMPAREVECTORS(inttype* t1, inttype* t2, inttype bk) {
	for (bk = 0; bk < d_sv_nints; bk++) {
		if (STRIPPEDSTATE(t1, bk) != STRIPPEDSTATE(t2, bk)) {
			return 0;
		}
	}
	return 1;
}

// find or put element, single thread version.
__device__ inttype FINDORPUT_SINGLE(inttype* t, inttype* d_q, inttype bi, inttype bj, inttype bk, inttype bl, inttype bitmask, inttype hashtmp) {
	for (bi = 0; bi < NR_HASH_FUNCTIONS; bi++) {
		HASHFUNCTION(bitmask, bi, t);
		for (bj = 0; bj < NREL_IN_BUCKET; bj++) {
			bl = d_q[bitmask+(bj*d_sv_nints)];
			if (bl == EMPTYVECT32) {
				bl = atomicCAS(&d_q[bitmask+(bj*d_sv_nints)], EMPTYVECT32, t[0]);
				if (bl == EMPTYVECT32) {
					for (bk = 1; bk < d_sv_nints; bk++) {
						d_q[bitmask+(bj*d_sv_nints)+bk] = t[bk];
					}
				}
			}
			if (bl != EMPTYVECT32) {
				if (COMPAREVECTORS(&d_q[bitmask+(bj*d_sv_nints)], t, bk) == 1) {
					break;
				}
			}
			else {
				SETOLDSTATE(t);
				if (ITERATIONS < d_kernel_iters-1) {
					bk = atomicAdd((inttype *) &OPENTILECOUNT, d_sv_nints);
					if (bk < d_sv_nints*(blockDim.x / d_nr_procs)) {
						d_q[bitmask+(bj*d_sv_nints)+(d_sv_nints-1)] = t[d_sv_nints-1];
						for (bl = 0; bl < d_sv_nints; bl++) {
							shared[OPENTILEOFFSET+bk+bl] = NEWSTATEPART(t, bl);
						}
					}
				}
				return 1;
			}
		}
	}
	return 0;
}

// find or put element, warp version. t is element stored in block cache
#define FINDORPUT_WARP(a, t)				{	(a) = 0; \
												for (bi = 0; bi < NR_HASH_FUNCTIONS; bi++) { \
													HASHFUNCTION(bitmask, bi, (t)); \
													bl = d_q[bitmask+LANE]; \
													bk = __ballot(STRIPPEDSTATE(&bl, 0) == STRIPPEDSTATE((t), ENTRY_ID)); \
													if (LANEPOINTSTOVALIDBUCKETPOS) { \
														(a) = 0; \
														SETBITS(LANE-ENTRY_ID, LANE-ENTRY_ID+d_sv_nints-1, (a)); \
														if (bk & (a) == (a)) { \
															if (ENTRY_ID == d_sv_nints-1) { \
																SETOLDCACHEINT((t)[ENTRY_ID]); \
															} \
														} \
													} \
													(a) = 0; \
													if (!ISOLDCACHEINT((t)[d_sv_nints-1])) { \
														for (bj = 0; bj < NREL_IN_BUCKET; bj++) { \
															if (LANE_POINTS_TO_EL(bj)) { \
																bk = atomicCAS(&d_q[bitmask+LANE], EMPTYVECT32, (t)[ENTRY_ID]); \
																if (bk == EMPTYVECT32) { \
																	if (ENTRY_ID == 0) { \
																		SETOLDCACHEINT((t)[d_sv_nints-1]); \
																	} \
																	if (ITERATIONS < d_kernel_iters-1) { \
																		if (ENTRY_ID == 0) { \
																			bk = atomicAdd((inttype *) &OPENTILECOUNT, d_sv_nints); \
																			if (bk < OPENTILELEN) { \
																				d_q[bitmask+LANE+(d_sv_nints-1)] = (t)[d_sv_nints-1]; \
																			} \
																		} \
																		bk = __shfl(bk, LANE-ENTRY_ID); \
																		if (bk < OPENTILELEN) { \
																			shared[OPENTILEOFFSET+bk+ENTRY_ID] = NEWSTATEPART((t), ENTRY_ID); \
																		} \
																	} \
																} \
															} \
															if (ISOLDCACHEINT((t)[d_sv_nints-1])) { \
																break; \
															} \
														} \
													} \
													if (ISOLDCACHEINT((t)[d_sv_nints-1])) { \
														(a) = 1; \
														break; \
													} \
												} \
											}

// macro to print state vector
//#define PRINTVECTOR(s) 							{	printf ("("); \
//													for (bk = 0; bk < d_nr_procs; bk++) { \
//														bitmask = 0; \
//														SETBITS(shared[VECTORPOSOFFSET+bk],shared[VECTORPOSOFFSET+bk+1],bitmask); \
//														bj = (s & bitmask) >> shared[VECTORPOSOFFSET+bk]; \
//														printf ("%d", bj); \
//														if (bk < (d_nr_procs-1)) { \
//															printf (","); \
//														} \
//													} \
//													printf (")\n"); \
//												}
#define PRINTVECTOR(s) 							{	printf ("("); \
													for (bk = 0; bk < d_nr_procs; bk++) { \
														GETSTATEVECTORSTATE(bj, (s), bk) \
														printf ("%d", bj); \
														if (bk < (d_nr_procs-1)) { \
															printf (","); \
														} \
													} \
													printf (")\n"); \
												}


//#define INCRSTATEVECTOR(t)						(sv_nints == 1 ? t[0]++ : (t[0] == EMPTYVECTOR ? t[1]++ : t[0]++))
//#define DECRSTATEVECTOR(t)						(sv_nints == 1 ? t[0]-- : (t[0] == 0 ? (t[1]--; t[0] = EMPTYVECTOR) : t[0]--))

int vmem = 0;

// GPU textures
texture<inttype, 1, hipReadModeElementType> tex_proc_offsets_start;
texture<inttype, 1, hipReadModeElementType> tex_proc_offsets;
texture<inttype, 1, hipReadModeElementType> tex_proc_trans_start;
texture<inttype, 1, hipReadModeElementType> tex_proc_trans;
texture<inttype, 1, hipReadModeElementType> tex_syncbits_offsets;
texture<inttype, 1, hipReadModeElementType> tex_syncbits;

/**
 * This macro checks return value of the CUDA runtime call and exits
 * the application if the call failed.
 */
#define CUDA_CHECK_RETURN(value) {											\
	hipError_t _m_cudaStat = value;										\
	if (_m_cudaStat != hipSuccess) {										\
		fprintf(stderr, "Error %s at line %d in file %s\n",					\
				hipGetErrorString(_m_cudaStat), __LINE__, __FILE__);		\
		exit(1);															\
	} }

//wrapper around hipMalloc to count allocated memory and check for error while allocating
int cudaMallocCount ( void ** ptr,int size) {
	hipError_t err = hipSuccess;
	vmem += size;
	err = hipMalloc(ptr,size);
	if (err) {
		printf("Error %s at line %d in file %s\n", hipGetErrorString(err), __LINE__, __FILE__);
		exit(1);
	}
	fprintf (stdout, "allocated %d\n", size);
	return size;
}

//test function to print a given state vector
void print_statevector(inttype *state, inttype *firstbit_statevector, inttype nr_procs) {
	inttype i, s, bitmask, bi;

	for (i = 0; i < nr_procs; i++) {
		bitmask = 0;
		if (firstbit_statevector[i]/INTSIZE == firstbit_statevector[i+1]/INTSIZE) {
			SETBITS(firstbit_statevector[i] % INTSIZE,firstbit_statevector[i+1] % INTSIZE, bitmask);
			s = (state[firstbit_statevector[i]/INTSIZE] & bitmask) >> (firstbit_statevector[i] % INTSIZE);
		}
		else {
			SETBITS(0, firstbit_statevector[i+1] % INTSIZE, bitmask);
			s = (state[firstbit_statevector[i]/INTSIZE] >> (firstbit_statevector[i] % INTSIZE)
					| (state[firstbit_statevector[i+1]/INTSIZE] & bitmask) << (INTSIZE - (firstbit_statevector[i] % INTSIZE))); \
		}
		fprintf (stdout, "%d", s);
		if (i < (nr_procs-1)) {
			fprintf (stdout, ",");
		}
	}
	fprintf (stdout, "\n");
}

//test function to print the contents of the device queue
void print_queue(inttype *d_q, inttype q_size, inttype *firstbit_statevector, inttype nr_procs, inttype sv_nints) {
	inttype *q_test = (inttype*) malloc(sizeof(inttype)*q_size);
	hipMemcpy(q_test, d_q, q_size*sizeof(inttype), hipMemcpyDeviceToHost);
	inttype nw;
	int count = 0;
	int newcount = 0;
	for (inttype i = 0; i < (q_size/WARPSIZE); i++) {
		for (inttype j = 0; j < (WARPSIZE/sv_nints); j++) {
			if (q_test[(i*WARPSIZE)+(j*sv_nints)+(sv_nints-1)] != EMPTYVECT32) {
				count++;
				nw = ISNEWSTATE_HOST(&q_test[(i*WARPSIZE)+(j*sv_nints)]);
				if (nw) {
					newcount++;
					fprintf (stdout, "new: ");
				}
				print_statevector(&(q_test[(i*WARPSIZE)+(j*sv_nints)]), firstbit_statevector, nr_procs);
			}
		}
	}
	fprintf (stdout, "nr. of states in hash table: %d (%d unexplored states)\n", count, newcount);
}

//test function to print the contents of the device queue
void print_local_queue(inttype *q, inttype q_size, inttype *firstbit_statevector, inttype nr_procs, inttype sv_nints) {
	int count = 0, newcount = 0;
	inttype nw;
	for (inttype i = 0; i < (q_size/WARPSIZE); i++) {
		for (inttype j = 0; j < (WARPSIZE/sv_nints); j++) {
			if (q[(i*WARPSIZE)+(j*sv_nints)+(sv_nints-1)] != EMPTYVECT32) {
				count++;
				nw = ISNEWSTATE_HOST(&q[(i*WARPSIZE)+(j*sv_nints)]);
				if (nw) {
					newcount++;
					fprintf (stdout, "new: ");
					//print_statevector(&(q[(i*32)+(j*sv_nints)]), firstbit_statevector, nr_procs);
				}
				print_statevector(&(q[(i*WARPSIZE)+(j*sv_nints)]), firstbit_statevector, nr_procs);
			}
		}
	}
	fprintf (stdout, "nr. of states in hash table: %d (%d unexplored states)\n", count, newcount);
}

//test function to count the contents of the device queue
void count_queue(inttype *d_q, inttype q_size, inttype *firstbit_statevector, inttype nr_procs, inttype sv_nints) {
	inttype *q_test = (inttype*) malloc(sizeof(inttype)*q_size);
	hipMemcpy(q_test, d_q, q_size*sizeof(inttype), hipMemcpyDeviceToHost);

	int count = 0;
	for (inttype i = 0; i < (q_size/WARPSIZE); i++) {
		for (inttype j = 0; j < (WARPSIZE/sv_nints); j++) {
			if (q_test[(i*WARPSIZE)+(j*sv_nints)+(sv_nints-1)] != EMPTYVECT32) {
				count++;
			}
		}
	}
	fprintf (stdout, "nr. of states in hash table: %d\n", count);
}

//test function to count the contents of the host queue
void count_local_queue(inttype *q, inttype q_size, inttype *firstbit_statevector, inttype nr_procs, inttype sv_nints) {
	int count = 0, newcount = 0;
	inttype nw;
	for (inttype i = 0; i < (q_size/WARPSIZE); i++) {
		for (inttype j = 0; j < (WARPSIZE/sv_nints); j++) {
			if (q[(i*WARPSIZE)+(j*sv_nints)+(sv_nints-1)] != EMPTYVECT32) {
				count++;
				nw = ISNEWSTATE_HOST(&q[(i*WARPSIZE)+(j*sv_nints)]);
				if (nw) {
					newcount++;
				}
			}
		}
	}
	fprintf (stdout, "nr. of states in hash table: %d (%d unexplored states)\n", count, newcount);
}

/**
 * CUDA kernel function to initialise the queue
 */
__global__ void init_queue(inttype *d_q, inttype n_elem) {
    inttype nthreads = blockDim.x*gridDim.x;
    inttype i = (blockIdx.x *blockDim.x) + threadIdx.x;

    for(; i < n_elem; i += nthreads) {
    	d_q[i] = (inttype) EMPTYVECT32;
    }
}

/**
 * CUDA kernel to store initial state in hash table
 */
__global__ void store_initial(inttype *d_q, inttype *d_h) {
	inttype bj, h;
	uint64_t hashtmp;
	inttype state[MAX_SIZE];

	for (bj = 0; bj < d_sv_nints; bj++) {
		state[bj] = 0;
	}
	SETNEWSTATE(state);
	FIRSTHASH(h, state);
	for (bj = 0; bj < d_sv_nints; bj++) {
		d_q[h+bj] = state[bj];
	}
}

/**
 * CUDA kernel function for BFS iteration state gathering
 * Order of data in the shared queue:
 * (0. index of process LTS states sizes)
 * (1. index of sync rules offsets)
 * (2. index of sync rules)
 * (1. index of open queue tile)
 * 0. the 'iterations' flag to count the number of iterations so far (nr of tiles processed by SM)
 * 1. the 'continue' flag for thread work
 * (4. index of threads buffer)
 * (5. index of hash table)
 * 2. constants for d_q hash functions (2 per function, in total 8 by default)
 * 3. state vector offsets (nr_procs+1 elements)
 * 4. sizes of states in process LTS states (nr_procs elements)
 * (9. sync rules + offsets (nr_syncbits_offsets + nr_syncbits elements))
 * 5. tile of open queue to be processed by block (sv_nints*(blockDim.x / nr_procs) elements)
 * 6. buffer for threads ((blockDim.x*max_buf_ints)+(blockDim.x/nr_procs) elements)
 * 7. hash table
 */
__global__ void gather(inttype *d_q, inttype *d_h, inttype *d_bits_state,
						inttype *d_firstbit_statevector, inttype *d_proc_offsets_start,
						inttype *d_proc_offsets, inttype *d_proc_trans, inttype *d_syncbits_offsets,
						inttype *d_syncbits, inttype *d_contBFS, inttype *d_property_violation, inttype scan) {
	//inttype global_id = (blockIdx.x * blockDim.x) + threadIdx.x;
	//inttype group_nr = threadIdx.x / nr_procs;
	inttype i, k, l, index, offset1, offset2, tmp, cont, act, sync_offset1, sync_offset2;
	inttype src_state[MAX_SIZE], tgt_state[MAX_SIZE];
	inttype bitmask, bi, bj, bk, bl;
	uint64_t hashtmp;
	int pos;
	// is at least one outgoing transition enabled for a given state (needed to detect deadlocks)
	inttype outtrans_enabled;

	for (i = threadIdx.x; i < d_shared_q_size; i += blockDim.x) {
		shared[i] = 0;
	}
	// Locally store the state sizes and syncbits
	i = threadIdx.x;
	if (i == 0) {
		ITERATIONS = 0;
		OPENTILECOUNT = 0;
	}
	if ((blockIdx.x*blockDim.x)+threadIdx.x == 0) {
		(*d_contBFS) = 0;
	}
	for (i = threadIdx.x; i < HASHCONSTANTSLEN; i += blockDim.x) {
		shared[i+HASHCONSTANTSOFFSET] = d_h[i];
	}
	for (i = threadIdx.x; i < VECTORPOSLEN; i += blockDim.x) {
		shared[i+VECTORPOSOFFSET] = d_firstbit_statevector[i];
	}
	for (i = threadIdx.x; i < LTSSTATESIZELEN; i += blockDim.x) {
		shared[i+LTSSTATESIZEOFFSET] = d_bits_state[i];
	}
	// Reset the open queue tile
	if (threadIdx.x < d_sv_nints*(blockDim.x / d_nr_procs)) {
		shared[OPENTILEOFFSET+threadIdx.x] = EMPTYVECT32;
	}
	// Clean the cache
	i = threadIdx.x;
	while (i < (d_shared_q_size - CACHEOFFSET)) {
		shared[CACHEOFFSET + i] = EMPTYVECT32;
		i += blockDim.x;
	}
	__syncthreads();
	while (ITERATIONS < d_kernel_iters) {
		// Scan the open set for work; we use the OPENTILECOUNT flag at this stage to count retrieved elements
		if (scan || ITERATIONS == 0) {
			for (i = GLOBAL_WARP_ID; i < d_nrbuckets && OPENTILECOUNT < OPENTILELEN; i += NR_WARPS) {
				src_state[0] = d_q[(i*WARPSIZE)+LANE];
				if (ENTRY_ID == (d_sv_nints-1)) {
					if (ISNEWINT(src_state[0])) {
						// try to increment the OPENTILECOUNT counter, if successful, store the state
						l = atomicAdd((inttype *) &OPENTILECOUNT, d_sv_nints);
						if (l >= OPENTILELEN) {
							src_state[0] = 0;
						}
						else {
							// vector has been claimed for exploration. make it old in the global hash table
							d_q[(i*WARPSIZE)+LANE] = OLDINT(src_state[0]);
						}
					}
					else {
						src_state[0] = 0;
					}
				}
				// all threads read the outcome of the above procedure, and if positive, store their part of the vector in the shared memory
				if (LANEPOINTSTOVALIDBUCKETPOS) {
					k = __shfl(src_state[0], LANE-ENTRY_ID+d_sv_nints-1);
					if (k != 0) {
						// get offset l
						l == __shfl(l, LANE-ENTRY_ID+d_sv_nints-1);
						// write part of vector to shared memory
						shared[OPENTILEOFFSET+l+ENTRY_ID] = src_state[0];
					}
				}
			}
		}
		__syncthreads();
		// if work has been retrieved, indicate this
		if (threadIdx.x == 0) {
			if (OPENTILECOUNT > 0) {
				(*d_contBFS) = 1;
			}
		}
		// is the thread part of an 'active' group?
		offset1 = 0;
		offset2 = 0;
		// both group 'leaders' (group id 0) and threads at the start of a warp should read vector states,
		// to be distributed among the other threads via shuffle operations
		if (THREADINGROUP) {
			if (LANE == 0 || GROUP_ID == 0) {
				for (l = 0; l < d_sv_nints; l++) {
					src_state[l] = OPENTILESTATEPART(l);
				}
			}
			// every thread reads the state from the appropriate leader thread
			for (l = 0; l < d_sv_nints; l++) {
				src_state[l] = __shfl(src_state[l], (GROUP_ID > LANE) ? 0 : LANE-GROUP_ID);
			}
		}
		// Reset the open queue tile
		if (threadIdx.x < d_sv_nints*(blockDim.x / d_nr_procs)) {
			shared[OPENTILEOFFSET+threadIdx.x] = EMPTYVECT32;
		}
		if (threadIdx.x == 0) {
			OPENTILECOUNT = 0;
		}
		__syncthreads();
		if (THREADINGROUP) {
			// Is there work?
			if (ISSTATE(src_state)) {
				// Gather the required transition information for all states in the tile
				i = tex1Dfetch(tex_proc_offsets_start, GROUP_ID);
				// Determine process state
				GETSTATEVECTORSTATE(cont, src_state, GROUP_ID);
				// Offset position
				index = cont/(INTSIZE/d_nbits_offset);
				pos = cont - (index*(INTSIZE/d_nbits_offset));
				tmp = tex1Dfetch(tex_proc_offsets, i+index);
				GETTRANSOFFSET(offset1, tmp, pos);
				if (pos == (INTSIZE/d_nbits_offset)-1) {
					tmp = tex1Dfetch(tex_proc_offsets, i+index+1);
					GETTRANSOFFSET(offset2, tmp, 0);
				}
				else {
					GETTRANSOFFSET(offset2, tmp, pos+1);
				}
			}
			if (GROUP_ID == 0) {
				// for later, when constructing successors for this state, set action counter to maximum
				THREADGROUPCOUNTER = (1 << d_bits_act);
			}
		}
		// iterate over the outgoing transitions of state 'cont'
		// variable cont is reused to indicate whether the buffer content of this thread still needs processing
		cont = 0;
		if (threadIdx.x == 0) {
			CONTINUE = 1;
		}
		__syncthreads();
		// while there is work to be done
		//int loopcounter = 0;
		outtrans_enabled = 0;
		while (CONTINUE == 1) {
		// for (loopcounter = 0; loopcounter < 10000 && CONTINUE == 1; loopcounter++) {
			if (offset1 < offset2 || cont) {
				if (!cont) {
					// reset act
					act = (1 << (d_bits_act));
					// reset buffer of this thread
					for (l = 0; l < d_max_buf_ints; l++) {
						THREADBUFFERGROUPPOS(GROUP_ID, l) = 0;
					}
					// if not sync, store in hash table
					while (offset1 < offset2) {
						tmp = tex1Dfetch(tex_proc_trans, offset1);
						GETPROCTRANSSYNC(bitmask, tmp);
						if (bitmask == 0) {
							// no deadlock
							outtrans_enabled = 1;
							// construct state
							for (l = 0; l < d_sv_nints; l++) {
								tgt_state[l] = src_state[l];
							}
							for (l = 1; l <= NR_OF_STATES_IN_TRANSENTRY(GROUP_ID); l++) {
								GETPROCTRANSSTATE(pos, tmp, l, GROUP_ID);
								if (pos > 0) {
									//printf ("%d %d\n", l, pos);
									SETSTATEVECTORSTATE(tgt_state, GROUP_ID, pos-1);
									// check for violation of safety property, if required
									if (d_property == 2) {
										if (GROUP_ID == d_nr_procs-1) {
											// pos contains state id + 1
											// error state is state 1
											if (pos == 2) {
												// error state found
												(*d_property_violation) = 1;
											}
										}
									}
									// store tgt_state in cache; if i == d_shared_q_size, state was found, duplicate detected
									// if i == d_shared_q_size+1, cache is full, immediately store in global hash table
									//printf ("src\n");
									//PRINTVECTOR(src_state);
									//printf ("storing\n");
									//PRINTVECTOR(tgt_state);
									// cache time-out; store directly in global hash table
									if (STOREINCACHE(tgt_state, bi, bj, bk, bl, bitmask, hashtmp) > d_shared_q_size) {
										if (FINDORPUT_SINGLE(tgt_state, d_q, bi, bj, bk, bl, bitmask, hashtmp)  == 0) {
											// ERROR! hash table too full. Set CONTINUE to 2
											CONTINUE = 2;
										}
									}
								}
								else {
									break;
								}
							}
							//printf ("out\n");
							offset1++;
						}
						else {
							break;
						}
					}

					// i is the current relative position in the buffer for this thread
					i = 0;
					if (offset1 < offset2) {
						GETPROCTRANSACT(act, tmp);
						//PRINTTHREAD(0, act);
						// store transition entry
						THREADBUFFERGROUPPOS(GROUP_ID,i) = tmp;
						cont = 1;
						i++;
						offset1++;
						while (offset1 < offset2) {
							tmp = tex1Dfetch(tex_proc_trans, offset1);
							GETPROCTRANSACT(bitmask, tmp);
							if (act == bitmask) {
								THREADBUFFERGROUPPOS(GROUP_ID,i) = tmp;
								i++;
								offset1++;
							}
							else {
								break;
							}
						}
					}
				}
			}
			// group leaders now need to set the counter to the next minimal action value.
			// To avoid bank conflicts afterwards when threads would need to read THREADGROUPCOUNTER,
			// the leader disables the SYNC bit of transition entries for those threads which need
			// to perform work next. In this way, threads can determine locally that they should proceed
			// without reading the counter.
			__syncthreads();
			if (THREADINGROUP) {
				if (GROUP_ID == 0) {
					if (THREADGROUPCOUNTER != EXPLORATION_DONE) {
						pos = (1 << d_bits_act);
						for (i = 0; i < d_nr_procs; i++) {
							l = THREADBUFFERGROUPPOS(i,0);
							if (l != 0) {
								GETPROCTRANSACT(bitmask, l);
								if (THREADGROUPCOUNTER == (1 << d_bits_act)) {
									if (bitmask < pos) {
										pos = bitmask;
									}
								}
								else {
									if (bitmask < pos && bitmask > THREADGROUPCOUNTER) {
										pos = bitmask;
									}
								}
							}
						}
						// if pos = THREADGROUPCOUNTER, no more work for this state is required, now or in future iterations
						if (pos == THREADGROUPCOUNTER) {
							THREADGROUPCOUNTER = EXPLORATION_DONE;
						}
						// store the value
						if (pos < (1 << d_bits_act)) {
							THREADGROUPCOUNTER = pos;
						}
						//if (threadIdx.x == 0) {
						//	printf ("counter value: %d\n", pos);
						//}
						// notify threads to work
						for (i = 0; i < d_nr_procs; i++) {
							l = THREADBUFFERGROUPPOS(i,0);
							if (l != 0) {
								GETPROCTRANSACT(bitmask, l);
								if (bitmask == THREADGROUPCOUNTER) {
									// notify
									SETPROCTRANSSYNC(THREADBUFFERGROUPPOS(i,0),0);
								}
							}
						}
					}
				}
			}
			//if (0) {
			__syncthreads();
			// only active threads should do something
			if (cont) {
				// Now, we have obtained the info needed to combine process transitions
				// if the sync bit has been disabled, come into action, creating successors
				GETPROCTRANSSYNC(index, THREADBUFFERGROUPPOS(GROUP_ID,0));
				if (index == 0) {
					//PRINTTHREAD(6, act);
					// syncbits Offset position
					i = act/(INTSIZE/d_nbits_syncbits_offset);
					pos = act - (i*(INTSIZE/d_nbits_syncbits_offset));
					l = tex1Dfetch(tex_syncbits_offsets, i);
					GETSYNCOFFSET(sync_offset1, l, pos);
					if (pos == (INTSIZE/d_nbits_syncbits_offset)-1) {
						l = tex1Dfetch(tex_syncbits_offsets, i+1);
						GETSYNCOFFSET(sync_offset2, l, 0);
					}
					else {
						GETSYNCOFFSET(sync_offset2, l, pos+1);
					}
					// iterate through the relevant syncbit filters
					tmp = 1;
					for (;sync_offset1 < sync_offset2 && tmp; sync_offset1++) {
						index = tex1Dfetch(tex_syncbits, sync_offset1);
						for (i = 0; i < (INTSIZE/d_nr_procs); i++) {
							GETSYNCRULE(tmp, index, i);
							//PRINTTHREAD(7, tmp);
							if (tmp != 0) {
								OWNSSYNCRULE(bitmask, tmp, GROUP_ID);
							}
							else {
								bitmask = 0;
							}
							if (bitmask) {
								// start combining entries in the buffer to create target states
								// if sync rule applicable, construct the first successor
								// copy src_state into tgt_state
								SYNCRULEISAPPLICABLE(l, tmp, act);
								if (l) {
									// source state is not a deadlock
									outtrans_enabled = 1;
									for (pos = 0; pos < d_sv_nints; pos++) {
										tgt_state[pos] = src_state[pos];
									}
									//printf("src\n");
									//PRINTVECTOR(src_state);
									//printf("%d\n", src_state[0]);
									//printf("%d\n", act);
									// construct first successor
									for (pos = 0; pos < d_nr_procs; pos++) {
										if (GETBIT(pos, tmp)) {
											// get first state
											GETPROCTRANSSTATE(k, THREADBUFFERGROUPPOS(pos,0), 1, pos);
											//PRINTTHREAD(1, THREADBUFFERGROUPPOS(pos,0));
											//PRINTTHREAD(2, k);
											SETSTATEVECTORSTATE(tgt_state, pos, k-1);
										}
									}
									//PRINTVECTOR(tgt_state);
									//STRIPSTATE(tgt_state);
									//PRINTTHREAD(3, tgt_state[0]);
									SETNEWSTATE(tgt_state);
									// while we keep getting new states, store them
									while (ISNEWSTATE(tgt_state)) {
										// check for violation of safety property, if required
										if (d_property == 2) {
											GETSTATEVECTORSTATE(pos, tgt_state, d_nr_procs-1);
											if (pos == 1) {
												// error state found
												(*d_property_violation) = 1;
											}
										}

										// store tgt_state in cache; if i == d_shared_q_size, state was found, duplicate detected
										// if i == d_shared_q_size+1, cache is full, immediately store in global hash table
										//printf ("store in cache\n");
										//PRINTTHREAD(0, 10);
										//PRINTTHREAD(0, 11);
										// state stored: search for it in global hash table
										//if (pos < d_shared_q_size) {
										//	LOOKUPINCLOSED(k, tgt_state);
										//	// state found, set old in shared cache
										//	if (k < q_size) {
										//		printf ("already in hash table\n");
										//		SETOLDSTATE(&shared[CACHEOFFSET+pos]);
										//	}
										//}
										// cache time-out; store directly in global hash table
										if (STOREINCACHE(tgt_state, bi, bj, bk, bl, bitmask, hashtmp) > d_shared_q_size) {
											if (FINDORPUT_SINGLE(d_q, tgt_state, bi, bj, bk, bl, bitmask, hashtmp) == 0) {
												// ERROR! hash table too full. Set CONTINUE to 2
												CONTINUE = 2;
											}
										}
										// get next successor
										for (pos = d_nr_procs-1; pos > (int) GROUP_ID-1; pos--) {
											if (GETBIT(pos,tmp)) {
												GETSTATEVECTORSTATE(cont, tgt_state, pos);
												act = 0;
												for (k = 0; k < d_max_buf_ints; k++) {
													for (l = 1; l <= NR_OF_STATES_IN_TRANSENTRY(pos); l++) {
														GETPROCTRANSSTATE(act, THREADBUFFERGROUPPOS(pos,k), l, pos);
														if (cont == (act-1)) {
															break;
														}
													}
													if (cont == (act-1)) {
														break;
													}
												}
												// Assumption: element has been found (otherwise, 'last' was not a valid successor)
												// Try to get the next element
												if (l == NR_OF_STATES_IN_TRANSENTRY(pos)) {
													if (k >= d_max_buf_ints-1) {
														act = 0;
													}
													else {
														k++;
														l = 1;
													}
												}
												else {
													l++;
												}
												// Retrieve next element, insert it in 'tgt_state' if it is not 0, and return result, otherwise continue
												if (act != 0) {
													GETPROCTRANSSTATE(act, THREADBUFFERGROUPPOS(pos,k), l, pos);
													if (act > 0) {
														SETSTATEVECTORSTATE(tgt_state, pos, act-1);
														SETNEWSTATE(tgt_state);
														break;
													}
												}
												// else, set this process state to first one, and continue to next process
												GETPROCTRANSSTATE(act, THREADBUFFERGROUPPOS(pos,0), 1, pos);
												SETSTATEVECTORSTATE(tgt_state, pos, act-1);
											}
										}
										// did we find a successor? if not, set tgt_state to old
										if (pos == (int) GROUP_ID-1) {
											SETOLDSTATE(tgt_state);
										}
									}
								}
							}
						}
					}
					cont = 0;
				}
			}
			// finished an iteration of adding states.
			// Is there still work? (is another iteration required?)
			if (threadIdx.x == 0) {
				if (CONTINUE != 2) {
					CONTINUE = 0;
				}
			}
			__syncthreads();
			if (THREADINGROUP) {
				if ((offset1 < offset2) || cont) {
					if (CONTINUE != 2) {
						CONTINUE = 1;
					}
				}
			}
			// FOR TEST PURPOSES!
//			if (threadIdx.x == 0) {
//				CONTINUE++;
//			}
			__syncthreads();
		} // END WHILE CONTINUE == 1
		// have we encountered a deadlock state?
		// we use the shared memory to communicate this to the group leaders
		if (d_property == 1) {
			if (THREADINGROUP) {
				if (ISSTATE(src_state)) {
					THREADBUFFERGROUPPOS(GROUP_ID, 0) = outtrans_enabled;
					// group leader collects results
					l = 0;
					if (GROUP_ID == 0) {
						for (i = 0; i < d_nr_procs; i++) {
							l += THREADBUFFERGROUPPOS(i, 0);
						}
						if (l == 0) {
							// deadlock state found
							(*d_property_violation) = 1;
						}
					}
				}
			}
		}
		//if (loopcounter == 10000 && CONTINUE == 1) {
		//	PRINTTHREAD(0,offset1)
		//	PRINTTHREAD(1,offset2)
		//	PRINTTHREAD(2,cont)
		//}
		//if (THREADINGROUP) {
		//	if (offset1 < offset2) {
		//		PRINTTHREAD(0,offset1)
		//		PRINTTHREAD(1,offset2)
		//		PRINTTHREAD(3,GROUP_ID)
		//		PRINTVECTOR(src_state)
		//	}
		//	if (cont) {
		//		PRINTTHREAD(2,cont)
		//	}
		//}
		// start scanning the local cache and write results to the global hash table
		k = (d_shared_q_size-CACHEOFFSET)/d_sv_nints;
		for (i = WARP_ID; i < k; i += (blockDim.x/WARPSIZE)) {
			if (ISNEWSTATE(&shared[CACHEOFFSET+(i*d_sv_nints)])) {
				// look for the state in the global hash table
				FINDORPUT_WARP(l, &shared[CACHEOFFSET+(i*d_sv_nints)]);
				if (l == 0) {
					// ERROR: hash table is full
					CONTINUE = 2;
				}
			}
		}
		__syncthreads();
		// Ready to start next iteration, if error has not occurred
		if (threadIdx.x == 0) {
			if (CONTINUE == 2) {
				(*d_contBFS) = 2;
				ITERATIONS = d_kernel_iters;
			}
			else {
				ITERATIONS++;
			}
			CONTINUE = 0;
		}
		__syncthreads();
	}
}

/**
 * Host function that prepares data array and passes it to the CUDA kernel.
 */
int main(int argc, char** argv) {
	FILE *fp;
	inttype nr_procs, bits_act, bits_statevector, sv_nints, nr_trans, proc_nrstates, nbits_offset, max_buf_ints, nr_syncbits_offsets, nr_syncbits, nbits_syncbits_offset;
	inttype *bits_state, *firstbit_statevector, *proc_offsets, *proc_trans, *proc_offsets_start, *syncbits_offsets, *syncbits;
	inttype contBFS;
	char stmp[BUFFERSIZE], fn[50];
	// to store constants for closed set hash functions
	int h[NR_HASH_FUNCTIONS*2];
	// size of global hash table
	size_t q_size = 0;
	// size of open set
	inttype opensize = 0;
	inttype check_property = 0;
	// nr of iterations in single kernel run
	int kernel_iters = KERNEL_ITERS;
	int nblocks = NR_OF_BLOCKS;
	int nthreadsperblock = BLOCK_SIZE;
	// level of verbosity (1=print level progress)
	int verbosity = 0;
	// clock to measure time
	clock_t start, stop;
	double runtime = 0.0;

	// Start timer
	assert((start = clock())!=-1);

	hipDeviceProp_t prop;
	int nDevices;

	// GPU side versions of the input
	inttype *d_bits_state, *d_firstbit_statevector, *d_proc_offsets_start, *d_proc_offsets, *d_proc_trans, *d_syncbits_offsets, *d_syncbits, *d_h;
	// flag to keep track of progress and whether hash table errors occurred (value==2)
	inttype *d_contBFS;
	// flag to keep track of property verification outcome
	inttype *d_property_violation;

	// GPU datastructures for calculation
	inttype *d_q;
	// GPU queue indices
	//inttype d_q_open, d_q_next, d_q_end;

	if (argc == 1) {
		fprintf(stderr, "ERROR: No input network given!\n");
		exit(1);
	}

	strcpy(fn, argv[1]);
	strcat(fn, ".gpf");

	int i = 2;
	while (i < argc) {
		printf ("%s\n", argv[i]);
		if (!strcmp(argv[i],"-k")) {
			// if nr. of iterations per kernel run is given, store it
			kernel_iters = atoi(argv[i+1]);
			i += 2;
		}
		else if (!strcmp(argv[i],"-b")) {
			// store nr of blocks to be used
			nblocks = atoi(argv[i+1]);
			i += 2;
		}
		else if (!strcmp(argv[i],"-t")) {
			// store nr of threads per block to be used
			nthreadsperblock = atoi(argv[i+1]);
			i += 2;
		}
		else if (!strcmp(argv[i],"-q")) {
			// store hash table size
			q_size = atoi(argv[i+1]);
			i += 2;
		}
		else if (!strcmp(argv[i],"-v")) {
			// store verbosity level
			verbosity = atoi(argv[i+1]);
			if (verbosity > 3) {
				verbosity = 3;
			}
			i += 2;
		}
		else if (!strcmp(argv[i],"-o")) {
			// store open set size
			opensize = atoi(argv[i+1]);
			i += 2;
		}
		else if (!strcmp(argv[i],"-d")) {
			// check for deadlocks
			check_property = 1;
			i += 1;
		}
		else if (!strcmp(argv[i],"-s")) {
			// check a safety property
			check_property = 2;
			i += 1;
		}
	}

	fp = fopen(fn, "r");
	if (fp) {
		// Read the input
		fgets(stmp, BUFFERSIZE, fp);
		nr_procs = atoi(stmp);
		fprintf(stdout, "nr of procs: %d\n", nr_procs);
		fgets(stmp, BUFFERSIZE, fp);
		bits_act = atoi(stmp);
		fprintf(stdout, "nr of bits for transition label: %d\n", bits_act);
		fgets(stmp, BUFFERSIZE, fp);
		proc_nrstates = atoi(stmp);
		fprintf(stdout, "min. nr. of proc. states that fit in 32-bit integer: %d\n", proc_nrstates);
		fgets(stmp, BUFFERSIZE, fp);
		bits_statevector = atoi(stmp);
		fprintf(stdout, "number of bits needed for a state vector: %d\n", bits_statevector);
		firstbit_statevector = (inttype*) malloc(sizeof(inttype)*(nr_procs+1));
		for (int i = 0; i <= nr_procs; i++) {
			fgets(stmp, BUFFERSIZE, fp);
			firstbit_statevector[i] = atoi(stmp);
			fprintf(stdout, "statevector offset %d: %d\n", i, firstbit_statevector[i]);
		}
		// determine the number of integers needed for a state vector
		sv_nints = (bits_statevector+31) / INTSIZE;
		bits_state = (inttype*) malloc(sizeof(inttype)*nr_procs);
		for (int i = 0; i < nr_procs; i++) {
			fgets(stmp, BUFFERSIZE, fp);
			bits_state[i] = atoi(stmp);
			fprintf(stdout, "bits for states of process LTS %d: %d\n", i, bits_state[i]);
		}
		fgets(stmp, BUFFERSIZE, fp);
		nbits_offset = atoi(stmp);
		fprintf(stdout, "size of offset in process LTSs: %d\n", nbits_offset);
		fgets(stmp, BUFFERSIZE, fp);
		max_buf_ints = atoi(stmp);
		fprintf(stdout, "maximum label-bounded branching factor: %d\n", max_buf_ints);
		proc_offsets_start = (inttype*) malloc(sizeof(inttype)*(nr_procs+1));
		for (int i = 0; i <= nr_procs; i++) {
			fgets(stmp, BUFFERSIZE, fp);
			proc_offsets_start[i] = atoi(stmp);
		}
		proc_offsets = (inttype*) malloc(sizeof(inttype)*proc_offsets_start[nr_procs]);
		for (int i = 0; i < proc_offsets_start[nr_procs]; i++) {
			fgets(stmp, BUFFERSIZE, fp);
			proc_offsets[i] = atoi(stmp);
		}
		fgets(stmp, BUFFERSIZE, fp);
		nr_trans = atoi(stmp);
		fprintf(stdout, "total number of transition entries in network: %d\n", nr_trans);
		proc_trans = (inttype*) malloc(sizeof(inttype)*nr_trans);
		for (int i = 0; i < nr_trans; i++) {
			fgets(stmp, BUFFERSIZE, fp);
			proc_trans[i] = atoi(stmp);
		}

		fgets(stmp, BUFFERSIZE, fp);
		nbits_syncbits_offset = atoi(stmp);
		//fprintf(stdout, "size of offset in sync rules: %d\n", nbits_syncbits_offset);
		fgets(stmp, BUFFERSIZE, fp);
		nr_syncbits_offsets = atoi(stmp);
		syncbits_offsets = (inttype*) malloc(sizeof(inttype)*nr_syncbits_offsets);
		for (int i = 0; i < nr_syncbits_offsets; i++) {
			fgets(stmp, BUFFERSIZE, fp);
			syncbits_offsets[i] = atoi(stmp);
			//fprintf(stdout, "syncbits offset %d: %d\n", i, syncbits_offsets[i]);
		}
		fgets(stmp, BUFFERSIZE, fp);
		nr_syncbits = atoi(stmp);
		syncbits = (inttype*) malloc(sizeof(inttype)*nr_syncbits);
		for (int i = 0; i < nr_syncbits; i++) {
			fgets(stmp, BUFFERSIZE, fp);
			syncbits[i] = atoi(stmp);
			//fprintf(stdout, "syncbits %d: %d\n", i, syncbits[i]);
		}
	}
	else {
		fprintf(stderr, "ERROR: input network does not exist!\n");
		exit(1);
	}

	// Randomly define the closed set hash functions
	srand(time(NULL));
	for (int i = 0; i < NR_HASH_FUNCTIONS*2; i++) {
		h[i] = rand();
	}

	// continue flags
	contBFS = 1;

	// Allocate memory on GPU
	cudaMallocCount((void **) &d_contBFS, sizeof(inttype));
	cudaMallocCount((void **) &d_property_violation, sizeof(inttype));
	cudaMallocCount((void **) &d_h, NR_HASH_FUNCTIONS*2*sizeof(inttype));
	cudaMallocCount((void **) &d_bits_state, nr_procs*sizeof(inttype));
	cudaMallocCount((void **) &d_firstbit_statevector, (nr_procs+1)*sizeof(inttype));
	cudaMallocCount((void **) &d_proc_offsets_start, (nr_procs+1)*sizeof(inttype));
	cudaMallocCount((void **) &d_proc_offsets, proc_offsets_start[nr_procs]*sizeof(inttype));
	cudaMallocCount((void **) &d_proc_trans, nr_trans*sizeof(inttype));
	cudaMallocCount((void **) &d_syncbits_offsets, nr_syncbits_offsets*sizeof(inttype));
	cudaMallocCount((void **) &d_syncbits, nr_syncbits*sizeof(inttype));

	// Copy data to GPU
	CUDA_CHECK_RETURN(hipMemcpy(d_contBFS, &contBFS, sizeof(inttype), hipMemcpyHostToDevice))
	CUDA_CHECK_RETURN(hipMemcpy(d_h, h, NR_HASH_FUNCTIONS*2*sizeof(inttype), hipMemcpyHostToDevice))
	CUDA_CHECK_RETURN(hipMemcpy(d_bits_state, bits_state, nr_procs*sizeof(inttype), hipMemcpyHostToDevice))
	CUDA_CHECK_RETURN(hipMemcpy(d_firstbit_statevector, firstbit_statevector, (nr_procs+1)*sizeof(inttype), hipMemcpyHostToDevice))
	CUDA_CHECK_RETURN(hipMemcpy(d_proc_offsets_start, proc_offsets_start, (nr_procs+1)*sizeof(inttype), hipMemcpyHostToDevice))
	CUDA_CHECK_RETURN(hipMemcpy(d_proc_offsets, proc_offsets, proc_offsets_start[nr_procs]*sizeof(inttype), hipMemcpyHostToDevice))
	CUDA_CHECK_RETURN(hipMemcpy(d_proc_trans, proc_trans, nr_trans*sizeof(inttype), hipMemcpyHostToDevice))
	CUDA_CHECK_RETURN(hipMemcpy(d_syncbits_offsets, syncbits_offsets, nr_syncbits_offsets*sizeof(inttype), hipMemcpyHostToDevice))
	CUDA_CHECK_RETURN(hipMemcpy(d_syncbits, syncbits, nr_syncbits*sizeof(inttype), hipMemcpyHostToDevice))

	// Bind data to textures
	hipBindTexture(NULL, tex_proc_offsets_start, d_proc_offsets_start, (nr_procs+1)*sizeof(inttype));
	hipBindTexture(NULL, tex_proc_offsets, d_proc_offsets, proc_offsets_start[nr_procs]*sizeof(inttype));
	hipBindTexture(NULL, tex_proc_trans, d_proc_trans, nr_trans*sizeof(inttype));
	hipBindTexture(NULL, tex_syncbits_offsets, d_syncbits_offsets, nr_syncbits_offsets*sizeof(inttype));
	hipBindTexture(NULL, tex_syncbits, d_syncbits, nr_syncbits*sizeof(inttype));

	// Query the device properties and determine data structure sizes
	hipGetDeviceCount(&nDevices);
	if (nDevices == 0) {
		fprintf (stderr, "ERROR: No CUDA compatible GPU detected!\n");
		exit(1);
	}
	hipGetDeviceProperties(&prop, 0);
	fprintf (stdout, "global mem: %d\n", (int) prop.totalGlobalMem);
	fprintf (stdout, "shared mem per block: %d\n", (int) prop.sharedMemPerBlock);
	fprintf (stdout, "max. threads per block: %d\n", (int) prop.maxThreadsPerBlock);
	fprintf (stdout, "max. grid size: %d\n", (int) prop.maxGridSize[0]);
	fprintf (stdout, "nr. of multiprocessors: %d\n", (int) prop.multiProcessorCount);

	size_t available, total;
	hipMemGetInfo(&available, &total);
	if (q_size == 0) {
		q_size = total / sizeof(inttype);
	}
	size_t el_per_Mb = Mb / sizeof(inttype);

	while(hipMalloc((void**)&d_q,  q_size * sizeof(inttype)) == hipErrorOutOfMemory)	{
		q_size -= el_per_Mb;
		if( q_size  < el_per_Mb) {
			// signal no free memory
			break;
		}
	}
	fprintf (stdout, "global mem queue size: %lu, number of entries: %d\n", q_size*sizeof(int), (int) q_size);

	if (opensize == 0) {
		opensize = q_size / 1000;
	}
	//fprintf (stdout, "global mem open set size: %lu, number of entries: %d\n", opensize*sizeof(int), (int) opensize);

	// calculate nr of buckets
	opensize = opensize / (WARPSIZE / nr_procs);

	inttype shared_q_size = (int) prop.sharedMemPerBlock / sizeof(inttype);
	//shared_q_size = 1000;
	fprintf (stdout, "shared mem queue size: %lu, number of entries: %u\n", shared_q_size*sizeof(inttype), shared_q_size);

	// copy symbols
	inttype nrbuckets = q_size / WARPSIZE;
	hipMemcpyToSymbol(HIP_SYMBOL(d_nrbuckets), &nrbuckets, sizeof(inttype));
	hipMemcpyToSymbol(HIP_SYMBOL(d_shared_q_size), &shared_q_size, sizeof(inttype));
	hipMemcpyToSymbol(HIP_SYMBOL(d_nr_procs), &nr_procs, sizeof(inttype));
	hipMemcpyToSymbol(HIP_SYMBOL(d_max_buf_ints), &max_buf_ints, sizeof(inttype));
	hipMemcpyToSymbol(HIP_SYMBOL(d_sv_nints), &sv_nints, sizeof(inttype));
	hipMemcpyToSymbol(HIP_SYMBOL(d_bits_act), &bits_act, sizeof(inttype));
	hipMemcpyToSymbol(HIP_SYMBOL(d_nbits_offset), &nbits_offset, sizeof(inttype));
	hipMemcpyToSymbol(HIP_SYMBOL(d_nbits_syncbits_offset), &nbits_syncbits_offset, sizeof(inttype));
	hipMemcpyToSymbol(HIP_SYMBOL(d_kernel_iters), &kernel_iters, sizeof(inttype));
	hipMemcpyToSymbol(HIP_SYMBOL(d_property), &check_property, sizeof(inttype));

	// determine actual nr of blocks
	nblocks = MAX(1,MIN(prop.maxGridSize[0],nblocks));

	// init the queue
	fprintf (stdout, "nr. of blocks: %d, block size: %d, nr of kernel iterations: %d\n", nblocks, nthreadsperblock, kernel_iters);
	init_queue<<<nblocks, nthreadsperblock>>>(d_q, q_size);
	//hipMemset(d_q, 0, q_size*sizeof(uint64_t));
	store_initial<<<1,1>>>(d_q, d_h);
	//print_queue(d_q, q_size, firstbit_statevector, nr_procs, sv_nints);
	for (int i = 0; i < 2*NR_HASH_FUNCTIONS; i++) {
		fprintf (stdout, "hash constant %d: %d\n", i, h[i]);
	}
	FIRSTHASHHOST(i);
	fprintf (stdout, "hash of initial state: %d\n", i);
	// set pointers
	//d_q_open = 0;
	//d_q_next = 1;
	//d_q_end = 1;

	// test: scan the entire queue
	//scan_queue<<<nblocks, nthreadsperblock>>>(d_q, q_size);

	// number of blocks required for iteration
	//nblocks = MAX(1,MIN(prop.maxGridSize[0],((d_q_next-d_q_open)/sv_nints) / (nthreadsperblock / nr_procs)));
	//nblocks = MAX(1,MIN(prop.maxGridSize[0], q_size/nthreadsperblock));

	inttype zero = 0;
	inttype *q_test = (inttype*) malloc(sizeof(inttype)*q_size);
	int j = 0;
	inttype scan = 0;
	CUDA_CHECK_RETURN(hipMemcpy(d_property_violation, &zero, sizeof(inttype), hipMemcpyHostToDevice))
	inttype property_violation = 0;
	while (contBFS == 1) {
		CUDA_CHECK_RETURN(hipMemcpy(d_contBFS, &zero, sizeof(inttype), hipMemcpyHostToDevice))
		gather<<<nblocks, nthreadsperblock, shared_q_size*sizeof(inttype)>>>(d_q, d_h, d_bits_state, d_firstbit_statevector, d_proc_offsets_start,
																		d_proc_offsets, d_proc_trans, d_syncbits_offsets, d_syncbits, d_contBFS, d_property_violation, scan);
		// copy progress result
		//CUDA_CHECK_RETURN(hipGetLastError());
		CUDA_CHECK_RETURN(hipDeviceSynchronize());
		CUDA_CHECK_RETURN(hipMemcpy(&contBFS, d_contBFS, sizeof(inttype), hipMemcpyDeviceToHost))
		if (check_property > 0) {
			CUDA_CHECK_RETURN(hipMemcpy(&property_violation, d_property_violation, sizeof(inttype), hipMemcpyDeviceToHost))
			if (property_violation == 1) {
				contBFS = 0;
			}
		}
		if (verbosity > 0) {
			if (verbosity == 1) {
				printf ("%d\n", j++);
			}
			else if (verbosity == 2) {
				hipMemcpy(q_test, d_q, q_size*sizeof(inttype), hipMemcpyDeviceToHost);
				count_local_queue(q_test, q_size, firstbit_statevector, nr_procs, sv_nints);
			}
			else if (verbosity == 3) {
				hipMemcpy(q_test, d_q, q_size*sizeof(inttype), hipMemcpyDeviceToHost);
				print_local_queue(q_test, q_size, firstbit_statevector, nr_procs, sv_nints);
			}
		}
		//j++;
		//if (j == 1) {
		scan = 1;
		//}
	}
	// determine runtime
	stop = clock();
	runtime = (double) (stop-start)/CLOCKS_PER_SEC;
	fprintf (stdout, "Run time: %f\n", runtime);

	if (property_violation == 1) {
		if (check_property == 1) {
			printf ("deadlock detected!\n");
		}
		else {
			printf ("safety property violation detected!\n");
		}
	}
	// report error if required
	//if (contBFS == 2) {
	//	fprintf (stderr, "ERROR: problem with hash table\n");
	//}
	//else {
		count_queue(d_q, q_size, firstbit_statevector, nr_procs, sv_nints);
	//}

	CUDA_CHECK_RETURN(hipDeviceSynchronize());	// Wait for the GPU launched work to complete
	//CUDA_CHECK_RETURN(hipGetLastError());

	return 0;
}
