#include "hip/hip_runtime.h"
/*
 ============================================================================
 Name        : GPUexplore.cu
 Author      : Anton Wijs
 Version     :
 Copyright   : Copyright Anton Wijs
 Description : CUDA GPUexplore: On the fly state space analysis
 ============================================================================
 */

#include <stdio.h>
#include <stdlib.h>
#include <stdint.h>
#include <string.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <assert.h>
#include <time.h>
#include <math.h>

// type of elements used
#define inttype uint32_t
// type of indices in hash table
#define indextype uint64_t

enum BucketEntryStatus { EMPTY, TAKEN, FOUND };
enum PropertyStatus { NONE, DEADLOCK, SAFETY, LIVENESS };

#define MIN(a,b) \
   ({ __typeof__ (a) _a = (a); \
       __typeof__ (b) _b = (b); \
     _a < _b ? _a : _b; })

#define MAX(a,b) \
   ({ __typeof__ (a) _a = (a); \
       __typeof__ (b) _b = (b); \
     _a > _b ? _a : _b; })

// Nr of tiles processed in single kernel launch
//#define TILEITERS 10

static const int WARPSIZE = 32;
static const int HALFWARPSIZE = 16;
static const int INTSIZE = 32;
static const int BUFFERSIZE = 50;

// GPU constants
__constant__ inttype d_nrbuckets;
__constant__ inttype d_shared_q_size;
__constant__ inttype d_nr_procs;
__constant__ inttype d_max_buf_ints;
__constant__ inttype d_sv_nints;
__constant__ inttype d_bits_act;
__constant__ inttype d_nbits_offset;
__constant__ inttype d_kernel_iters;
__constant__ inttype d_nbits_syncbits_offset;
__constant__ PropertyStatus d_property;

// GPU shared memory array
extern __shared__ inttype shared[];

// thread ids
#define WARP_ID							(threadIdx.x / WARPSIZE)
#define GLOBAL_WARP_ID					(((blockDim.x / WARPSIZE)*blockIdx.x)+WARP_ID)
#define NR_WARPS						((blockDim.x / WARPSIZE)*gridDim.x)
#define LANE							(threadIdx.x % WARPSIZE)
#define HALFLANE						(threadIdx.x % HALFWARPSIZE)
//#define ENTRY_ID						(LANE % d_sv_nints)
#define ENTRY_ID						(HALFLANE % d_sv_nints)
#define GROUP_ID						(threadIdx.x % d_nr_procs)
#define GROUP_GID						(threadIdx.x / d_nr_procs)

//#define NREL_IN_BUCKET					((WARPSIZE / d_sv_nints))
#define NREL_IN_BUCKET					((HALFWARPSIZE / d_sv_nints)*2)
#define NREL_IN_BUCKET_HOST				((HALFWARPSIZE / sv_nints)*2)

// constant for cuckoo hashing (Alcantara et al)
static const inttype P = 334214459;
// Retry constant to determine number of retries for element insertion
#define RETRYFREQ 7
#define NR_HASH_FUNCTIONS 8
// Number of retries in local cache
#define CACHERETRYFREQ 20
// Maximum size of state vectors (in nr. of 32-bit integers)
#define MAX_SIZE 9
// Empty state vectors
static const inttype EMPTYVECT32 = 0x7FFFFFFF;
// Constant to indicate that no more work is required
# define EXPLORATION_DONE 0x7FFFFFFF
// offset in shared memory from which loaded data can be read
static const int SH_OFFSET = 5;
//static const int KERNEL_ITERS = 10;
//static const int NR_OF_BLOCKS = 3120;
//static const int BLOCK_SIZE = 512;
static const int KERNEL_ITERS = 1;
static const int NR_OF_BLOCKS = 1;
static const int BLOCK_SIZE = 32;
const size_t Mb = 1<<20;

// test macros
#define PRINTTHREADID()						{printf("Hello thread %d\n", (blockIdx.x*blockDim.x)+threadIdx.x);}
#define PRINTTHREAD(j, i)					{printf("%d: Seen by thread %d: %d\n", (j), (blockIdx.x*blockDim.x)+threadIdx.x, (i));}

// Offsets calculations for shared memory arrays
#define HASHCONSTANTSLEN				(2*NR_HASH_FUNCTIONS)
#define VECTORPOSLEN					(d_nr_procs+1)
#define LTSSTATESIZELEN					(d_nr_procs)
#define OPENTILELEN						(d_sv_nints*(blockDim.x/d_nr_procs))
#define TGTSTATELEN						(blockDim.x*d_sv_nints)
#define THREADBUFFERLEN					((blockDim.x/d_nr_procs)*(THREADBUFFERSHARED+(d_nr_procs*d_max_buf_ints)))

#define HASHCONSTANTSOFFSET 			(SH_OFFSET)
#define VECTORPOSOFFSET 				(HASHCONSTANTSOFFSET+HASHCONSTANTSLEN)
#define LTSSTATESIZEOFFSET 				(VECTORPOSOFFSET+VECTORPOSLEN)
#define OPENTILEOFFSET 					(LTSSTATESIZEOFFSET+LTSSTATESIZELEN)
#define TGTSTATEOFFSET		 			(OPENTILEOFFSET+OPENTILELEN)
#define THREADBUFFEROFFSET	 			(TGTSTATEOFFSET+TGTSTATELEN)
#define CACHEOFFSET 					(THREADBUFFEROFFSET+THREADBUFFERLEN)

// One int for sync action counter
// One int for POR counter
#define THREADBUFFERSHARED				2
// parameter is thread id
#define THREADBUFFERGROUPSTART(i)		(THREADBUFFEROFFSET+(((i) / d_nr_procs)*(THREADBUFFERSHARED+(d_nr_procs*d_max_buf_ints))))
// parameter is group id
#define THREADBUFFERGROUPPOS(i, j)		shared[THREADBUFFERGROUPSTART(threadIdx.x)+THREADBUFFERSHARED+((i)*d_max_buf_ints)+(j)]
#define THREADGROUPCOUNTER				shared[(THREADBUFFERGROUPSTART(threadIdx.x))]
#define THREADGROUPPOR					shared[(THREADBUFFERGROUPSTART(threadIdx.x)) + 1]
#define OPENTILESTATEPART(i)			shared[OPENTILEOFFSET+(d_sv_nints*(threadIdx.x / d_nr_procs))+(i)]

#define THREADINGROUP					(threadIdx.x < (blockDim.x/d_nr_procs)*d_nr_procs)

#define STATESIZE(i)					(shared[LTSSTATESIZEOFFSET+(i)])
#define VECTORSTATEPOS(i)				(shared[VECTORPOSOFFSET+(i)])
#define NR_OF_STATES_IN_TRANSENTRY(i)	((31 - d_bits_act) / shared[LTSSTATESIZEOFFSET+(i)])
// SM local progress flags
#define ITERATIONS						(shared[0])
#define CONTINUE						(shared[1])
#define OPENTILECOUNT					(shared[2])
#define WORKSCANRESULT					(shared[3])
#define SCAN							(shared[4])

// BIT MANIPULATION MACROS

#define SETBIT(i, x)							{(x) = ((1L<<(i)) | (x));}
#define GETBIT(i, x)							(((x) >> (i)) & 1L)
#define SETBITS(i, j, x)						{(x) = (x) | (((1L<<(j))-1)^((1L<<(i))-1));}
#define GETPROCTRANSACT(a, t)					{bitmask = 0; SETBITS(1, 1+d_bits_act, bitmask); (a) = ((t) & bitmask) >> 1;}
#define GETPROCTRANSSYNC(a, t)					{(a) = ((t) & 1);}
#define GETPROCTRANSSTATE(a, t, i, j)			{bitmask = 0; SETBITS(1+d_bits_act+((i)-1)*shared[LTSSTATESIZEOFFSET+(j)], \
								1+d_bits_act+(i)*shared[LTSSTATESIZEOFFSET+(j)],bitmask); \
								(a) = ((t) & bitmask) >> 1+d_bits_act+(((i)-1)*shared[LTSSTATESIZEOFFSET+(j)]);}
#define GETTRANSOFFSET(a, t, i)					{bitmask = 0; SETBITS((i)*d_nbits_offset, ((i)+1)*d_nbits_offset, bitmask); (a) = ((t) & bitmask) >> ((i)*d_nbits_offset);}
#define GETSYNCOFFSET(a, t, i)					{bitmask = 0; SETBITS((i)*d_nbits_syncbits_offset, ((i)+1)*d_nbits_syncbits_offset, bitmask); \
													(a) = ((t) & bitmask) >> ((i)*d_nbits_syncbits_offset);}
#define GETSTATEVECTORSTATE(a, t, i)			{bitmask = 0; 	if (shared[VECTORPOSOFFSET+(i)]/INTSIZE == (shared[VECTORPOSOFFSET+(i)+1]-1)/INTSIZE) { \
																	SETBITS((shared[VECTORPOSOFFSET+(i)] % INTSIZE), \
																			(((shared[VECTORPOSOFFSET+(i)+1]-1) % INTSIZE)+1), bitmask); \
																	(a) = ((t)[shared[VECTORPOSOFFSET+(i)]/INTSIZE] & bitmask) >> (shared[VECTORPOSOFFSET+(i)] % INTSIZE); \
																} \
																else { \
																	SETBITS(0,(shared[VECTORPOSOFFSET+(i)+1] % INTSIZE),bitmask); \
																	(a) = (t)[shared[VECTORPOSOFFSET+(i)]/INTSIZE] >> (shared[VECTORPOSOFFSET+(i)] % INTSIZE) \
																		 | \
																		((t)[shared[VECTORPOSOFFSET+(i)+1]/INTSIZE] & bitmask) << \
																		(INTSIZE - (shared[VECTORPOSOFFSET+(i)] % INTSIZE)); \
																} \
												}
#define SETPROCTRANSACT(t, x)					{bitmask = 0; SETBITS(1, d_bits_act+1,bitmask); (t) = ((t) & ~bitmask) | ((x) << 1);}
#define SETPROCTRANSSYNC(t, x)					{(t) = ((t) & ~1) | (x);}
#define SETPROCTRANSSTATE(t, i, x, j)			{bitmask = 0; SETBITS(1+d_bits_act+((i)-1)*shared[LTSSTATESIZEOFFSET+(j)],1+d_bits_act+(i)*shared[LTSSTATESIZEOFFSET+(j)],bitmask); \
													(t) = ((t) & ~bitmask) | ((x) << (1+d_bits_act+((i)-1)*shared[LTSSTATESIZEOFFSET+(j)]));}
#define SETSTATEVECTORSTATE(t, i, x)			{bitmask = 0; 	if (shared[VECTORPOSOFFSET+(i)]/INTSIZE == (shared[VECTORPOSOFFSET+(i)+1]-1)/INTSIZE) { \
																	SETBITS((shared[VECTORPOSOFFSET+(i)] % INTSIZE), \
																			(((shared[VECTORPOSOFFSET+(i)+1]-1) % INTSIZE)+1),bitmask); \
																	(t)[shared[VECTORPOSOFFSET+(i)]/INTSIZE] = ((t)[shared[VECTORPOSOFFSET+(i)]/INTSIZE] & ~bitmask) | \
																	((x) << (shared[VECTORPOSOFFSET+(i)] % INTSIZE)); \
																} \
																else { \
																	SETBITS(0,(shared[VECTORPOSOFFSET+(i)] % INTSIZE), bitmask); \
																	(t)[shared[VECTORPOSOFFSET+(i)]/INTSIZE] = ((t)[shared[VECTORPOSOFFSET+(i)]/INTSIZE] & bitmask) | \
																	((x) << (shared[VECTORPOSOFFSET+(i)] % INTSIZE)); \
																	bitmask = 0; \
																	SETBITS((shared[VECTORPOSOFFSET+(i)+1] % INTSIZE), INTSIZE, bitmask); \
																	(t)[shared[VECTORPOSOFFSET+(i)+1]/INTSIZE] = ((t)[shared[VECTORPOSOFFSET+(i)+1]/INTSIZE] & bitmask) | \
																		((x) >> (INTSIZE - (shared[VECTORPOSOFFSET+(i)] % INTSIZE))); \
																} \
												}
// NEEDS FIX: USE BIT 32 OF FIRST INTEGER TO INDICATE STATE OR NOT (1 or 0), IN CASE MULTIPLE INTEGERS ARE USED FOR STATE VECTOR!!!
//#define ISSTATE(t)								((t)[(d_sv_nints-1)] != EMPTYVECT32)
#define ISSTATE(t)								((t)[0] != EMPTYVECT32)
#define SETNEWSTATE(t)							{	(t)[(d_sv_nints-1)] = (t)[(d_sv_nints-1)] | 0x80000000;}
#define SETOLDSTATE(t)							{	(t)[(d_sv_nints-1)] = (t)[(d_sv_nints-1)] & 0x7FFFFFFF;}
#define ISNEWSTATE(t)							((t)[(d_sv_nints-1)] >> 31)
#define ISNEWSTATE_HOST(t)						((t)[(sv_nints-1)] >> 31)
#define ISNEWINT(t)								((t) >> 31)
#define OLDINT(t)								((t) & 0x7FFFFFFF)
#define NEWINT(t)								((t) | 0x80000000)
#define STRIPSTATE(t)							{(t)[(d_sv_nints-1)] = (t)[(d_sv_nints-1)] & 0x7FFFFFFF;}
#define STRIPPEDSTATE(t, i)						((i == d_sv_nints-1) ? ((t)[i] & 0x7FFFFFFF) : (t)[i])
#define STRIPPEDENTRY(t, i)						((i == d_sv_nints-1) ? ((t) & 0x7FFFFFFF) : (t))
#define STRIPPEDENTRY_HOST(t, i)				((i == sv_nints-1) ? ((t) & 0x7FFFFFFF) : (t))
#define NEWSTATEPART(t, i)						(((i) == d_sv_nints-1) ? ((t)[d_sv_nints-1] | 0x80000000) : (t)[(i)])
#define COMPAREENTRIES(t1, t2)					(((t1) & 0x7FFFFFFF) == ((t2) & 0x7FFFFFFF))
#define OWNSSYNCRULE(a, t, i)					{if (GETBIT((i),(t))) { \
													bitmask = 0; SETBITS(0,(i),bitmask); if ((t & bitmask) > 0) {(a) = 0;} else {(a) = 1;}} \
													else {(a) = 0;}}
#define GETSYNCRULE(a, t, i)					{bitmask = 0; SETBITS((i)*d_nr_procs,((i)+1)*d_nr_procs,bitmask); (a) = ((t) & bitmask) >> ((i)*d_nr_procs);}
#define SYNCRULEISAPPLICABLE(a, t, ac)			{(a) = 1; for (bk = 0; bk < d_nr_procs; bk++) { \
													if (GETBIT(bk,(t))) { \
														bj = THREADBUFFERGROUPPOS((inttype) bk,0); \
														if (bj == 0) { \
															(a) = 0; \
														} \
														else { \
															GETPROCTRANSACT(k, bj); \
															if (k != (ac)) { \
																(a) = 0; \
															} \
														}\
													} \
												} \
												}

// HASH TABLE MACROS

// Return 0 if not found, 1 if found, 2 if cache is full
__device__ inttype STOREINCACHE(inttype* t, inttype* d_q, inttype* address) {
	inttype bi, bj, bk, bl, bitmask;
	indextype hashtmp;
	STRIPSTATE(t);
	hashtmp = 0;
	for (bi = 0; bi < d_sv_nints; bi++) {
		hashtmp += t[bi];
	}
	bitmask = d_sv_nints*((inttype) (hashtmp % ((d_shared_q_size - CACHEOFFSET) / d_sv_nints)));
	SETNEWSTATE(t);
	bl = 0;
	while (bl < CACHERETRYFREQ) {
		bi = atomicCAS((inttype *) &shared[CACHEOFFSET+bitmask+(d_sv_nints-1)], EMPTYVECT32, t[d_sv_nints-1]);
		if (bi == EMPTYVECT32) {
			for (bj = 0; bj < d_sv_nints-1; bj++) {
				shared[CACHEOFFSET+bitmask+bj] = t[bj];
			}
			*address = bitmask;
			return 0;
		}
		if (COMPAREENTRIES(bi, t[d_sv_nints-1])) {
			if (d_sv_nints == 1) {
				*address = bitmask;
				return 1;
			}
			else {
				for (bj = 0; bj < d_sv_nints-1; bj++) {
					if (shared[CACHEOFFSET+bitmask+bj] != (t)[bj]) {
						break;
					}
				}
				if (bj == d_sv_nints-1) {
					*address = bitmask;
					return 1;
				}
			}
		}
		if (!ISNEWINT(bi)) {
			bj = atomicCAS((inttype *) &shared[CACHEOFFSET+bitmask+(d_sv_nints-1)], bi, t[d_sv_nints-1]);
			if (bi == bj) {
				for (bk = 0; bk < d_sv_nints-1; bk++) {
					shared[CACHEOFFSET+bitmask+bk] = t[bk];
				}
				*address = bitmask;
				return 0;
			}
		}
		bl++;
		bitmask += d_sv_nints;
		if ((bitmask+(d_sv_nints-1)) >= (d_shared_q_size - CACHEOFFSET)) {
			bitmask = 0;
		}
	}
	return 2;
}

// hash functions use bj variable
#define FIRSTHASH(a, t)							{	hashtmp = 0; \
													for (bj = 0; bj < d_sv_nints; bj++) { \
														hashtmp += (indextype) (d_h[0]*(STRIPPEDSTATE(t, bj))+d_h[1]); \
													} \
													(a) = WARPSIZE*((inttype) ((hashtmp % P) % d_nrbuckets)); \
												}
#define FIRSTHASHHOST(a)						{	indextype hashtmp = 0; \
													hashtmp = (indextype) h[1]; \
													(a) = WARPSIZE*((inttype) ((hashtmp % P) % q_size/WARPSIZE)); \
												}
#define HASHALL(a, i, t)						{	hashtmp = 0; \
													for (bj = 0; bj < d_sv_nints; bj++) { \
														hashtmp += (indextype) (shared[HASHCONSTANTSOFFSET+(2*(i))]*(STRIPPEDSTATE(t, bj))+shared[HASHCONSTANTSOFFSET+(2*(i))+1]); \
													} \
													(a) = WARPSIZE*((inttype) ((hashtmp % P) % d_nrbuckets)); \
												}
#define HASHFUNCTION(a, i, t)					((HASHALL((a), (i), (t))))

#define COMPAREVECTORS(a, t1, t2)				{	(a) = 1; \
													for (bk = 0; bk < d_sv_nints-1; bk++) { \
														if ((t1)[bk] != (t2)[bk]) { \
															(a) = 0; break; \
														} \
													} \
													if ((a)) { \
														if (STRIPPEDSTATE((t1),bk) != STRIPPEDSTATE((t2),bk)) { \
															(a) = 0; \
														} \
													} \
												}

// check if bucket element associated with lane is a valid position to store data
#define LANEPOINTSTOVALIDBUCKETPOS						(HALFLANE < ((HALFWARPSIZE / d_sv_nints)*d_sv_nints))
//#define LANEPOINTSTOVALIDBUCKETPOS						true

__device__ inttype LANE_POINTS_TO_EL(inttype i)	{
	if (i < HALFWARPSIZE / d_sv_nints) {
		return (LANE >= i*d_sv_nints && LANE < (i+1)*d_sv_nints);
	}
	else {
		return (LANE >= HALFWARPSIZE+(i-(HALFWARPSIZE / d_sv_nints))*d_sv_nints && LANE < HALFWARPSIZE+(i-(HALFWARPSIZE / d_sv_nints)+1)*d_sv_nints);
	}
}

//__device__ inttype LANE_POINTS_TO_EL(inttype i)	{
//	return (LANE >= i*d_sv_nints && LANE < (i+1)*d_sv_nints);
//}

// start position of element i in bucket
#define STARTPOS_OF_EL_IN_BUCKET(i)			((i < (HALFWARPSIZE / d_sv_nints)) ? (i*d_sv_nints) : (HALFWARPSIZE + (i-(HALFWARPSIZE/d_sv_nints))*d_sv_nints))
//#define STARTPOS_OF_EL_IN_BUCKET(i)			(i*d_sv_nints)
#define STARTPOS_OF_EL_IN_BUCKET_HOST(i)	((i < (HALFWARPSIZE / sv_nints)) ? (i*sv_nints) : (HALFWARPSIZE + (i-(HALFWARPSIZE/sv_nints))*sv_nints))
//#define STARTPOS_OF_EL_IN_BUCKET_HOST(i)	(i*sv_nints)

// find or put element, single thread version.
__device__ inttype FINDORPUT_SINGLE(inttype* t, inttype* d_q, volatile inttype* d_newstate_flags) {
	inttype bi, bj, bk, bl;
	indextype hashtmp;
	for (bi = 0; bi < NR_HASH_FUNCTIONS; bi++) {
		HASHFUNCTION(hashtmp, bi, t);
		for (bj = 0; bj < NREL_IN_BUCKET; bj++) {
			bl = d_q[hashtmp+STARTPOS_OF_EL_IN_BUCKET(bj)+(d_sv_nints-1)];
			if (bl == EMPTYVECT32) {
				bl = atomicCAS(&d_q[hashtmp+STARTPOS_OF_EL_IN_BUCKET(bj)+(d_sv_nints-1)], EMPTYVECT32, t[d_sv_nints-1]);
				if (bl == EMPTYVECT32) {
					// Write was successful
					if (d_sv_nints > 1) {
						for (bk = 0; bk < d_sv_nints-1; bk++) {
							d_q[hashtmp+STARTPOS_OF_EL_IN_BUCKET(bj)+bk] = t[bk];
						}
					}
					threadfence();
					// There is work available for some block
					d_newstate_flags[(hashtmp / blockDim.x) % gridDim.x] = 1;
				}
			}
			if (bl != EMPTYVECT32) {
				COMPAREVECTORS(bk, &d_q[hashtmp+STARTPOS_OF_EL_IN_BUCKET(bj)], t); \
				if (bk == 1) {
					// Found state in global memory
					return 1;
				}
			}
			else {
				SETOLDSTATE(t);
				return 1;
			}
		}
	}
	return 0;
}

// find or put element, warp version. t is element stored in block cache
__device__ inttype FINDORPUT_WARP(inttype* t, inttype* d_q, volatile inttype* d_newstate_flags)	{
	inttype bi, bj, bk, bl, bitmask;
	indextype hashtmp;
	BucketEntryStatus threadstatus;
	// prepare bitmask once to reason about results of threads in the same (state vector) group
	bitmask = 0;
	if (LANEPOINTSTOVALIDBUCKETPOS) {
		SETBITS(LANE-ENTRY_ID, LANE-ENTRY_ID+d_sv_nints, bitmask);
	}
	for (bi = 0; bi < NR_HASH_FUNCTIONS; bi++) {
		HASHFUNCTION(hashtmp, bi, t);
		bl = d_q[hashtmp+LANE];
		bk = __ballot(STRIPPEDENTRY(bl, ENTRY_ID) == STRIPPEDSTATE(t, ENTRY_ID));
		// threadstatus is used to determine whether full state vector has been found
		threadstatus = EMPTY;
		if (LANEPOINTSTOVALIDBUCKETPOS) {
			if ((bk & bitmask) == bitmask) {
				threadstatus = FOUND;
			}
		}
		if (__ballot(threadstatus == FOUND) != 0) {
			// state vector has been found in bucket. mark local copy as old.
			if (LANE == 0) {
				SETOLDSTATE(t);
			}
			return 1;
		}
		// try to find empty position to insert new state vector
		threadstatus = (bl == EMPTYVECT32 && LANEPOINTSTOVALIDBUCKETPOS) ? EMPTY : TAKEN;
		// let bk hold the smallest index of an available empty position
		bk = __ffs(__ballot(threadstatus == EMPTY));
		while (bk != 0) {
			// write the state vector
			bk--;
			if (LANE >= bk && LANE < bk+d_sv_nints) {
				bl = atomicCAS(&(d_q[hashtmp+LANE]), EMPTYVECT32, t[ENTRY_ID]);
				if (bl == EMPTYVECT32) {
					// success
					if (ENTRY_ID == d_sv_nints-1) {
						SETOLDSTATE(t);
					}
					// try to claim the state vector for future work
					bl = OPENTILELEN;
					if (ENTRY_ID == d_sv_nints-1) {
						// try to increment the OPENTILECOUNT counter
						bl = atomicAdd((inttype *) &OPENTILECOUNT, d_sv_nints);
						if (bl < OPENTILELEN) {
							d_q[hashtmp+LANE] = t[d_sv_nints-1];
						} else {
							// There is work available for some block
							__threadfence();
							d_newstate_flags[(hashtmp / blockDim.x) % gridDim.x] = 1;
						}
					}
					// all active threads read the OPENTILECOUNT value of the first thread, and possibly store their part of the vector in the shared memory
					bl = __shfl(bl, LANE-ENTRY_ID+d_sv_nints-1);
					if (bl < OPENTILELEN) {
						// write part of vector to shared memory
						shared[OPENTILEOFFSET+bl+ENTRY_ID] = NEWSTATEPART(t, ENTRY_ID);
					}
					// write was successful. propagate this to the whole warp by setting threadstatus to FOUND
					threadstatus = FOUND;
				}
				else {
					// write was not successful. check if the state vector now in place equals the one we are trying to insert
					bk = __ballot(STRIPPEDENTRY(bl, ENTRY_ID) == STRIPPEDSTATE(t, ENTRY_ID));
					if ((bk & bitmask) == bitmask) {
						// state vector has been found in bucket. mark local copy as old.
						if (LANE == bk) {
							SETOLDSTATE(t);
						}
						// propagate this result to the whole warp
						threadstatus = FOUND;
					}
					else {
						// state vector is different, and position in bucket is taken
						threadstatus = TAKEN;
					}
				}
			}
			// check if the state vector was either encountered or inserted
			if (__ballot(threadstatus == FOUND) != 0) {
				return 1;
			}
			// recompute bk
			bk = __ffs(__ballot(threadstatus == EMPTY));
		}
	}
	return 0;
}

// find element, warp version. t is element stored in block cache
__device__ inttype FIND_WARP(inttype* t, inttype* d_q)	{
	inttype bi, bj, bk, bl, bitmask;
	indextype hashtmp;
	BucketEntryStatus threadstatus;
	// prepare bitmask once to reason about results of threads in the same (state vector) group
	bitmask = 0;
	if (LANEPOINTSTOVALIDBUCKETPOS) {
		SETBITS(LANE-ENTRY_ID, LANE-ENTRY_ID+d_sv_nints, bitmask);
	}
	for (bi = 0; bi < NR_HASH_FUNCTIONS; bi++) {
		HASHFUNCTION(hashtmp, bi, t);
		bl = d_q[hashtmp+LANE];
		bk = __ballot(STRIPPEDENTRY(bl, ENTRY_ID) == STRIPPEDSTATE(t, ENTRY_ID));
		// threadstatus is used to determine whether full state vector has been found
		threadstatus = EMPTY;
		if (LANEPOINTSTOVALIDBUCKETPOS) {
			if ((bk & bitmask) == bitmask) {
				threadstatus = FOUND;
			}
		}
		if (__ballot(threadstatus == FOUND) != 0) {
			// state vector has been found in bucket. mark local copy as old.
			if (threadstatus == FOUND & ISNEWINT(bl) == 0 & ENTRY_ID == d_sv_nints - 1) {
				SETOLDSTATE(t);
			}
			return __ballot(threadstatus == FOUND & ISNEWINT(bl) == 0 & ENTRY_ID == d_sv_nints - 1);
		}
		// try to find empty position
		threadstatus = (bl == EMPTYVECT32 && LANEPOINTSTOVALIDBUCKETPOS) ? EMPTY : TAKEN;
		// let bk hold the smallest index of an available empty position
		bk = __ffs(__ballot(threadstatus == EMPTY));
		if(bk < 32) {
			// There is an empty slot in this bucket and the state vector was not found
			// State will also not be found after rehashing, so we return 0
			return 0;
		}
	}
	return 0;
}

__device__ inttype FINDORPUT_WARP_ORIG(inttype* t, inttype* d_q, inttype bi, inttype bj, inttype bk, inttype bl, inttype bitmask, indextype hashtmp) {
	for (bi = 0; bi < NR_HASH_FUNCTIONS; bi++) {
		HASHFUNCTION(hashtmp, bi, t);
		bl = d_q[hashtmp+LANE];
		if (ENTRY_ID == (d_sv_nints-1)) {
			if (bl != EMPTYVECT32) {
				COMPAREVECTORS(bl, &d_q[hashtmp+LANE-(d_sv_nints-1)], (t));
				if (bl) {
					SETOLDSTATE((t));
				}
			}
		}
		if (ISNEWSTATE(t)) {
			for (bj = 0; bj < NREL_IN_BUCKET; bj++) {
				if (d_q[hashtmp+STARTPOS_OF_EL_IN_BUCKET(bj)+(d_sv_nints-1)] == EMPTYVECT32) {
					if (LANE == 0) {
						bl = atomicCAS(&d_q[hashtmp+STARTPOS_OF_EL_IN_BUCKET(bj)+(d_sv_nints-1)], EMPTYVECT32, t[d_sv_nints-1]);
						if (bl == EMPTYVECT32) {
							SETOLDSTATE(t);
							shared[THREADBUFFEROFFSET+WARP_ID] = OPENTILELEN;
							if (ITERATIONS < d_kernel_iters-1) {
								bk = atomicAdd((inttype *) &OPENTILECOUNT, d_sv_nints);
								if (bk < OPENTILELEN) {
									shared[THREADBUFFEROFFSET+WARP_ID] = bk;
									d_q[hashtmp+STARTPOS_OF_EL_IN_BUCKET(bj)+(d_sv_nints-1)] = t[d_sv_nints-1];
								}
							}
						}
					}
					if (!ISNEWSTATE(t)) {
						if (LANE < d_sv_nints - 1) {
							d_q[hashtmp+STARTPOS_OF_EL_IN_BUCKET(bj)+LANE] = t[LANE];
						}
						bk = shared[THREADBUFFEROFFSET+WARP_ID];
						if (bk != OPENTILELEN) {
							if (LANE < d_sv_nints) {
								shared[OPENTILEOFFSET+bk+LANE] = NEWSTATEPART(t, LANE);
							}
							if (LANE == 0) {
								shared[THREADBUFFEROFFSET+WARP_ID] = 0;
							}
						}
					}
				}
				if (!ISNEWSTATE((t))) {
					return 1;
				}
			}
		}
		if (!ISNEWSTATE((t))) {
			return 1;
		}
	}
	return 0;
}

// macro to print state vector
#define PRINTVECTOR(s) 							{	printf ("("); \
													for (bk = 0; bk < d_nr_procs; bk++) { \
														GETSTATEVECTORSTATE(bj, (s), bk) \
														printf ("%d", bj); \
														if (bk < (d_nr_procs-1)) { \
															printf (","); \
														} \
													} \
													printf (")\n"); \
												}


//#define INCRSTATEVECTOR(t)						(sv_nints == 1 ? t[0]++ : (t[0] == EMPTYVECTOR ? t[1]++ : t[0]++))
//#define DECRSTATEVECTOR(t)						(sv_nints == 1 ? t[0]-- : (t[0] == 0 ? (t[1]--; t[0] = EMPTYVECTOR) : t[0]--))

int vmem = 0;

// GPU textures
texture<inttype, 1, hipReadModeElementType> tex_proc_offsets_start;
texture<inttype, 1, hipReadModeElementType> tex_proc_offsets;
texture<inttype, 1, hipReadModeElementType> tex_proc_trans_start;
texture<inttype, 1, hipReadModeElementType> tex_proc_trans;
texture<inttype, 1, hipReadModeElementType> tex_syncbits_offsets;
texture<inttype, 1, hipReadModeElementType> tex_syncbits;

/**
 * This macro checks return value of the CUDA runtime call and exits
 * the application if the call failed.
 */
#define CUDA_CHECK_RETURN(value) {											\
	hipError_t _m_cudaStat = value;										\
	if (_m_cudaStat != hipSuccess) {										\
		fprintf(stderr, "Error %s at line %d in file %s\n",					\
				hipGetErrorString(_m_cudaStat), __LINE__, __FILE__);		\
		exit(1);															\
	} }

//wrapper around hipMalloc to count allocated memory and check for error while allocating
int cudaMallocCount ( void ** ptr,int size) {
	hipError_t err = hipSuccess;
	vmem += size;
	err = hipMalloc(ptr,size);
	if (err) {
		printf("Error %s at line %d in file %s\n", hipGetErrorString(err), __LINE__, __FILE__);
		exit(1);
	}
	fprintf (stdout, "allocated %d\n", size);
	return size;
}

//test function to print a given state vector
void print_statevector(FILE* stream, inttype *state, inttype *firstbit_statevector, inttype nr_procs, inttype sv_nints) {
	inttype i, s, bitmask;

	for (i = 0; i < nr_procs; i++) {
		bitmask = 0;
		if (firstbit_statevector[i]/INTSIZE == firstbit_statevector[i+1]/INTSIZE) {
			SETBITS(firstbit_statevector[i] % INTSIZE,firstbit_statevector[i+1] % INTSIZE, bitmask);
			s = (state[firstbit_statevector[i]/INTSIZE] & bitmask) >> (firstbit_statevector[i] % INTSIZE);
		}
		else {
			SETBITS(0, firstbit_statevector[i+1] % INTSIZE, bitmask);
			s = (state[firstbit_statevector[i]/INTSIZE] >> (firstbit_statevector[i] % INTSIZE)
					| (state[firstbit_statevector[i+1]/INTSIZE] & bitmask) << (INTSIZE - (firstbit_statevector[i] % INTSIZE))); \
		}
		fprintf (stream, "%d", s);
		if (i < (nr_procs-1)) {
			fprintf (stream, ",");
		}
	}
	fprintf (stream, " ");
	for (i = 0; i < sv_nints; i++) {
		fprintf (stream, "%d ", STRIPPEDENTRY_HOST(state[i], i));
	}
	fprintf (stream, "\n");
}

//test function to print the contents of the device queue
void print_queue(inttype *d_q, inttype q_size, inttype *firstbit_statevector, inttype nr_procs, inttype sv_nints) {
	inttype *q_test = (inttype*) malloc(sizeof(inttype)*q_size);
	hipMemcpy(q_test, d_q, q_size*sizeof(inttype), hipMemcpyDeviceToHost);
	inttype nw;
	int count = 0;
	int newcount = 0;
	for (inttype i = 0; i < (q_size/WARPSIZE); i++) {
		for (inttype j = 0; j < NREL_IN_BUCKET_HOST; j++) {
			if (q_test[(i*WARPSIZE)+STARTPOS_OF_EL_IN_BUCKET_HOST(j)+(sv_nints-1)] != EMPTYVECT32) {
				count++;
				nw = ISNEWSTATE_HOST(&q_test[(i*WARPSIZE)+STARTPOS_OF_EL_IN_BUCKET_HOST(j)]);
				if (nw) {
					newcount++;
					fprintf (stdout, "new: ");
				}
				print_statevector(stdout, &(q_test[(i*WARPSIZE)+STARTPOS_OF_EL_IN_BUCKET_HOST(j)]), firstbit_statevector, nr_procs, sv_nints);
			}
		}
	}
	fprintf (stdout, "nr. of states in hash table: %d (%d unexplored states)\n", count, newcount);
}

//test function to print the contents of the device queue
void print_local_queue(FILE* stream, inttype *q, inttype q_size, inttype *firstbit_statevector, inttype nr_procs, inttype sv_nints) {
	int count = 0, newcount = 0;
	inttype nw;
	for (inttype i = 0; i < (q_size/WARPSIZE); i++) {
		for (inttype j = 0; j < NREL_IN_BUCKET_HOST; j++) {
			if (q[(i*WARPSIZE)+STARTPOS_OF_EL_IN_BUCKET_HOST(j)+(sv_nints-1)] != EMPTYVECT32) {
				count++;

//				if (j == 0) {
//					fprintf (stdout, "-----------\n");
//				}
				nw = ISNEWSTATE_HOST(&q[(i*WARPSIZE)+STARTPOS_OF_EL_IN_BUCKET_HOST(j)]);
				if (nw) {
					newcount++;
					fprintf (stream, "new: ");
					//print_statevector(&(q[(i*WARPSIZE)+(j*sv_nints)]), firstbit_statevector, nr_procs);
				}
				print_statevector(stream, &(q[(i*WARPSIZE)+STARTPOS_OF_EL_IN_BUCKET_HOST(j)]), firstbit_statevector, nr_procs, sv_nints);
			}
		}
	}
	fprintf (stream, "nr. of states in hash table: %d (%d unexplored states)\n", count, newcount);
}

//test function to count the contents of the device queue
void count_queue(inttype *d_q, inttype q_size, inttype *firstbit_statevector, inttype nr_procs, inttype sv_nints) {
	inttype *q_test = (inttype*) malloc(sizeof(inttype)*q_size);
	hipMemcpy(q_test, d_q, q_size*sizeof(inttype), hipMemcpyDeviceToHost);

	int count = 0;
	for (inttype i = 0; i < (q_size/WARPSIZE); i++) {
		for (inttype j = 0; j < NREL_IN_BUCKET_HOST; j++) {
			if (q_test[(i*WARPSIZE)+STARTPOS_OF_EL_IN_BUCKET_HOST(j)+(sv_nints-1)] != EMPTYVECT32) {
				count++;
			}
		}
	}
	fprintf (stdout, "nr. of states in hash table: %d\n", count);
}

//test function to count the contents of the host queue
void count_local_queue(inttype *q, inttype q_size, inttype *firstbit_statevector, inttype nr_procs, inttype sv_nints) {
	int count = 0, newcount = 0;
	inttype nw;
	inttype nrbuckets = q_size / WARPSIZE;
	inttype nrels = NREL_IN_BUCKET_HOST;
	for (inttype i = 0; i < nrbuckets; i++) {
		for (inttype j = 0; j < nrels; j++) {
			inttype elpos = STARTPOS_OF_EL_IN_BUCKET_HOST(j);
			inttype abselpos = (i*WARPSIZE)+elpos+sv_nints-1;
			inttype q_abselpos = q[abselpos];
			if (q_abselpos != EMPTYVECT32) {
				count++;
				nw = ISNEWSTATE_HOST(&q[(i*WARPSIZE)+elpos]);
				if (nw) {
					newcount++;
				}
			}
		}
	}
	fprintf (stdout, "nr. of states in hash table: %d (%d unexplored states)\n", count, newcount);
}

/**
 * CUDA kernel function to initialise the queue
 */
__global__ void init_queue(inttype *d_q, inttype n_elem) {
    inttype nthreads = blockDim.x*gridDim.x;
    inttype i = (blockIdx.x *blockDim.x) + threadIdx.x;

    for(; i < n_elem; i += nthreads) {
    	d_q[i] = (inttype) EMPTYVECT32;
    }
}

/**
 * CUDA kernel to store initial state in hash table
 */
__global__ void store_initial(inttype *d_q, inttype *d_h, inttype *d_newstate_flags, inttype blockdim, inttype griddim) {
	inttype bj;
	indextype hashtmp;
	inttype state[MAX_SIZE];

	for (bj = 0; bj < d_sv_nints; bj++) {
		state[bj] = 0;
	}
	SETNEWSTATE(state);
	FIRSTHASH(hashtmp, state);
	for (bj = 0; bj < d_sv_nints; bj++) {
		d_q[hashtmp+bj] = state[bj];
	}
	d_newstate_flags[(hashtmp / blockdim) % griddim] = 1;
}

/**
 * CUDA kernel function for BFS iteration state gathering
 * Order of data in the shared queue:
 * (0. index of process LTS states sizes)
 * (1. index of sync rules offsets)
 * (2. index of sync rules)
 * (1. index of open queue tile)
 * 0. the 'iterations' flag to count the number of iterations so far (nr of tiles processed by SM)
 * 1. the 'continue' flag for thread work
 * (4. index of threads buffer)
 * (5. index of hash table)
 * 2. constants for d_q hash functions (2 per function, in total 8 by default)
 * 3. state vector offsets (nr_procs+1 elements)
 * 4. sizes of states in process LTS states (nr_procs elements)
 * (9. sync rules + offsets (nr_syncbits_offsets + nr_syncbits elements))
 * 5. tile of open queue to be processed by block (sv_nints*(blockDim.x / nr_procs) elements)
 * 6. buffer for threads ((blockDim.x*max_buf_ints)+(blockDim.x/nr_procs) elements)
 * 7. hash table
 */
__global__ void
__launch_bounds__(512, 2)
gather(inttype *d_q, inttype *d_h, inttype *d_bits_state,
						inttype *d_firstbit_statevector, inttype *d_proc_offsets_start,
						inttype *d_proc_offsets, inttype *d_proc_trans, inttype *d_syncbits_offsets,
						inttype *d_syncbits, inttype *d_contBFS, inttype *d_property_violation,
						volatile inttype *d_newstate_flags, inttype scan) {
	//inttype global_id = (blockIdx.x * blockDim.x) + threadIdx.x;
	//inttype group_nr = threadIdx.x / nr_procs;
	inttype i, k, l, index, offset1, offset2, tmp, cont, act, sync_offset1, sync_offset2;
	int32_t local_action_counter;
	inttype* src_state = &shared[OPENTILEOFFSET+(threadIdx.x/d_nr_procs)*d_sv_nints];
	inttype* tgt_state = &shared[TGTSTATEOFFSET+threadIdx.x*d_sv_nints];
	inttype bitmask, bi, bj, bk;
	int pos;
	// TODO: remove this
	inttype TMPVAR;
	// is at least one outgoing transition enabled for a given state (needed to detect deadlocks)
	inttype outtrans_enabled;

	for (i = threadIdx.x; i < d_shared_q_size; i += blockDim.x) {
		shared[i] = 0;
	}
	// Locally store the state sizes and syncbits
	i = threadIdx.x;
	if (i == 0) {
		ITERATIONS = 0;
		OPENTILECOUNT = 0;
		WORKSCANRESULT = 0;
		SCAN = 0;
	}
	if ((blockIdx.x*blockDim.x)+threadIdx.x == 0) {
		(*d_contBFS) = 0;
	}
	for (i = threadIdx.x; i < HASHCONSTANTSLEN; i += blockDim.x) {
		shared[i+HASHCONSTANTSOFFSET] = d_h[i];
	}
	for (i = threadIdx.x; i < VECTORPOSLEN; i += blockDim.x) {
		shared[i+VECTORPOSOFFSET] = d_firstbit_statevector[i];
	}
	for (i = threadIdx.x; i < LTSSTATESIZELEN; i += blockDim.x) {
		shared[i+LTSSTATESIZEOFFSET] = d_bits_state[i];
	}
	// Reset the open queue tile
	if (threadIdx.x < d_sv_nints*(blockDim.x / d_nr_procs)) {
		shared[OPENTILEOFFSET+threadIdx.x] = EMPTYVECT32;
	}
	// Clean the cache
	i = threadIdx.x;
	while (i < (d_shared_q_size - CACHEOFFSET)) {
		shared[CACHEOFFSET + i] = EMPTYVECT32;
		i += blockDim.x;
	}
	__syncthreads();
	if(scan) {
		//Copy the work tile from global mem
		if (threadIdx.x < OPENTILELEN) {
			shared[OPENTILEOFFSET+threadIdx.x] = d_q[d_nrbuckets*WARPSIZE + (OPENTILELEN+1) * blockIdx.x + threadIdx.x];
		}
		if(threadIdx.x == 0) {
			OPENTILECOUNT = d_q[d_nrbuckets*WARPSIZE + (OPENTILELEN+1) * blockIdx.x + OPENTILELEN];
		}
	}
	__syncthreads();
	inttype last_search_location = 0;
	while (ITERATIONS < d_kernel_iters) {
		if (threadIdx.x == 0 && OPENTILECOUNT < OPENTILELEN && d_newstate_flags[blockIdx.x]) {
			d_newstate_flags[blockIdx.x] = 2;
			SCAN = 1;
		}
		__syncthreads();
		// Scan the open set for work; we use the OPENTILECOUNT flag at this stage to count retrieved elements
		if (SCAN) {
			// This block should be able to find a new state
			int found_new_state = 0;
			for (i = GLOBAL_WARP_ID; i < d_nrbuckets && OPENTILECOUNT < OPENTILELEN; i += NR_WARPS) {
				int loc = i + last_search_location;
				if(loc >= d_nrbuckets) {
					last_search_location = -i + GLOBAL_WARP_ID;
					loc = i + last_search_location;
				}
				tmp = d_q[loc*WARPSIZE+LANE];
				l = EMPTYVECT32;
				if (ENTRY_ID == (d_sv_nints-1)) {
					if (ISNEWINT(tmp)) {
						found_new_state = 1;
						// try to increment the OPENTILECOUNT counter, if successful, store the state
						l = atomicAdd((uint32_t *) &OPENTILECOUNT, d_sv_nints);
						if (l < OPENTILELEN) {
							d_q[loc*WARPSIZE+LANE] = OLDINT(tmp);
						}
					}
				}
				// all threads read the OPENTILECOUNT value of the 'tail' thread, and possibly store their part of the vector in the shared memory
				if (LANEPOINTSTOVALIDBUCKETPOS) {
					l = __shfl(l, LANE-ENTRY_ID+d_sv_nints-1);
					if (l < OPENTILELEN) {
						// write part of vector to shared memory
						shared[OPENTILEOFFSET+l+ENTRY_ID] = tmp;
					}
				}
			}
			if(i < d_nrbuckets) {
				last_search_location = i - GLOBAL_WARP_ID;
			} else {
				last_search_location = 0;
			}
			if(found_new_state || i < d_nrbuckets) {
				WORKSCANRESULT = 1;
			}
		}
		__syncthreads();
		// if work has been retrieved, indicate this
		if (threadIdx.x == 0) {
			if (OPENTILECOUNT > 0) {
				(*d_contBFS) = 1;
			}
			if(SCAN && WORKSCANRESULT == 0 && d_newstate_flags[blockIdx.x] == 2) {
				// No new states were found by this block, save this information to prevent
				// unnecessary scanning later on
				d_newstate_flags[blockIdx.x] = 0;
			} else {
				WORKSCANRESULT = 0;
			}
			scan = 0;
		}
		// is the thread part of an 'active' group?
		offset1 = 0;
		offset2 = 0;
		if (threadIdx.x == 0) {
			OPENTILECOUNT = 0;
		}
		__syncthreads();
		if (THREADINGROUP) {
			// Is there work?
			if (ISSTATE(src_state)) {
				// Gather the required transition information for all states in the tile
				i = tex1Dfetch(tex_proc_offsets_start, GROUP_ID);
				// Determine process state
				GETSTATEVECTORSTATE(cont, src_state, GROUP_ID);
				// TODO: remove
				TMPVAR = cont;
				// Offset position
				index = cont/(INTSIZE/d_nbits_offset);
				pos = cont - (index*(INTSIZE/d_nbits_offset));
				tmp = tex1Dfetch(tex_proc_offsets, i+index);
				GETTRANSOFFSET(offset1, tmp, pos);
				if (pos == (INTSIZE/d_nbits_offset)-1) {
					tmp = tex1Dfetch(tex_proc_offsets, i+index+1);
					GETTRANSOFFSET(offset2, tmp, 0);
				}
				else {
					GETTRANSOFFSET(offset2, tmp, pos+1);
				}
			}
			if (GROUP_ID == 0) {
				// for later, when constructing successors for this state, set action counter to maximum
				THREADGROUPCOUNTER = (1 << d_bits_act);
				THREADGROUPPOR = 0;
			}
		}
		// iterate over the outgoing transitions of state 'cont'
		// variable cont is reused to indicate whether the buffer content of this thread still needs processing
		cont = 0;
		if (threadIdx.x == 0) {
			CONTINUE = 1;
		}
		__syncthreads();
		// while there is work to be done
		//int loopcounter = 0;
		outtrans_enabled = 0;
		local_action_counter = 0;
		while (CONTINUE == 1) {
			if (offset1 < offset2 || cont) {
				if (!cont) {
					// reset act
					act = (1 << (d_bits_act));
					// reset buffer of this thread
					for (l = 0; l < d_max_buf_ints; l++) {
						THREADBUFFERGROUPPOS(GROUP_ID, l) = 0;
					}
					// if not sync, store in hash table
					while (offset1 < offset2) {
						tmp = tex1Dfetch(tex_proc_trans, offset1);
						GETPROCTRANSSYNC(bitmask, tmp);
						if (bitmask == 0) {
							// no deadlock
							outtrans_enabled = 1;
							// construct state
							for (l = 0; l < d_sv_nints; l++) {
								tgt_state[l] = src_state[l];
							}
							for (l = 1; l <= NR_OF_STATES_IN_TRANSENTRY(GROUP_ID); l++) {
								GETPROCTRANSSTATE(pos, tmp, l, GROUP_ID);
								if (pos > 0) {
									SETSTATEVECTORSTATE(tgt_state, GROUP_ID, pos-1);
									// check for violation of safety property, if required
									if (d_property == SAFETY) {
										if (GROUP_ID == d_nr_procs-1) {
											// pos contains state id + 1
											// error state is state 1
											if (pos == 2) {
												// error state found
												(*d_property_violation) = 1;
											}
										}
									}
									// store tgt_state in cache; if i == d_shared_q_size, state was found, duplicate detected
									// if i == d_shared_q_size+1, cache is full, immediately store in global hash table
									k = STOREINCACHE(tgt_state, d_q, &bi);
									if (k == 2) {
										// cache time-out; store directly in global hash table
										if (FINDORPUT_SINGLE(tgt_state, d_q, d_newstate_flags) == 0) {
											// ERROR! hash table too full. Set CONTINUE to 2
											CONTINUE = 2;
										}
									} else if(local_action_counter != -1){
										// Only keep unique pointers in the buffer
										for(bk = bj = 0; bj < local_action_counter && bk == 0; bj++) {
											if((THREADBUFFERGROUPPOS(GROUP_ID, bj) & 0x7FFFFFFF) == bi) {
												bk = 1;
											}
										}
										if(bk == 0) {
											// Set the most-significant bit if we are not the owner
											THREADBUFFERGROUPPOS(GROUP_ID,local_action_counter) = bi | (k << 31);
											local_action_counter++;
										}
									}
								}
								else {
									break;
								}
							}
							offset1++;
						}
						else {
							break;
						}
					}

					// i is the current relative position in the buffer for this thread
					i = 0;
					if (offset1 < offset2) {
						GETPROCTRANSACT(act, tmp);
						// store transition entry
						THREADBUFFERGROUPPOS(GROUP_ID,i) = tmp;
						cont = 1;
						i++;
						offset1++;
						while (offset1 < offset2) {
							tmp = tex1Dfetch(tex_proc_trans, offset1);
							GETPROCTRANSACT(bitmask, tmp);
							if (act == bitmask) {
								THREADBUFFERGROUPPOS(GROUP_ID,i) = tmp;
								i++;
								offset1++;
							}
							else {
								break;
							}
						}
					} else {
						THREADGROUPPOR = 0x80000000 | d_nr_procs;
					}
				}
			}
			// All states following from local transitions
			// are now checked in global memory to see if
			// they are new.
			k = (d_shared_q_size-CACHEOFFSET)/d_sv_nints;
			int por_possible = 0;
			int32_t index = -1;
			int32_t c = 0;
			while(1) {
				if(__all(c >= local_action_counter)) {
					break;
				}
				if(c < local_action_counter) {
					index = THREADBUFFERGROUPPOS(GROUP_ID,c);
				}
				int active_lane = __ffs(__ballot((index & 0x80000000) == 0)) - 1;

				if(active_lane == -1) {
					c++;
					continue;
				}
				int cache_index = __shfl(index, active_lane);
				if(FIND_WARP((inttype*) &shared[CACHEOFFSET+cache_index], d_q) == 0) {
					por_possible |= active_lane == LANE;
				}
				if(active_lane == LANE || ((index & 0x80000000) != 0)) {
					c++;
					index = -1;
				}
			}
			__syncthreads();
			if(por_possible) {
				// At least one state following from a local
				// transition is new, report that POR
				// can be applied.
				atomicMin((unsigned int*)&THREADGROUPPOR, 0x80000000 | GROUP_ID);
			}
			__syncthreads();
			// Apply partial-order reduction by only retaining
			// new states from one thread as new.
			// Reset some variables to prevent further successor
			// generation.
			int do_por = 0;
			if(THREADINGROUP) {
				do_por = THREADGROUPPOR < (0x80000000 | d_nr_procs) && THREADGROUPPOR != 0;
			}
			__syncthreads();
			if(do_por) {
				// Cycle proviso is satisfied
				for(int32_t c = 0; c < local_action_counter; c++) {
					if((THREADGROUPPOR & 0x7FFFFFFF) != GROUP_ID) {
						SETOLDSTATE( &shared[CACHEOFFSET+ (THREADBUFFERGROUPPOS(GROUP_ID,c) & 0x7FFFFFFF)] );
					}
					THREADBUFFERGROUPPOS(GROUP_ID,c) = 0;
				}
				cont = 0;
				offset1 = offset2;
				if (THREADINGROUP && GROUP_ID == 0) {
					THREADGROUPCOUNTER = EXPLORATION_DONE;
					THREADGROUPPOR = 0;
				}
			}
			__syncthreads();
			if(do_por == 0){
				for(int32_t c = 0; c < local_action_counter; c++) {
					SETNEWSTATE( &shared[CACHEOFFSET+ (THREADBUFFERGROUPPOS(GROUP_ID,c) & 0x7FFFFFFF)] );
					THREADBUFFERGROUPPOS(GROUP_ID,c) = 0;
				}
			}
			local_action_counter = -1;
			// group leaders now need to set the counter to the next minimal action value.
			// To avoid bank conflicts afterwards when threads would need to read GETTHREADGROUPCOUNTER,
			// the leader disables the SYNC bit of transition entries for those threads which need
			// to perform work next. In this way, threads can determine locally that they should proceed
			// without reading the counter.
			__syncthreads();
			if (THREADINGROUP) {
				if (GROUP_ID == 0) {
					if (THREADGROUPCOUNTER != EXPLORATION_DONE) {
						pos = (1 << d_bits_act);
						for (i = 0; i < d_nr_procs; i++) {
							l = THREADBUFFERGROUPPOS(i,0);
							if (l != 0) {
								GETPROCTRANSACT(bitmask, l);
								if (THREADGROUPCOUNTER == (1 << d_bits_act)) {
									if (bitmask < pos) {
										pos = bitmask;
									}
								}
								else {
									if (bitmask < pos && bitmask > THREADGROUPCOUNTER) {
										pos = bitmask;
									}
								}
							}
						}
						// if pos = THREADGROUPCOUNTER, no more work for this state is required, now or in future iterations
						if (pos == THREADGROUPCOUNTER) {
							THREADGROUPCOUNTER = EXPLORATION_DONE;
						}
						// store the value
						if (pos < (1 << d_bits_act)) {
							THREADGROUPCOUNTER = pos;
						}
						// notify threads to work
						for (i = 0; i < d_nr_procs; i++) {
							l = THREADBUFFERGROUPPOS(i,0);
							if (l != 0) {
								GETPROCTRANSACT(bitmask, l);
								if (bitmask == THREADGROUPCOUNTER) {
									// notify
									SETPROCTRANSSYNC(THREADBUFFERGROUPPOS(i,0),0);
								}
							}
						}
					}
				}
			}
			__syncthreads();
			// only active threads should do something
			if (cont) {
				// Now, we have obtained the info needed to combine process transitions
				// if the sync bit has been disabled, come into action, creating successors
				GETPROCTRANSSYNC(index, THREADBUFFERGROUPPOS(GROUP_ID,0));
				if (index == 0) {
					// syncbits Offset position
					i = act/(INTSIZE/d_nbits_syncbits_offset);
					pos = act - (i*(INTSIZE/d_nbits_syncbits_offset));
					l = tex1Dfetch(tex_syncbits_offsets, i);
					GETSYNCOFFSET(sync_offset1, l, pos);
					if (pos == (INTSIZE/d_nbits_syncbits_offset)-1) {
						l = tex1Dfetch(tex_syncbits_offsets, i+1);
						GETSYNCOFFSET(sync_offset2, l, 0);
					}
					else {
						GETSYNCOFFSET(sync_offset2, l, pos+1);
					}
					// iterate through the relevant syncbit filters
					tmp = 1;
					for (;sync_offset1 < sync_offset2 && tmp; sync_offset1++) {
						index = tex1Dfetch(tex_syncbits, sync_offset1);
						for (i = 0; i < (INTSIZE/d_nr_procs); i++) {
							GETSYNCRULE(tmp, index, i);
							if (tmp != 0) {
								OWNSSYNCRULE(bitmask, tmp, GROUP_ID);
							}
							else {
								bitmask = 0;
							}
							if (bitmask) {
								// start combining entries in the buffer to create target states
								// if sync rule applicable, construct the first successor
								// copy src_state into tgt_state
								SYNCRULEISAPPLICABLE(l, tmp, act);
								if (l) {
									// source state is not a deadlock
									outtrans_enabled = 1;
									for (pos = 0; pos < d_sv_nints; pos++) {
										tgt_state[pos] = src_state[pos];
									}
									// construct first successor
									for (pos = 0; pos < d_nr_procs; pos++) {
										if (GETBIT(pos, tmp)) {
											// get first state
											GETPROCTRANSSTATE(k, THREADBUFFERGROUPPOS(pos,0), 1, pos);
											SETSTATEVECTORSTATE(tgt_state, pos, k-1);
										}
									}
									SETNEWSTATE(tgt_state);
									// while we keep getting new states, store them
									while (ISNEWSTATE(tgt_state)) {
										// check for violation of safety property, if required
										if (d_property == SAFETY) {
											GETSTATEVECTORSTATE(pos, tgt_state, d_nr_procs-1);
											if (pos == 1) {
												// error state found
												(*d_property_violation) = 1;
											}
										}

										// store tgt_state in cache; if i == d_shared_q_size, state was found, duplicate detected
										// if i == d_shared_q_size+1, cache is full, immediately store in global hash table
										TMPVAR = STOREINCACHE(tgt_state, d_q, &bitmask);
										if (TMPVAR == 2) {
											// cache time-out; store directly in global hash table
											if (FINDORPUT_SINGLE(tgt_state, d_q, d_newstate_flags) == 0) {
												// ERROR! hash table too full. Set CONTINUE to 2
												CONTINUE = 2;
											}
										}
										// get next successor
										for (pos = d_nr_procs-1; pos > (int) GROUP_ID-1; pos--) {
											if (GETBIT(pos,tmp)) {
												GETSTATEVECTORSTATE(cont, tgt_state, pos);
												int st = 0;
												for (k = 0; k < d_max_buf_ints; k++) {
													for (l = 1; l <= NR_OF_STATES_IN_TRANSENTRY(pos); l++) {
														GETPROCTRANSSTATE(st, THREADBUFFERGROUPPOS(pos,k), l, pos);
														if (cont == (st-1)) {
															break;
														}
													}
													if (cont == (st-1)) {
														break;
													}
												}
												// Assumption: element has been found (otherwise, 'last' was not a valid successor)
												// Try to get the next element
												if (l == NR_OF_STATES_IN_TRANSENTRY(pos)) {
													if (k >= d_max_buf_ints-1) {
														st = 0;
													}
													else {
														k++;
														l = 1;
													}
												}
												else {
													l++;
												}
												// Retrieve next element, insert it in 'tgt_state' if it is not 0, and return result, otherwise continue
												if (st != 0) {
													GETPROCTRANSSTATE(st, THREADBUFFERGROUPPOS(pos,k), l, pos);
													if (st > 0) {
														SETSTATEVECTORSTATE(tgt_state, pos, st-1);
														SETNEWSTATE(tgt_state);
														break;
													}
												}
												// else, set this process state to first one, and continue to next process
												GETPROCTRANSSTATE(st, THREADBUFFERGROUPPOS(pos,0), 1, pos);
												SETSTATEVECTORSTATE(tgt_state, pos, st-1);
											}
										}
										// did we find a successor? if not, set tgt_state to old
										if (pos == (int) GROUP_ID-1) {
											SETOLDSTATE(tgt_state);
										}
									}
								}
							}
						}
					}
					cont = 0;
				}
			}
			// finished an iteration of adding states.
			// Is there still work? (is another iteration required?)
			if (threadIdx.x == 0) {
				if (CONTINUE != 2) {
					CONTINUE = 0;
				}
			}
			__syncthreads();
			if (THREADINGROUP) {
				if ((offset1 < offset2) || cont) {
					if (CONTINUE != 2) {
						CONTINUE = 1;
					}
				}
			}
			// FOR TEST PURPOSES!
//			if (threadIdx.x == 0) {
//				CONTINUE++;
//			}
			__syncthreads();
		} // END WHILE CONTINUE == 1
		// have we encountered a deadlock state?
		// we use the shared memory to communicate this to the group leaders
		if (d_property == DEADLOCK) {
			if (THREADINGROUP) {
				if (ISSTATE(src_state)) {
					THREADBUFFERGROUPPOS(GROUP_ID, 0) = outtrans_enabled;
					// group leader collects results
					l = 0;
					if (GROUP_ID == 0) {
						for (i = 0; i < d_nr_procs; i++) {
							l += THREADBUFFERGROUPPOS(i, 0);
						}
						if (l == 0) {
							// deadlock state found
							(*d_property_violation) = 1;
						}
					}
				}
			}
		}
		// Reset the open queue tile
		if (threadIdx.x < OPENTILELEN) {
			shared[OPENTILEOFFSET+threadIdx.x] = EMPTYVECT32;
		}
		if (threadIdx.x == 0) {
			OPENTILECOUNT = 0;
		}
		__syncthreads();
		// start scanning the local cache and write results to the global hash table
		k = (d_shared_q_size-CACHEOFFSET)/d_sv_nints;
		int c;
		for (i = WARP_ID; i * WARPSIZE < k; i += (blockDim.x / WARPSIZE)) {
			int have_new_state = i * WARPSIZE + LANE < k && ISNEWSTATE(&shared[CACHEOFFSET+(i*WARPSIZE+LANE)*d_sv_nints]);
			while (c = __ballot(have_new_state)) {
				int active_lane = __ffs(c) - 1;
				if(FINDORPUT_WARP((inttype*) &shared[CACHEOFFSET + (i*WARPSIZE+active_lane)*d_sv_nints], d_q, d_newstate_flags) == 0) {
					CONTINUE = 2;
				}
				if (LANE == active_lane) {
					have_new_state = 0;
				}
			}
		}
		__syncthreads();
		// Ready to start next iteration, if error has not occurred
		if (threadIdx.x == 0) {
			if (CONTINUE == 2) {
				(*d_contBFS) = 2;
				ITERATIONS = d_kernel_iters;
			}
			else {
				ITERATIONS++;
			}
			CONTINUE = 0;
		}
		__syncthreads();
	}

	//Copy the work tile to global mem
	if (threadIdx.x < OPENTILELEN) {
		d_q[d_nrbuckets*WARPSIZE + (OPENTILELEN+1) * blockIdx.x + threadIdx.x] = shared[OPENTILEOFFSET+threadIdx.x];
	}
	if(threadIdx.x == 0) {
		d_q[d_nrbuckets*WARPSIZE + (OPENTILELEN+1) * blockIdx.x + OPENTILELEN] = OPENTILECOUNT;
	}
}

/**
 * Host function that prepares data array and passes it to the CUDA kernel.
 */
int main(int argc, char** argv) {
	FILE *fp;
	inttype nr_procs, bits_act, bits_statevector, sv_nints, nr_trans, proc_nrstates, nbits_offset, max_buf_ints, nr_syncbits_offsets, nr_syncbits, nbits_syncbits_offset;
	inttype *bits_state, *firstbit_statevector, *proc_offsets, *proc_trans, *proc_offsets_start, *syncbits_offsets, *syncbits;
	inttype contBFS;
	char stmp[BUFFERSIZE], fn[50];
	// to store constants for closed set hash functions
	int h[NR_HASH_FUNCTIONS*2];
	// size of global hash table
	size_t q_size = 0;
	PropertyStatus check_property = NONE;
	// nr of iterations in single kernel run
	int kernel_iters = KERNEL_ITERS;
	int nblocks = NR_OF_BLOCKS;
	int nthreadsperblock = BLOCK_SIZE;
	// level of verbosity (1=print level progress)
	int verbosity = 0;
	// clock to measure time
	clock_t start, stop;
	double runtime = 0.0;

	// Start timer
	assert((start = clock())!=-1);

	hipDeviceProp_t prop;
	int nDevices;

	// GPU side versions of the input
	inttype *d_bits_state, *d_firstbit_statevector, *d_proc_offsets_start, *d_proc_offsets, *d_proc_trans, *d_syncbits_offsets, *d_syncbits, *d_h;
	// flag to keep track of progress and whether hash table errors occurred (value==2)
	inttype *d_contBFS;
	// flags to track which blocks have new states
	inttype *d_newstate_flags;
	// flag to keep track of property verification outcome
	inttype *d_property_violation;

	// GPU datastructures for calculation
	inttype *d_q;

	if (argc == 1) {
		fprintf(stderr, "ERROR: No input network given!\n");
		exit(1);
	}

	strcpy(fn, argv[1]);
	strcat(fn, ".gpf");

	int i = 2;
	while (i < argc) {
		printf ("%s\n", argv[i]);
		if (!strcmp(argv[i],"-k")) {
			// if nr. of iterations per kernel run is given, store it
			kernel_iters = atoi(argv[i+1]);
			i += 2;
		}
		else if (!strcmp(argv[i],"-b")) {
			// store nr of blocks to be used
			nblocks = atoi(argv[i+1]);
			i += 2;
		}
		else if (!strcmp(argv[i],"-t")) {
			// store nr of threads per block to be used
			nthreadsperblock = atoi(argv[i+1]);
			i += 2;
		}
		else if (!strcmp(argv[i],"-q")) {
			// store hash table size
			q_size = atoi(argv[i+1]);
			i += 2;
		}
		else if (!strcmp(argv[i],"-v")) {
			// store verbosity level
			verbosity = atoi(argv[i+1]);
			if (verbosity > 3) {
				verbosity = 3;
			}
			i += 2;
		}
		else if (!strcmp(argv[i],"-d")) {
			// check for deadlocks
			check_property = DEADLOCK;
			i += 1;
		}
		else if (!strcmp(argv[i],"-p")) {
			// check a property
			check_property = SAFETY;
			i += 1;
		}
	}

	fp = fopen(fn, "r");
	if (fp) {
		// Read the input
		fgets(stmp, BUFFERSIZE, fp);
		if (check_property == SAFETY) {
			i = atoi(stmp);
			fprintf(stdout, "Property to check is ");
			if (i == 0) {
				fprintf(stdout, "not ");
			}
			fprintf(stdout, "a liveness property\n");
			if (i == 1) {
				check_property = LIVENESS;
			}
		}
		fgets(stmp, BUFFERSIZE, fp);
		nr_procs = atoi(stmp);
		fprintf(stdout, "nr of procs: %d\n", nr_procs);
		fgets(stmp, BUFFERSIZE, fp);
		bits_act = atoi(stmp);
		fprintf(stdout, "nr of bits for transition label: %d\n", bits_act);
		fgets(stmp, BUFFERSIZE, fp);
		proc_nrstates = atoi(stmp);
		fprintf(stdout, "min. nr. of proc. states that fit in 32-bit integer: %d\n", proc_nrstates);
		fgets(stmp, BUFFERSIZE, fp);
		bits_statevector = atoi(stmp);
		fprintf(stdout, "number of bits needed for a state vector: %d\n", bits_statevector);
		firstbit_statevector = (inttype*) malloc(sizeof(inttype)*(nr_procs+1));
		for (int i = 0; i <= nr_procs; i++) {
			fgets(stmp, BUFFERSIZE, fp);
			firstbit_statevector[i] = atoi(stmp);
			fprintf(stdout, "statevector offset %d: %d\n", i, firstbit_statevector[i]);
		}
		// determine the number of integers needed for a state vector
		sv_nints = (bits_statevector+31) / INTSIZE;
		bits_state = (inttype*) malloc(sizeof(inttype)*nr_procs);
		for (int i = 0; i < nr_procs; i++) {
			fgets(stmp, BUFFERSIZE, fp);
			bits_state[i] = atoi(stmp);
			fprintf(stdout, "bits for states of process LTS %d: %d\n", i, bits_state[i]);
		}
		fgets(stmp, BUFFERSIZE, fp);
		nbits_offset = atoi(stmp);
		fprintf(stdout, "size of offset in process LTSs: %d\n", nbits_offset);
		fgets(stmp, BUFFERSIZE, fp);
		max_buf_ints = atoi(stmp);
		fprintf(stdout, "maximum label-bounded branching factor: %d\n", max_buf_ints);
		proc_offsets_start = (inttype*) malloc(sizeof(inttype)*(nr_procs+1));
		for (int i = 0; i <= nr_procs; i++) {
			fgets(stmp, BUFFERSIZE, fp);
			proc_offsets_start[i] = atoi(stmp);
		}
		proc_offsets = (inttype*) malloc(sizeof(inttype)*proc_offsets_start[nr_procs]);
		for (int i = 0; i < proc_offsets_start[nr_procs]; i++) {
			fgets(stmp, BUFFERSIZE, fp);
			proc_offsets[i] = atoi(stmp);
		}
		fgets(stmp, BUFFERSIZE, fp);
		nr_trans = atoi(stmp);
		fprintf(stdout, "total number of transition entries in network: %d\n", nr_trans);
		proc_trans = (inttype*) malloc(sizeof(inttype)*nr_trans);
		for (int i = 0; i < nr_trans; i++) {
			fgets(stmp, BUFFERSIZE, fp);
			proc_trans[i] = atoi(stmp);
		}

		fgets(stmp, BUFFERSIZE, fp);
		nbits_syncbits_offset = atoi(stmp);
		//fprintf(stdout, "size of offset in sync rules: %d\n", nbits_syncbits_offset);
		fgets(stmp, BUFFERSIZE, fp);
		nr_syncbits_offsets = atoi(stmp);
		syncbits_offsets = (inttype*) malloc(sizeof(inttype)*nr_syncbits_offsets);
		for (int i = 0; i < nr_syncbits_offsets; i++) {
			fgets(stmp, BUFFERSIZE, fp);
			syncbits_offsets[i] = atoi(stmp);
			//fprintf(stdout, "syncbits offset %d: %d\n", i, syncbits_offsets[i]);
		}
		fgets(stmp, BUFFERSIZE, fp);
		nr_syncbits = atoi(stmp);
		syncbits = (inttype*) malloc(sizeof(inttype)*nr_syncbits);
		for (int i = 0; i < nr_syncbits; i++) {
			fgets(stmp, BUFFERSIZE, fp);
			syncbits[i] = atoi(stmp);
			//fprintf(stdout, "syncbits %d: %d\n", i, syncbits[i]);
		}
	}
	else {
		fprintf(stderr, "ERROR: input network does not exist!\n");
		exit(1);
	}

	// Randomly define the closed set hash functions
//	srand(time(NULL));
//	for (int i = 0; i < NR_HASH_FUNCTIONS*2; i++) {
//		h[i] = rand();
//	}
	// TODO: make random again
	h[0] = 483319424;
	h[1] = 118985421;
	h[2] = 1287157904;
	h[3] = 1162380012;
	h[4] = 1231274815;
	h[5] = 1344969351;
	h[6] = 527997957;
	h[7] = 735456672;
	h[8] = 1774251664;
	h[9] = 23102285;
	h[10] = 2089529600;
	h[11] = 2083003102;
	h[12] = 908039861;
	h[13] = 1913855526;
	h[14] = 1515282600;
	h[15] = 1691511413;

	// continue flags
	contBFS = 1;

	// Query the device properties and determine data structure sizes
	hipGetDeviceCount(&nDevices);
	if (nDevices == 0) {
		fprintf (stderr, "ERROR: No CUDA compatible GPU detected!\n");
		exit(1);
	}
	hipGetDeviceProperties(&prop, 0);
	fprintf (stdout, "global mem: %lu\n", (uint64_t) prop.totalGlobalMem);
	fprintf (stdout, "shared mem per block: %d\n", (int) prop.sharedMemPerBlock);
	fprintf (stdout, "max. threads per block: %d\n", (int) prop.maxThreadsPerBlock);
	fprintf (stdout, "max. grid size: %d\n", (int) prop.maxGridSize[0]);
	fprintf (stdout, "nr. of multiprocessors: %d\n", (int) prop.multiProcessorCount);

	// determine actual nr of blocks
	nblocks = MAX(1,MIN(prop.maxGridSize[0],nblocks));

	// Allocate memory on GPU
	cudaMallocCount((void **) &d_contBFS, sizeof(inttype));
	cudaMallocCount((void **) &d_property_violation, sizeof(inttype));
	cudaMallocCount((void **) &d_h, NR_HASH_FUNCTIONS*2*sizeof(inttype));
	cudaMallocCount((void **) &d_bits_state, nr_procs*sizeof(inttype));
	cudaMallocCount((void **) &d_firstbit_statevector, (nr_procs+1)*sizeof(inttype));
	cudaMallocCount((void **) &d_proc_offsets_start, (nr_procs+1)*sizeof(inttype));
	cudaMallocCount((void **) &d_proc_offsets, proc_offsets_start[nr_procs]*sizeof(inttype));
	cudaMallocCount((void **) &d_proc_trans, nr_trans*sizeof(inttype));
	cudaMallocCount((void **) &d_syncbits_offsets, nr_syncbits_offsets*sizeof(inttype));
	cudaMallocCount((void **) &d_syncbits, nr_syncbits*sizeof(inttype));
	cudaMallocCount((void **) &d_newstate_flags, nblocks*sizeof(inttype));

	// Copy data to GPU
	CUDA_CHECK_RETURN(hipMemcpy(d_contBFS, &contBFS, sizeof(inttype), hipMemcpyHostToDevice))
	CUDA_CHECK_RETURN(hipMemcpy(d_h, h, NR_HASH_FUNCTIONS*2*sizeof(inttype), hipMemcpyHostToDevice))
	CUDA_CHECK_RETURN(hipMemcpy(d_bits_state, bits_state, nr_procs*sizeof(inttype), hipMemcpyHostToDevice))
	CUDA_CHECK_RETURN(hipMemcpy(d_firstbit_statevector, firstbit_statevector, (nr_procs+1)*sizeof(inttype), hipMemcpyHostToDevice))
	CUDA_CHECK_RETURN(hipMemcpy(d_proc_offsets_start, proc_offsets_start, (nr_procs+1)*sizeof(inttype), hipMemcpyHostToDevice))
	CUDA_CHECK_RETURN(hipMemcpy(d_proc_offsets, proc_offsets, proc_offsets_start[nr_procs]*sizeof(inttype), hipMemcpyHostToDevice))
	CUDA_CHECK_RETURN(hipMemcpy(d_proc_trans, proc_trans, nr_trans*sizeof(inttype), hipMemcpyHostToDevice))
	CUDA_CHECK_RETURN(hipMemcpy(d_syncbits_offsets, syncbits_offsets, nr_syncbits_offsets*sizeof(inttype), hipMemcpyHostToDevice))
	CUDA_CHECK_RETURN(hipMemcpy(d_syncbits, syncbits, nr_syncbits*sizeof(inttype), hipMemcpyHostToDevice))
	CUDA_CHECK_RETURN(hipMemset(d_newstate_flags, 0, nblocks*sizeof(inttype)));

	// Bind data to textures
	hipBindTexture(NULL, tex_proc_offsets_start, d_proc_offsets_start, (nr_procs+1)*sizeof(inttype));
	hipBindTexture(NULL, tex_proc_offsets, d_proc_offsets, proc_offsets_start[nr_procs]*sizeof(inttype));
	hipBindTexture(NULL, tex_proc_trans, d_proc_trans, nr_trans*sizeof(inttype));
	hipBindTexture(NULL, tex_syncbits_offsets, d_syncbits_offsets, nr_syncbits_offsets*sizeof(inttype));
	hipBindTexture(NULL, tex_syncbits, d_syncbits, nr_syncbits*sizeof(inttype));

	size_t available, total;
	hipMemGetInfo(&available, &total);
	if (q_size == 0) {
		q_size = total / sizeof(inttype);
	}
	size_t el_per_Mb = Mb / sizeof(inttype);


	while(hipMalloc((void**)&d_q,  q_size * sizeof(inttype)) == hipErrorOutOfMemory)	{
		q_size -= el_per_Mb;
		if( q_size  < el_per_Mb) {
			// signal no free memory
			break;
		}
	}

	fprintf (stdout, "global mem queue size: %lu, number of entries: %lu\n", q_size*sizeof(inttype), (indextype) q_size);

	inttype shared_q_size = (int) prop.sharedMemPerBlock / sizeof(inttype);
	fprintf (stdout, "shared mem queue size: %lu, number of entries: %u\n", shared_q_size*sizeof(inttype), shared_q_size);
	fprintf (stdout, "nr. of blocks: %d, block size: %d, nr of kernel iterations: %d\n", nblocks, nthreadsperblock, kernel_iters);

	// copy symbols
	inttype tablesize = q_size - nblocks * (sv_nints*(nthreadsperblock/nr_procs)+1);
	inttype nrbuckets = tablesize / WARPSIZE;
	hipMemcpyToSymbol(HIP_SYMBOL(d_nrbuckets), &nrbuckets, sizeof(inttype));
	hipMemcpyToSymbol(HIP_SYMBOL(d_shared_q_size), &shared_q_size, sizeof(inttype));
	hipMemcpyToSymbol(HIP_SYMBOL(d_nr_procs), &nr_procs, sizeof(inttype));
	hipMemcpyToSymbol(HIP_SYMBOL(d_max_buf_ints), &max_buf_ints, sizeof(inttype));
	hipMemcpyToSymbol(HIP_SYMBOL(d_sv_nints), &sv_nints, sizeof(inttype));
	hipMemcpyToSymbol(HIP_SYMBOL(d_bits_act), &bits_act, sizeof(inttype));
	hipMemcpyToSymbol(HIP_SYMBOL(d_nbits_offset), &nbits_offset, sizeof(inttype));
	hipMemcpyToSymbol(HIP_SYMBOL(d_nbits_syncbits_offset), &nbits_syncbits_offset, sizeof(inttype));
	hipMemcpyToSymbol(HIP_SYMBOL(d_kernel_iters), &kernel_iters, sizeof(inttype));
	hipMemcpyToSymbol(HIP_SYMBOL(d_property), &check_property, sizeof(inttype));

	// init the queue
	init_queue<<<nblocks, nthreadsperblock>>>(d_q, q_size);
	store_initial<<<1,1>>>(d_q, d_h, d_newstate_flags,nthreadsperblock,nblocks);
	for (int i = 0; i < 2*NR_HASH_FUNCTIONS; i++) {
		fprintf (stdout, "hash constant %d: %d\n", i, h[i]);
	}
	FIRSTHASHHOST(i);
	fprintf (stdout, "hash of initial state: %d\n", i);

	inttype zero = 0;
	inttype *q_test = (inttype*) malloc(sizeof(inttype)*tablesize);
	int j = 0;
	inttype scan = 0;
	CUDA_CHECK_RETURN(hipMemcpy(d_property_violation, &zero, sizeof(inttype), hipMemcpyHostToDevice))
	inttype property_violation = 0;
	while (contBFS == 1) {
		CUDA_CHECK_RETURN(hipMemcpy(d_contBFS, &zero, sizeof(inttype), hipMemcpyHostToDevice))
		gather<<<nblocks, nthreadsperblock, shared_q_size*sizeof(inttype)>>>(d_q, d_h, d_bits_state, d_firstbit_statevector, d_proc_offsets_start,
																		d_proc_offsets, d_proc_trans, d_syncbits_offsets, d_syncbits, d_contBFS, d_property_violation, d_newstate_flags, scan);
		// copy progress result
		//CUDA_CHECK_RETURN(hipGetLastError());
		CUDA_CHECK_RETURN(hipDeviceSynchronize());
		CUDA_CHECK_RETURN(hipMemcpy(&contBFS, d_contBFS, sizeof(inttype), hipMemcpyDeviceToHost))
		if (check_property > 0) {
			CUDA_CHECK_RETURN(hipMemcpy(&property_violation, d_property_violation, sizeof(inttype), hipMemcpyDeviceToHost))
			if (property_violation == 1) {
				contBFS = 0;
			}
		}
		if (verbosity > 0) {
			if (verbosity == 1) {
				printf ("%d\n", j++);
			}
			else if (verbosity == 2) {
				hipMemcpy(q_test, d_q, tablesize*sizeof(inttype), hipMemcpyDeviceToHost);
				count_local_queue(q_test, tablesize, firstbit_statevector, nr_procs, sv_nints);
			}
			else if (verbosity == 3) {
				hipMemcpy(q_test, d_q, tablesize*sizeof(inttype), hipMemcpyDeviceToHost);
				print_local_queue(stdout, q_test, tablesize, firstbit_statevector, nr_procs, sv_nints);
			}
		}
		scan = 1;
	}
	// determine runtime
	stop = clock();
	runtime = (double) (stop-start)/CLOCKS_PER_SEC;
	fprintf (stdout, "Run time: %f\n", runtime);

	if (property_violation == 1) {
		switch (check_property) {
			case DEADLOCK:
				printf ("deadlock detected!\n");
				break;
			case SAFETY:
				printf ("safety property violation detected!\n");
				break;
			case LIVENESS:
				printf ("liveness property violation detected!\n");
				break;
		}
	}
	// report error if required
	if (contBFS == 2) {
		fprintf (stderr, "ERROR: problem with hash table\n");
	}
	count_queue(d_q, tablesize, firstbit_statevector, nr_procs, sv_nints);

	// Debugging functionality: print states to file
//	FILE* fout;
//	fout = fopen("/tmp/gpuexplore.debug", "w");
//	hipMemcpy(q_test, d_q, tablesize*sizeof(inttype), hipMemcpyDeviceToHost);
//	print_local_queue(fout, q_test, tablesize, firstbit_statevector, nr_procs, sv_nints);
//	fclose(fout);

	CUDA_CHECK_RETURN(hipDeviceSynchronize());	// Wait for the GPU launched work to complete
	//CUDA_CHECK_RETURN(hipGetLastError());

	return 0;
}
